#include "hip/hip_runtime.h"
#include <hip/hip_vector_types.h>

#ifndef __HIPCC__  
#define __HIPCC__
#endif

#ifndef  __CUDART_API_PER_THREAD_DEFAULT_STREAM
#define  __CUDART_API_PER_THREAD_DEFAULT_STREAM
#endif
#include <hip/hip_runtime_api.h>


#ifndef __CUDACC_RTC__ 
#define __CUDACC_RTC__

#endif

#include <algorithm>
#include <iostream>

#include ""
#include "Convolution.cuh"
#include <random>
#include <cmath>

#ifndef __HIPCC__  
#define __HIPCC__
#endif

#ifndef __CUDACC_RTC__ 
#define __CUDACC_RTC__
#endif

#include <hip/device_functions.h>
using namespace std;

__global__ void ConvolutionKernel(float* d_Input, float* d_Filter, float* d_Output, int _outputWidth, int _convolutionInputWidth, int filterHeight, int filterWidth)
{
	//These define indecies for output matrix;
	int outputRowIndex = blockIdx.x;
	int outputColumnIndex = threadIdx.x;

	//Starts from "top left" of current block of pixels being processed
	int inputRowIndex = blockIdx.x;
	int inputColumnIndex = threadIdx.x;

	int outputArrayIndex = outputRowIndex * _outputWidth + outputColumnIndex;

	int inputArrayIndex = 0;

	float result = 0;
	int filterIndex = 0;
	int temp = 0;
	for (int row = 0; row < filterHeight; row++)
	{
		inputColumnIndex = threadIdx.x;

		for (int col = 0; col < filterWidth; col++)
		{
			inputArrayIndex = inputRowIndex * _convolutionInputWidth + inputColumnIndex;

			result += d_Input[inputArrayIndex] * d_Filter[filterIndex];
			filterIndex++;
			inputColumnIndex += 1;
		}
		inputRowIndex += 1;

	}

	d_Output[outputArrayIndex] = result;
}

__global__ void LayerConvolutionKernel(float** _inputs, float** _filters, float** _outputs, float** _biases, int _numberOfInputs, int _outputWidth, int _inputWidth, int _filterSize)
{
	int inputSelectionIndex = threadIdx.x;
	int filterSelectionIndex = blockIdx.z * _numberOfInputs + threadIdx.x;
	int biasSelectionIndex = blockIdx.z * _numberOfInputs + threadIdx.x;
	int outputSelectionIndex = blockIdx.z;

	float* selectedInput = _inputs[inputSelectionIndex];
	float* selectedFilter = _filters[filterSelectionIndex];
	float* selectedBias = _biases[biasSelectionIndex];
	float* selectedOutput = _outputs[outputSelectionIndex];

	int inputStartReadRowIndex = blockIdx.x;
	int inputStartReadColumnIndex = blockIdx.y;

	int outputWriteRowIndex = blockIdx.x;
	int outputWriteColumnIndex = blockIdx.y;

	int inputArrayIndex = 0;

	int outputArrayIndex = outputWriteRowIndex * _outputWidth + outputWriteColumnIndex;

	float result = 0;
	int filterIndex = 0;
	int temp = 0;
	for (int row = 0; row < _filterSize; row++)
	{
		inputStartReadColumnIndex = blockIdx.y;

		for (int col = 0; col < _filterSize; col++)
		{
			inputArrayIndex = inputStartReadRowIndex * _inputWidth + inputStartReadColumnIndex;

			result += selectedInput[inputArrayIndex] * selectedFilter[filterIndex];
			filterIndex++;
			inputStartReadColumnIndex += 1;
		}
		inputStartReadRowIndex += 1;
	}


	result += selectedBias[outputArrayIndex];
	atomicAdd(&selectedOutput[outputArrayIndex], result);
};

__global__ void LayerBackpropConvolutionKernel(float** _inputs, float** _filters, float** _outputs, int _numberOfOutputs, int _outputWidth, int _inputWidth, int _filterSize)
{
	int inputSelectionIndex = threadIdx.x;
	int filterSelectionIndex = blockIdx.z * _numberOfOutputs + threadIdx.x;
	int outputSelectionIndex = blockIdx.z;

	float* selectedInput = _inputs[inputSelectionIndex];
	float* selectedFilter = _filters[filterSelectionIndex];
	float* selectedOutput = _outputs[outputSelectionIndex];

	int inputStartReadRowIndex = blockIdx.x;
	int inputStartReadColumnIndex = blockIdx.y;

	int outputWriteRowIndex = blockIdx.x;
	int outputWriteColumnIndex = blockIdx.y;

	int inputArrayIndex = 0;

	int outputArrayIndex = outputWriteRowIndex * _outputWidth + outputWriteColumnIndex;

	float result = 0;
	int filterIndex = 0;
	int temp = 0;
	for (int row = 0; row < _filterSize; row++)
	{
		inputStartReadColumnIndex = blockIdx.y;

		for (int col = 0; col < _filterSize; col++)
		{
			inputArrayIndex = inputStartReadRowIndex * _inputWidth + inputStartReadColumnIndex;

			result += selectedInput[inputArrayIndex] * selectedFilter[filterIndex];
			filterIndex++;
			inputStartReadColumnIndex += 1;
		}
		inputStartReadRowIndex += 1;
	}
	if (blockIdx.z == 0 && threadIdx.x == 0)
	{

	}
	//selectedOutput[0] = _numberOfOutputs;




	atomicAdd(&selectedOutput[outputArrayIndex], result);
}

__global__ void LayerFilterBackpropKernel(float** _inputs, float** _outputs, float** _filterEquivalents, int _inputsWidth, int _outputsWidth, int _filterEquivsHeight, int _filterEquivsWidth, int _numberOfInputs)
{
	int inputSelectionIndex = threadIdx.x;
	int filterSelectionIndex = blockIdx.z;
	int outputSelectionIndex = blockIdx.z * _numberOfInputs + threadIdx.x;

	float* selectedInput = _inputs[inputSelectionIndex];
	float* selectedFilter = _filterEquivalents[filterSelectionIndex];
	float* selectedOutput = _outputs[outputSelectionIndex];

	int inputStartReadRowIndex = blockIdx.x;
	int inputStartReadColumnIndex = blockIdx.y;

	int outputWriteRowIndex = blockIdx.x;
	int outputWriteColumnIndex = blockIdx.y;

	int inputArrayIndex = 0;

	int outputArrayIndex = outputWriteRowIndex * _outputsWidth + outputWriteColumnIndex;

	float result = 0;
	int filterIndex = 0;

	for (int row = 0; row < _filterEquivsHeight; row++)
	{
		inputStartReadColumnIndex = blockIdx.y;

		for (int col = 0; col < _filterEquivsWidth; col++)
		{
			inputArrayIndex = inputStartReadRowIndex * _inputsWidth + inputStartReadColumnIndex;

			result += selectedInput[inputArrayIndex] * selectedFilter[filterIndex];
			filterIndex++;
			inputStartReadColumnIndex += 1;
		}
		inputStartReadRowIndex += 1;
	}

	selectedOutput[outputArrayIndex] = result;
}

__global__ void LayerFilterFlipKernel(float** _inputFilters, float** _outputFilters, int _filterSize)
{
	float* filterToFlip = _inputFilters[blockIdx.x];
	int filterArraySize = _filterSize * _filterSize;
	float* flippedOutput = _outputFilters[blockIdx.x];
	int k = 0;

	//Loop from back and assign value to new array
	for (int i = filterArraySize - 1; i >= 0; ) {
		flippedOutput[k++] = filterToFlip[i--];
	}
}

__global__ void LayerPaddingKernel(float** __inputs, float** _outputs, int _unpaddedWidth, int _paddedWidth)
{
	int inputSelectionIndex = blockIdx.y;
	int outputSelectionIndex = blockIdx.y;

	float* selectedInput = __inputs[inputSelectionIndex];
	float* selectedOutput = _outputs[outputSelectionIndex];

	int rowWriteIndex = (blockIdx.x + 1) * 2 + threadIdx.x;
	int columnWriteIndex = 2;

	int inputRowReadIndex = (blockIdx.x * 2) + threadIdx.x;
	int inputColumnReadIndex = 0;

	int arrayPosition = rowWriteIndex * _paddedWidth + columnWriteIndex;
	int inputArrayPosition = inputRowReadIndex * _unpaddedWidth + inputColumnReadIndex;

	int var = 0;
	for (int i = 0; i < _unpaddedWidth; i++)
	{
		selectedOutput[arrayPosition] = selectedInput[inputArrayPosition];
		arrayPosition++;
		inputArrayPosition++;
	}
}

__global__ void ConvolutionPaddingKernel(float* d_UnpaddedInput, float* d_Output, int _paddedInputWidth, int _unpaddedInputWidth, int _unpaddedInputHeight)
{
	int rowWriteIndex = (blockIdx.x + 1) * 2 + threadIdx.x;
	int columnWriteIndex = 2;

	int inputRowReadIndex = (blockIdx.x * 2) + threadIdx.x;
	int inputColumnReadIndex = 0;

	int arrayPosition = rowWriteIndex * _paddedInputWidth + columnWriteIndex;
	int inputArrayPosition = inputRowReadIndex * _unpaddedInputWidth + inputColumnReadIndex;

	int var = 0;
	for (int i = 0; i < _unpaddedInputWidth; i++)
	{
		d_Output[arrayPosition] = d_UnpaddedInput[inputArrayPosition];
		arrayPosition++;
		inputArrayPosition++;
	}
}


__global__ void FilterUpdateKernel(float** _currentFilters, float** _filterGradients, float** _VMatricies, float** _SMatricies, float** _V_CorrectedMatrices, float** _S_CorrectedMatricies, int _filterSize, float _HyperParam_Beta1, float _HyperParam_Beta2, float _HyperParam_T, float _HyperParam_alpha, float _HyperParam_Epsilon)
{
	float* selectedFilter = _currentFilters[blockIdx.x];
	float* selectedGradient = _filterGradients[blockIdx.x];
	float* selected_V_Matrix = _VMatricies[blockIdx.x];
	float* selected_S_Matrix = _SMatricies[blockIdx.x];
	float* selected_Corrected_V_Matrix = _V_CorrectedMatrices[blockIdx.x];
	float* selected_Corrected_S_Matrix = _S_CorrectedMatricies[blockIdx.x];

	for (int rowIndex = 0; rowIndex < _filterSize; ++rowIndex)
	{
		for (int columnIndex = 0; columnIndex < _filterSize; ++columnIndex)
		{
			int index = rowIndex * _filterSize + columnIndex;

			float filterBackpropValue = selectedGradient[index];
			float oldV = selected_V_Matrix[index];
			float oldS = selected_S_Matrix[index];

			float newV = _HyperParam_Beta1 * oldV + (1 - _HyperParam_Beta1) * filterBackpropValue;
			float newS = _HyperParam_Beta2 * oldS + (1 - _HyperParam_Beta2) * filterBackpropValue;

			float newVCorrected = newV / (1 - pow(_HyperParam_Beta1, _HyperParam_T));
			float newSCorrected = newS / (1 - pow(_HyperParam_Beta2, _HyperParam_T));

			selected_V_Matrix[index] = newV;
			selected_S_Matrix[index] = newS;

			selected_Corrected_V_Matrix[index] = newVCorrected;
			selected_Corrected_S_Matrix[index] = newSCorrected;

			float oldFilterValue = selectedFilter[index];
			float newF = oldFilterValue - _HyperParam_alpha * (newVCorrected / sqrt(newSCorrected + _HyperParam_Epsilon));

			selectedFilter[index] = newF;
		}
	}
}

__global__ void BiasUpdateKernel(float** _currentFilters, float** _filterGradients, float** _VMatricies, float** _SMatricies, float** _V_CorrectedMatrices, float** _S_CorrectedMatricies, int _height, int _width, float _HyperParam_Beta1, float _HyperParam_Beta2, float _HyperParam_T, float _HyperParam_alpha, float _HyperParam_Epsilon)
{
	int filterSelectionIndex = blockIdx.x * blockDim.x + threadIdx.x;
	float* selectedFilter = _currentFilters[filterSelectionIndex];
	float* selectedGradient = _filterGradients[blockIdx.x];
	float* selected_V_Matrix = _VMatricies[filterSelectionIndex];
	float* selected_S_Matrix = _SMatricies[filterSelectionIndex];
	float* selected_Corrected_V_Matrix = _V_CorrectedMatrices[filterSelectionIndex];
	float* selected_Corrected_S_Matrix = _S_CorrectedMatricies[filterSelectionIndex];

	for (int rowIndex = 0; rowIndex < _height; ++rowIndex)
	{
		for (int columnIndex = 0; columnIndex < _width; ++columnIndex)
		{
			int index = rowIndex * _width + columnIndex;

			float filterBackpropValue = selectedGradient[index];
			float oldV = selected_V_Matrix[index];
			float oldS = selected_S_Matrix[index];

			float newV = _HyperParam_Beta1 * oldV + (1 - _HyperParam_Beta1) * filterBackpropValue;
			float newS = _HyperParam_Beta2 * oldS + (1 - _HyperParam_Beta2) * filterBackpropValue;

			float newVCorrected = newV / (1 - pow(_HyperParam_Beta1, _HyperParam_T));
			float newSCorrected = newS / (1 - pow(_HyperParam_Beta2, _HyperParam_T));

			selected_V_Matrix[index] = newV;
			selected_S_Matrix[index] = newS;

			selected_Corrected_V_Matrix[index] = newVCorrected;
			selected_Corrected_S_Matrix[index] = newSCorrected;

			float oldFilterValue = selectedFilter[index];
			float newF = oldFilterValue - _HyperParam_alpha * (newVCorrected / sqrt(newSCorrected + _HyperParam_Epsilon));

			selectedFilter[index] = newF;
		}
	}
}

Convolution::Convolution(int _filterSize, int _paddingSize, int _numberOfInputs, int _numberOfOutputs, int _inputHeight, int _inputWidth)
{
	m_FilterSize = _filterSize;
	m_PaddingSize = _paddingSize;
	m_AdamOptimizer_VMatrix = new float[m_FilterSize * m_FilterSize];
	m_AdamOptimizer_SMatrix = new float[m_FilterSize * m_FilterSize];

	m_AdamOptimizer_Corrected_VMatrix = new float[m_FilterSize * m_FilterSize];
	m_AdamOptimizer_Corrected_SMatrix = new float[m_FilterSize * m_FilterSize];

	memset(m_AdamOptimizer_VMatrix, 0, m_FilterSize * m_FilterSize * sizeof(float));
	memset(m_AdamOptimizer_SMatrix, 0, m_FilterSize * m_FilterSize * sizeof(float));

	L_FORWARD_NumberOf_INPUTS = _numberOfInputs;
	L_FORWARD_NumberOf_OUTPUTS = _numberOfOutputs;

	L_BACKWARD_NumberOf_INPUTS = L_FORWARD_NumberOf_OUTPUTS;
	L_BACKWARD_NumberOf_OUTPUTS = L_FORWARD_NumberOf_INPUTS;

	L_FORWARD_InputLayer_HEIGHT = _inputHeight;
	L_FORWARD_InputLayer_WIDTH = _inputWidth;

	L_FORWARD_OutputLayer_HEIGHT = _inputHeight - 2;
	L_FORWARD_OutputLayer_WIDTH = _inputWidth - 2;

	L_BACKWARD_InputLayer_HEIGHT = L_FORWARD_OutputLayer_HEIGHT;
	L_BACKWARD_InputLayer_WIDTH = L_FORWARD_OutputLayer_WIDTH;

	L_BACKWARD_OutputLayer_HEIGHT = L_FORWARD_InputLayer_HEIGHT;
	L_BACKWARD_OutputLayer_WIDTH = L_FORWARD_InputLayer_WIDTH;

	L_FORWARD_Pass_INPUTS = new float* [L_FORWARD_NumberOf_INPUTS];
	L_FORWARD_Pass_OUTPUTS = new float* [L_FORWARD_NumberOf_OUTPUTS];

	L_BACKWARD_Pass_INPUTS = new float* [L_BACKWARD_NumberOf_INPUTS];
	L_BACKWARD_Pass_PADDED_INPUTS = new float* [L_BACKWARD_NumberOf_INPUTS];
	L_BACKWARD_Pass_OUTPUTS = new float* [L_BACKWARD_NumberOf_OUTPUTS];

	L_NumberOf_FILTERS = L_FORWARD_NumberOf_INPUTS * L_FORWARD_NumberOf_OUTPUTS;

	L_Filters = new float* [L_NumberOf_FILTERS];
	L_FLIPPED_Filters = new float* [L_NumberOf_FILTERS];
	L_Filter_BACKPROP_RESULTS = new float* [L_NumberOf_FILTERS];

	L_Baises = new float* [L_NumberOf_FILTERS];
	L_PrevBiases = new float* [L_NumberOf_FILTERS];


	L_AdamOptimizer_V_Matrix = new float* [L_NumberOf_FILTERS];
	L_AdamOptimizer_S_Matrix = new float* [L_NumberOf_FILTERS];
	L_AdamOptimizer_Corrected_V_Matrix = new float* [L_NumberOf_FILTERS];
	L_AdamOptimizer_Corrected_S_Matrix = new float* [L_NumberOf_FILTERS];

	L_BIAS_AdamOptimizer_V_Matrix = new float* [L_NumberOf_FILTERS];
	L_BIAS_AdamOptimizer_S_Matrix = new float* [L_NumberOf_FILTERS];
	L_BIAS_AdamOptimizer_Corrected_V_Matrix = new float* [L_NumberOf_FILTERS];
	L_BIAS_AdamOptimizer_Corrected_S_Matrix = new float* [L_NumberOf_FILTERS];

	for (int i = 0; i < L_NumberOf_FILTERS; ++i)
	{
		L_AdamOptimizer_V_Matrix[i] = new float[m_FilterSize * m_FilterSize];
		L_AdamOptimizer_S_Matrix[i] = new float[m_FilterSize * m_FilterSize];
		L_AdamOptimizer_Corrected_V_Matrix[i] = new float[m_FilterSize * m_FilterSize];
		L_AdamOptimizer_Corrected_S_Matrix[i] = new float[m_FilterSize * m_FilterSize];
		L_Biases = new float* [L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH];
		size_t byteCount = m_FilterSize * m_FilterSize * sizeof(float);
		memset(L_AdamOptimizer_V_Matrix[i], 0, byteCount);
		memset(L_AdamOptimizer_S_Matrix[i], 0, byteCount);
		memset(L_AdamOptimizer_Corrected_V_Matrix[i], 0, byteCount);
		memset(L_AdamOptimizer_Corrected_S_Matrix[i], 0, byteCount);

		L_Biases = new float* [L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH];
		L_BIAS_AdamOptimizer_V_Matrix[i] = new float[L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH];
		L_BIAS_AdamOptimizer_S_Matrix[i] = new float[L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH];
		L_BIAS_AdamOptimizer_Corrected_V_Matrix[i] = new float[L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH];
		L_BIAS_AdamOptimizer_Corrected_S_Matrix[i] = new float[L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH];

		size_t biasByteCount = L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH * sizeof(float);
		memset(L_BIAS_AdamOptimizer_V_Matrix[i], 0, biasByteCount);
		memset(L_BIAS_AdamOptimizer_S_Matrix[i], 0, biasByteCount);
		memset(L_BIAS_AdamOptimizer_Corrected_V_Matrix[i], 0, biasByteCount);
		memset(L_BIAS_AdamOptimizer_Corrected_S_Matrix[i], 0, biasByteCount);
	}

	LayerFilterInitialization();
	LayerBiasInitialization();
}


void Convolution::ForwardPass(float* forwardPassInput, int fwdPassHeight, int fwdPassWidth)
{

	m_InputMatrixHeight = fwdPassHeight;
	m_InputMatrixWidth = fwdPassWidth;

	m_OutputMatrixHeight = m_InputMatrixHeight - 2;
	m_OutputMatrixWidth = m_InputMatrixWidth - 2;

	int arrayLength = fwdPassHeight * fwdPassWidth;
	size_t inputSize = arrayLength * sizeof(float);

	m_InputMatrix = new float[arrayLength];
	m_OutputMatrix = new float[m_OutputMatrixHeight * m_OutputMatrixWidth];

	memcpy(m_InputMatrix, forwardPassInput, inputSize);


	int rowShifts = m_OutputMatrixHeight;
	int columnShifts = m_OutputMatrixWidth;


	dim3 blockGrid(rowShifts, 1, 1);
	dim3 threads(columnShifts, 1, 1);

	size_t inputElementCount = m_InputMatrixHeight * m_InputMatrixWidth;
	size_t filterMatrixElementCount = m_FilterSize * m_FilterSize;
	size_t outputElementCount = m_OutputMatrixHeight * m_OutputMatrixWidth;

	int inputByteCount = inputElementCount * sizeof(float);
	int filterByteCount = filterMatrixElementCount * sizeof(float);
	int outputByteCount = outputElementCount * sizeof(float);


	//Define pointers for deviceMemory locations
	float* d_Input;
	float* d_Filter;
	float* d_Output;


	//Allocate memory
	hipMalloc((void**)&d_Input, inputByteCount);
	hipMalloc((void**)&d_Filter, filterByteCount);
	hipMalloc((void**)&d_Output, outputByteCount);


	//Copy m_Filter into global device memory m_InputMatrix -> d_Input
	hipMemcpy(d_Input, m_InputMatrix, inputByteCount, hipMemcpyHostToDevice);
	hipMemcpy(d_Filter, m_Filter, filterByteCount, hipMemcpyHostToDevice);

	ConvolutionKernel << <blockGrid, threads >> > (d_Input, d_Filter, d_Output, m_OutputMatrixWidth, m_InputMatrixWidth, m_FilterSize, m_FilterSize);
	hipDeviceSynchronize();

	hipMemcpy(m_OutputMatrix, d_Output, outputByteCount, hipMemcpyDeviceToHost);

}


void Convolution::BackwardPass(float* backpropInput, int backPassHeight, int backPassWidth)
{
	m_BackpropInputMatrixHeight = backPassHeight; ///OK
	m_BackpropInputMatrixWidth = backPassWidth; ///OK

	m_BackpropOutputMatrixHeight = m_BackpropInputMatrixHeight + 2; ///OK
	m_BackpropOutputMatrixWidth = m_BackpropInputMatrixWidth + 2; ///OK

	int arrayLength = backPassHeight * backPassWidth; ///OK
	size_t inputSize = arrayLength * sizeof(float); ///OK

	m_BackPropInputMatrix = new float[arrayLength]; ///OK
	m_BackpropagationOutput = new float[m_BackpropOutputMatrixHeight * m_BackpropOutputMatrixWidth]; ///OK

	memcpy(m_BackPropInputMatrix, backpropInput, inputSize); ///OK

	//Main backprop

	FlipFilter(); ///OK
	PadBackpropInput(); ///OK

	int rowShifts = m_BackpropOutputMatrixHeight;
	int columnShifts = m_BackpropOutputMatrixWidth;


	dim3 blockGrid(rowShifts, 1, 1); ///OK
	dim3 threads(columnShifts, 1, 1); ///OK

	size_t inputElementCount = m_PaddedInputHeight * m_PaddedInputWidth; ///OK
	size_t filterMatrixElementCount = m_FilterSize * m_FilterSize; ///OK
	size_t outputElementCount = m_BackpropOutputMatrixHeight * m_BackpropOutputMatrixWidth; ///OK

	int inputByteCount = inputElementCount * sizeof(float); ///OK
	int filterByteCount = filterMatrixElementCount * sizeof(float); ///OK
	int outputByteCount = outputElementCount * sizeof(float); ///OK

	//Define pointers for deviceMemory locations
	float* d_Input; ///OK
	float* d_Filter; ///OK
	float* d_Output; ///OK


	//Allocate memory
	hipMalloc((void**)&d_Input, inputByteCount); ///OK
	hipMalloc((void**)&d_Filter, filterByteCount); ///OK
	hipMalloc((void**)&d_Output, outputByteCount); ///OK


	//Copy m_Filter into global device memory m_InputMatrix -> d_Input
	hipMemcpy(d_Input, m_PaddedBackpropInput, inputByteCount, hipMemcpyHostToDevice); ///OK
	hipMemcpy(d_Filter, m_FlippedFilter, filterByteCount, hipMemcpyHostToDevice); ///OK

	ConvolutionKernel << <blockGrid, threads >> > (d_Input, d_Filter, d_Output, m_BackpropOutputMatrixWidth, m_PaddedInputWidth, m_FilterSize, m_FilterSize);
	hipDeviceSynchronize(); ///OK


	hipMemcpy(m_BackpropagationOutput, d_Output, outputByteCount, hipMemcpyDeviceToHost); ///OK

	//Filter backprop
	FilterBackprop(backpropInput, backPassHeight, backPassWidth); ///OK


	hipFree(d_Input); ///OK
	hipFree(d_Filter); ///OK
	hipFree(d_Output); ///OK
}

void Convolution::FilterBackprop(float* backpropInput, int backPassHeight, int backPassWidth)
{
	size_t fwdInputElementCount = m_InputMatrixHeight * m_InputMatrixWidth; ///OK
	size_t filterEqivElementCount = m_OutputMatrixHeight * m_OutputMatrixWidth; ///OK
	size_t  filterOutputElementCount = m_FilterSize * m_FilterSize; ///OK

	m_FilterBackpropResult = new float[filterOutputElementCount];

	int fwdInputByteCount = fwdInputElementCount * sizeof(float); ///OK
	int filterEqivByteCount = filterEqivElementCount * sizeof(float); ///OK
	int filterOutputByteCount = filterOutputElementCount * sizeof(float); ///OK

	float* d_FwdInput; ///OK
	float* d_FilterEquiv; ///OK
	float* d_FilterOutput; ///OK

	//Allocate memory
	hipMalloc((void**)&d_FwdInput, fwdInputByteCount); ///OK
	hipMalloc((void**)&d_FilterEquiv, filterEqivByteCount); ///OK
	hipMalloc((void**)&d_FilterOutput, filterOutputByteCount); ///OK

	//Copy m_Filter into global device memory m_InputMatrix -> d_Input
	hipMemcpy(d_FwdInput, m_InputMatrix, fwdInputByteCount, hipMemcpyHostToDevice); ///OK
	hipMemcpy(d_FilterEquiv, m_BackPropInputMatrix, filterEqivByteCount, hipMemcpyHostToDevice); ///OK

	int rowShifts = m_FilterSize; ///OK
	int columnShifts = m_FilterSize; ///OK

	dim3 blockGrid(rowShifts, 1, 1); ///OK
	dim3 threads(columnShifts, 1, 1); ///OK

	ConvolutionKernel << <blockGrid, threads >> > (d_FwdInput, d_FilterEquiv, d_FilterOutput, m_FilterSize, m_InputMatrixWidth, m_OutputMatrixHeight, m_OutputMatrixWidth);
	hipDeviceSynchronize();

	hipMemcpy(m_FilterBackpropResult, d_FilterOutput, filterOutputByteCount, hipMemcpyDeviceToHost);
}


void Convolution::LayerFilterBackprop()
{
	int inputSize = L_FORWARD_InputLayer_HEIGHT * L_FORWARD_InputLayer_WIDTH;

	int filterEquivalentSize = L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH;

	int outputSize = m_FilterSize * m_FilterSize;

	size_t inputByteCount = inputSize * sizeof(float);
	size_t filterEquivalentByteCount = filterEquivalentSize * sizeof(float);
	size_t outputByteCount = outputSize * sizeof(float);


	for (int outputNumber = 0; outputNumber < L_NumberOf_FILTERS; ++outputNumber)
	{
		L_Filter_BACKPROP_RESULTS[outputNumber] = new float[outputSize];
	}


	int numberOfBlockx_X = m_FilterSize;
	int numberOfBlocks_Y = m_FilterSize;
	int numberOfBlocks_Z = L_BACKWARD_NumberOf_INPUTS;
	int numberOfThreadsPerBlock = L_FORWARD_NumberOf_INPUTS;

	// create intermediate host array for storage of device row-pointers

	// create top-level device array pointer
	float** h_Inputs = new float* [L_FORWARD_NumberOf_INPUTS];  //(float**)malloc(L_FORWARD_NumberOf_INPUTS * sizeof(float*));
	float** h_Filters = new float* [L_BACKWARD_NumberOf_INPUTS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_Outputs = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_FORWARD_NumberOf_OUTPUTS * sizeof(float*));



	float** d_InputPointerArray;
	hipMalloc((void**)&d_InputPointerArray, L_FORWARD_NumberOf_INPUTS * sizeof(float*));

	float** d_FilterPointerArray;
	hipMalloc((void**)&d_FilterPointerArray, L_BACKWARD_NumberOf_INPUTS * sizeof(float*));

	float** d_OutputPointerArray;
	hipMalloc((void**)&d_OutputPointerArray, L_NumberOf_FILTERS * sizeof(float*));


	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_FORWARD_NumberOf_INPUTS; i++) {
		hipMalloc(&h_Inputs[i], inputByteCount);
		hipMemcpy(h_Inputs[i], L_FORWARD_Pass_INPUTS[i], inputByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_BACKWARD_NumberOf_INPUTS; i++) {
		hipMalloc(&h_Filters[i], filterEquivalentByteCount);
		hipMemcpy(h_Filters[i], L_BACKWARD_Pass_INPUTS[i], filterEquivalentByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Outputs[i], outputByteCount);
		hipMemset(h_Outputs[i], 0, outputByteCount);
	}

	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_InputPointerArray, h_Inputs, L_FORWARD_NumberOf_INPUTS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_FilterPointerArray, h_Filters, L_BACKWARD_NumberOf_INPUTS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_OutputPointerArray, h_Outputs, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);

	dim3 blockGrid(numberOfBlockx_X, numberOfBlocks_Y, numberOfBlocks_Z); ///OK
	dim3 threads(numberOfThreadsPerBlock, 1, 1); ///OK

	LayerFilterBackpropKernel << <blockGrid, threads >> > (d_InputPointerArray, d_OutputPointerArray, d_FilterPointerArray, L_FORWARD_InputLayer_WIDTH, m_FilterSize, L_BACKWARD_InputLayer_HEIGHT, L_BACKWARD_InputLayer_WIDTH, L_FORWARD_NumberOf_INPUTS);
	hipDeviceSynchronize();

	float* temp = new float[outputSize];
	hipMemcpy(h_Outputs, d_OutputPointerArray, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {

		hipMemcpy(temp, h_Outputs[i], outputByteCount, hipMemcpyDeviceToHost);
		memcpy(L_Filter_BACKPROP_RESULTS[i], temp, outputByteCount);
		hipFree(h_Outputs[i]);
	}
	hipFree(d_OutputPointerArray);
	delete[] h_Outputs;

	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_FORWARD_NumberOf_INPUTS; i++) {
		hipFree(h_Inputs[i]);
	}
	hipFree(d_InputPointerArray);
	delete[] h_Inputs;

	for (size_t i = 0; i < L_BACKWARD_NumberOf_INPUTS; i++) {
		hipFree(h_Filters[i]);
	}
	hipFree(d_FilterPointerArray);
	delete[] h_Filters;
}


void Convolution::LayerForwardPass(float** _inputs)
{
	int inputSize = L_FORWARD_InputLayer_HEIGHT * L_FORWARD_InputLayer_WIDTH;

	int filterSize = m_FilterSize * m_FilterSize;

	int biasSize = L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH;

	int outputSize = L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH;

	size_t inputByteCount = inputSize * sizeof(float);
	size_t filterByteCount = filterSize * sizeof(float);
	size_t biasByteCount = biasSize * sizeof(float);
	size_t outputByteCount = outputSize * sizeof(float);

	for (int inputNumber = 0; inputNumber < L_FORWARD_NumberOf_INPUTS; ++inputNumber)
	{
		L_FORWARD_Pass_INPUTS[inputNumber] = new float[inputSize];
		memcpy(L_FORWARD_Pass_INPUTS[inputNumber], _inputs[inputNumber], inputByteCount);
	}

	for (int outputNumber = 0; outputNumber < L_FORWARD_NumberOf_OUTPUTS; ++outputNumber)
	{
		L_FORWARD_Pass_OUTPUTS[outputNumber] = new float[outputSize];
	}


	int numberOfBlockx_X = L_FORWARD_OutputLayer_HEIGHT;
	int numberOfBlocks_Y = L_FORWARD_OutputLayer_WIDTH;
	int numberOfBlocks_Z = L_FORWARD_NumberOf_OUTPUTS;
	int numberOfThreadsPerBlock = L_FORWARD_NumberOf_INPUTS;

	// create intermediate host array for storage of device row-pointers

	// create top-level device array pointer
	float** h_Inputs = new float* [L_FORWARD_NumberOf_INPUTS];  //(float**)malloc(L_FORWARD_NumberOf_INPUTS * sizeof(float*));
	float** h_Filters = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_Biases = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_Outputs = new float* [L_FORWARD_NumberOf_OUTPUTS]; //(float**)malloc(L_FORWARD_NumberOf_OUTPUTS * sizeof(float*));

	float** d_BiasesPointerArray;
	hipMalloc((void**)&d_BiasesPointerArray, L_NumberOf_FILTERS * sizeof(float*));

	float** d_InputPointerArray;
	hipMalloc((void**)&d_InputPointerArray, L_FORWARD_NumberOf_INPUTS * sizeof(float*));

	float** d_FilterPointerArray;
	hipMalloc((void**)&d_FilterPointerArray, L_NumberOf_FILTERS * sizeof(float*));

	float** d_OutputPointerArray;
	hipMalloc((void**)&d_OutputPointerArray, L_FORWARD_NumberOf_OUTPUTS * sizeof(float*));


	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_FORWARD_NumberOf_INPUTS; i++) {
		hipMalloc((void**)&h_Inputs[i], inputByteCount);
		hipMemcpy(h_Inputs[i], L_FORWARD_Pass_INPUTS[i], inputByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc((void**)&h_Filters[i], filterByteCount);
		hipMemcpy(h_Filters[i], L_Filters[i], filterByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc((void**)&h_Biases[i], biasByteCount);
		hipMemcpy(h_Biases[i], L_Biases[i], biasByteCount, hipMemcpyHostToDevice);
	}
	CudaErrHandler("Convolution", "FlipFilterLayer - CudaMalloc for OutputPointerArray");


	for (size_t i = 0; i < L_FORWARD_NumberOf_OUTPUTS; i++) {
		hipMalloc((void**)&h_Outputs[i], outputByteCount);
		hipMemset(h_Outputs[i], 0, outputByteCount);
	}
	int ptrCopySize = L_NumberOf_FILTERS * sizeof(float*);
	int ptrCopySize2 = L_FORWARD_NumberOf_OUTPUTS * sizeof(float*);
	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_InputPointerArray, h_Inputs, L_FORWARD_NumberOf_INPUTS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_FilterPointerArray, h_Filters, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_BiasesPointerArray, h_Biases, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_OutputPointerArray, h_Outputs, L_FORWARD_NumberOf_OUTPUTS * sizeof(int**), hipMemcpyHostToDevice);

	dim3 blockGrid(numberOfBlockx_X, numberOfBlocks_Y, numberOfBlocks_Z); ///OK
	dim3 threads(numberOfThreadsPerBlock, 1, 1); ///OK

	LayerConvolutionKernel << <blockGrid, threads >> > (d_InputPointerArray, d_FilterPointerArray, d_OutputPointerArray, d_BiasesPointerArray, L_FORWARD_NumberOf_INPUTS, L_FORWARD_OutputLayer_WIDTH, L_FORWARD_InputLayer_WIDTH, m_FilterSize);
	hipDeviceSynchronize();
	float* temp = new float[outputSize];
	hipMemcpy(h_Outputs, d_OutputPointerArray, L_FORWARD_NumberOf_OUTPUTS * sizeof(float*), hipMemcpyDeviceToHost);
	for (size_t i = 0; i < L_FORWARD_NumberOf_OUTPUTS; i++) {

		hipMemcpy(temp, h_Outputs[i], outputByteCount, hipMemcpyDeviceToHost);
		memcpy(L_FORWARD_Pass_OUTPUTS[i], temp, outputByteCount);
		hipFree(h_Outputs[i]);
	}
	hipFree(d_OutputPointerArray);
	delete[] h_Outputs;

	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_FORWARD_NumberOf_INPUTS; i++) {
		hipFree(h_Inputs[i]);
	}
	hipFree(d_InputPointerArray);
	delete[] h_Inputs;

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) 
	{
		hipFree(h_Filters[i]);
	}
	hipFree(d_FilterPointerArray);
	delete[] h_Filters;
}


void Convolution::LayerBackwardPass(float** _backpropInput)
{
	int inputSize = L_BACKWARD_InputLayer_HEIGHT * L_BACKWARD_InputLayer_WIDTH;


	int filterSize = m_FilterSize * m_FilterSize;

	int outputSize = L_BACKWARD_OutputLayer_HEIGHT * L_BACKWARD_OutputLayer_WIDTH;

	size_t inputByteCount = inputSize * sizeof(float);


	size_t filterByteCount = filterSize * sizeof(float);
	size_t outputByteCount = outputSize * sizeof(float);

	for (int inputNumber = 0; inputNumber < L_BACKWARD_NumberOf_INPUTS; ++inputNumber)
	{
		L_BACKWARD_Pass_INPUTS[inputNumber] = new float[inputSize];
		memcpy(L_BACKWARD_Pass_INPUTS[inputNumber], _backpropInput[inputNumber], inputByteCount);
	}

	for (int outputNumber = 0; outputNumber < L_BACKWARD_NumberOf_OUTPUTS; ++outputNumber)
	{
		L_BACKWARD_Pass_OUTPUTS[outputNumber] = new float[outputSize];
	}

	LayerFlipFilter();
	LayerPadBackpropInput();

	int paddedInputSize = L_BACKWARD_InputLayer_PADDED_HEIGHT * L_BACKWARD_InputLayer_PADDED_WIDTH;
	size_t paddedInputByteCount = paddedInputSize * sizeof(float);


	int numberOfBlockx_X = L_BACKWARD_OutputLayer_HEIGHT;//L_FORWARD_OutputLayer_HEIGHT;
	int numberOfBlocks_Y = L_BACKWARD_OutputLayer_WIDTH;//L_FORWARD_OutputLayer_WIDTH;
	int numberOfBlocks_Z = L_BACKWARD_NumberOf_OUTPUTS;//L_FORWARD_NumberOf_OUTPUTS;
	int numberOfThreadsPerBlock = L_BACKWARD_NumberOf_INPUTS;//L_FORWARD_NumberOf_INPUTS;

	float** h_Inputs = new float* [L_BACKWARD_NumberOf_INPUTS];  //(float**)malloc(L_FORWARD_NumberOf_INPUTS * sizeof(float*));
	float** h_Filters = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_Outputs = new float* [L_BACKWARD_NumberOf_OUTPUTS]; //(float**)malloc(L_FORWARD_NumberOf_OUTPUTS * sizeof(float*));


	float** d_InputPointerArray;
	hipMalloc((void**)&d_InputPointerArray, L_BACKWARD_NumberOf_INPUTS * sizeof(float*));

	float** d_FilterPointerArray;
	hipMalloc((void**)&d_FilterPointerArray, L_NumberOf_FILTERS * sizeof(float*));

	float** d_OutputPointerArray;
	hipMalloc((void**)&d_OutputPointerArray, L_BACKWARD_NumberOf_OUTPUTS * sizeof(float*));

	float* something = new float[paddedInputSize];
	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_BACKWARD_NumberOf_INPUTS; i++) {
		hipMalloc(&h_Inputs[i], paddedInputByteCount);
		memcpy(something, L_BACKWARD_Pass_PADDED_INPUTS[i], paddedInputByteCount);
		hipError_t err = hipMemcpy(h_Inputs[i], L_BACKWARD_Pass_PADDED_INPUTS[i], paddedInputByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Filters[i], filterByteCount);
		hipMemcpy(h_Filters[i], L_FLIPPED_Filters[i], filterByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_BACKWARD_NumberOf_OUTPUTS; i++) {
		hipMalloc(&h_Outputs[i], outputByteCount);
		hipMemset(h_Outputs[i], 0, outputByteCount);
	}

	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_InputPointerArray, h_Inputs, L_BACKWARD_NumberOf_INPUTS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_FilterPointerArray, h_Filters, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_OutputPointerArray, h_Outputs, L_BACKWARD_NumberOf_OUTPUTS * sizeof(float*), hipMemcpyHostToDevice);

	dim3 blockGrid(numberOfBlockx_X, numberOfBlocks_Y, numberOfBlocks_Z); ///OK
	dim3 threads(numberOfThreadsPerBlock, 1, 1); ///OK

	LayerBackpropConvolutionKernel << <blockGrid, threads >> > (d_InputPointerArray, d_FilterPointerArray, d_OutputPointerArray, L_BACKWARD_NumberOf_INPUTS, L_BACKWARD_OutputLayer_WIDTH, L_BACKWARD_InputLayer_PADDED_WIDTH, m_FilterSize);
	hipDeviceSynchronize();
	float* temp = new float[outputSize];
	hipMemcpy(h_Outputs, d_OutputPointerArray, L_BACKWARD_NumberOf_OUTPUTS * sizeof(float*), hipMemcpyDeviceToHost);
	for (size_t i = 0; i < L_BACKWARD_NumberOf_OUTPUTS; i++) {

		hipMemcpy(temp, h_Outputs[i], outputByteCount, hipMemcpyDeviceToHost);
		memcpy(L_BACKWARD_Pass_OUTPUTS[i], temp, outputByteCount);
		hipFree(h_Outputs[i]);
	}
	hipFree(d_OutputPointerArray);
	delete[] h_Outputs;

	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_BACKWARD_NumberOf_INPUTS; i++) {
		hipFree(h_Inputs[i]);
	}
	hipFree(d_InputPointerArray);
	delete[] h_Inputs;

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipFree(h_Filters[i]);
	}
	hipFree(d_FilterPointerArray);
	delete[] h_Filters;

	LayerFilterBackprop();
	LayerUpdate();
	LayerBiasUpdate();
}


void Convolution::PadBackpropInput()
{
	m_PaddedInputHeight = m_BackpropInputMatrixHeight + 2 * m_PaddingSize;
	m_PaddedInputWidth = m_BackpropInputMatrixWidth + 2 * m_PaddingSize;
	int elementsInPaddedInput = m_PaddedInputHeight * m_PaddedInputWidth;

	m_PaddedBackpropInput = new float[elementsInPaddedInput];

	memset(m_PaddedBackpropInput, 0, elementsInPaddedInput * sizeof(float));

	float* d_Output;
	float* d_UnpaddedInput;

	size_t outputByteCount = elementsInPaddedInput * sizeof(float);
	size_t unpaddedInputByteCount = (m_BackpropInputMatrixHeight * m_BackpropInputMatrixWidth) * sizeof(float);

	hipMalloc((void**)&d_Output, outputByteCount);
	hipMalloc((void**)&d_UnpaddedInput, unpaddedInputByteCount);

	hipMemcpy(d_UnpaddedInput, m_BackPropInputMatrix, unpaddedInputByteCount, hipMemcpyHostToDevice);
	hipMemcpy(d_Output, m_PaddedBackpropInput, unpaddedInputByteCount, hipMemcpyHostToDevice);

	int numberOfBlocks = m_BackpropInputMatrixHeight / 2;
	cout << "Launching blocks: " << numberOfBlocks << endl;
	dim3 blockGrid(numberOfBlocks, 1, 1);
	dim3 threads(2, 1, 1);

	ConvolutionPaddingKernel << <blockGrid, threads >> > (d_UnpaddedInput, d_Output, m_PaddedInputWidth, m_BackpropInputMatrixWidth, m_BackpropInputMatrixHeight);
	hipDeviceSynchronize();

	hipMemcpy(m_PaddedBackpropInput, d_Output, outputByteCount, hipMemcpyDeviceToHost);
}


void Convolution::LayerPadBackpropInput()
{
	L_BACKWARD_InputLayer_PADDED_HEIGHT = L_BACKWARD_InputLayer_HEIGHT + 2 * m_PaddingSize;
	L_BACKWARD_InputLayer_PADDED_WIDTH = L_BACKWARD_InputLayer_WIDTH + 2 * m_PaddingSize;

	int inputSize = L_BACKWARD_InputLayer_HEIGHT * L_BACKWARD_InputLayer_WIDTH;

	int outputSize = L_BACKWARD_InputLayer_PADDED_HEIGHT * L_BACKWARD_InputLayer_PADDED_WIDTH;

	size_t inputByteCount = inputSize * sizeof(float);
	size_t outputByteCount = outputSize * sizeof(float);


	for (int inputNumber = 0; inputNumber < L_BACKWARD_NumberOf_INPUTS; ++inputNumber)
	{
		L_BACKWARD_Pass_PADDED_INPUTS[inputNumber] = new float[outputSize];
	}

	int numberOfBlockx_X = L_BACKWARD_InputLayer_HEIGHT / 2;
	int numberOfBlocks_Y = L_BACKWARD_NumberOf_INPUTS;
	int numberOfThreadsPerBlock = 2;

	// create intermediate host array for storage of device row-pointers

	// create top-level device array pointer
	float** h_Inputs = new float* [L_BACKWARD_NumberOf_INPUTS];  //(float**)malloc(L_FORWARD_NumberOf_INPUTS * sizeof(float*));
	float** h_Outputs = new float* [L_BACKWARD_NumberOf_INPUTS]; //(float**)malloc(L_FORWARD_NumberOf_OUTPUTS * sizeof(float*));


	float** d_InputPointerArray;
	hipMalloc((void**)&d_InputPointerArray, L_BACKWARD_NumberOf_INPUTS * sizeof(float*));

	float** d_OutputPointerArray;
	hipMalloc((void**)&d_OutputPointerArray, L_BACKWARD_NumberOf_INPUTS * sizeof(float*));


	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_BACKWARD_NumberOf_INPUTS; i++) {
		hipMalloc(&h_Inputs[i], inputByteCount);
		hipMemcpy(h_Inputs[i], L_BACKWARD_Pass_INPUTS[i], inputByteCount, hipMemcpyHostToDevice);
	}


	for (size_t i = 0; i < L_BACKWARD_NumberOf_INPUTS; i++) {
		hipMalloc(&h_Outputs[i], outputByteCount);
		hipMemset(h_Outputs[i], 0, outputByteCount);
	}

	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_InputPointerArray, h_Inputs, L_BACKWARD_NumberOf_INPUTS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_OutputPointerArray, h_Outputs, L_BACKWARD_NumberOf_INPUTS * sizeof(float*), hipMemcpyHostToDevice);

	dim3 blockGrid(numberOfBlockx_X, numberOfBlocks_Y, 1); ///OK
	dim3 threads(numberOfThreadsPerBlock, 1, 1); ///OK

	LayerPaddingKernel << <blockGrid, threads >> > (d_InputPointerArray, d_OutputPointerArray, L_BACKWARD_InputLayer_WIDTH, L_BACKWARD_InputLayer_PADDED_WIDTH);
	hipDeviceSynchronize();

	float* temp = new float[outputSize];
	hipMemcpy(h_Outputs, d_OutputPointerArray, L_BACKWARD_NumberOf_INPUTS * sizeof(float*), hipMemcpyDeviceToHost);
	for (size_t i = 0; i < L_BACKWARD_NumberOf_INPUTS; i++) {

		hipMemcpy(temp, h_Outputs[i], outputByteCount, hipMemcpyDeviceToHost);
		memcpy(L_BACKWARD_Pass_PADDED_INPUTS[i], temp, outputByteCount);
		hipFree(h_Outputs[i]);
	}
	hipFree(d_OutputPointerArray);
	delete[] h_Outputs;

	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_BACKWARD_NumberOf_INPUTS; i++) {
		hipFree(h_Inputs[i]);
	}
	hipFree(d_InputPointerArray);
	delete h_Inputs;
}


void Convolution::UpdateModule()
{
	for (int rowIndex = 0; rowIndex < m_FilterSize; ++rowIndex)
	{
		for (int columnIndex = 0; columnIndex < m_FilterSize; ++columnIndex)
		{
			int index = rowIndex * m_FilterSize + columnIndex;

			float filterBackpropValue = m_FilterBackpropResult[index];
			float oldV = m_AdamOptimizer_VMatrix[index];
			float oldS = m_AdamOptimizer_SMatrix[index];

			float newV = m_HyperParam_Beta1 * oldV + (1 - m_HyperParam_Beta1) * filterBackpropValue;
			float newS = m_HyperParam_Beta2 * oldS + (1 - m_HyperParam_Beta2) * filterBackpropValue;

			float newVCorrected = newV / (1 - pow(m_HyperParam_Beta1, m_HyperParam_T));
			float newSCorrected = newS / (1 - pow(m_HyperParam_Beta2, m_HyperParam_T));

			m_AdamOptimizer_VMatrix[index] = newV;
			m_AdamOptimizer_SMatrix[index] = newS;

			m_AdamOptimizer_Corrected_VMatrix[index] = newVCorrected;
			m_AdamOptimizer_Corrected_SMatrix[index] = newSCorrected;

			float oldFilterValue = m_Filter[index];
			float newF = oldFilterValue - m_HyperParam_alpha * (newVCorrected / sqrt(newSCorrected + m_HyperParam_Epsilon));

			m_Filter[index] = newF;
		}
	}
}


void Convolution::LayerUpdate()
{
	int filterSize = m_FilterSize * m_FilterSize;

	size_t filterByteCount = filterSize * sizeof(float);


	int numberOfBlocks_X = L_NumberOf_FILTERS;
	int numberOfThreadsPerBlock = 1;

	// create intermediate host array for storage of device row-pointers

	// create top-level device array pointer
	float** h_Filters = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_FilterGradients = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_V_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_S_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_V_CORRECTED_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_S_CORRECTED_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));


	float** d_FilterPointers;
	hipMalloc((void**)&d_FilterPointers, L_NumberOf_FILTERS * sizeof(float*));

	float** d_FilterGradientPointers;
	hipMalloc((void**)&d_FilterGradientPointers, L_NumberOf_FILTERS * sizeof(float*));

	float** d_V_Matricies;
	hipMalloc((void**)&d_V_Matricies, L_NumberOf_FILTERS * sizeof(float*));

	float** d_S_Matricies;
	hipMalloc((void**)&d_S_Matricies, L_NumberOf_FILTERS * sizeof(float*));

	float** d_V_CORRECTED_Matricies;
	hipMalloc((void**)&d_V_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*));

	float** d_S_CORRECTED_Matricies;
	hipMalloc((void**)&d_S_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*));



	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Filters[i], filterByteCount);
		hipMemcpy(h_Filters[i], L_Filters[i], filterByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_FilterGradients[i], filterByteCount);
		hipMemcpy(h_FilterGradients[i], L_Filter_BACKPROP_RESULTS[i], filterByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_V_Matricies[i], filterByteCount);
		hipMemcpy(h_V_Matricies[i], L_AdamOptimizer_V_Matrix[i], filterByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_S_Matricies[i], filterByteCount);
		hipMemcpy(h_S_Matricies[i], L_AdamOptimizer_S_Matrix[i], filterByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_V_CORRECTED_Matricies[i], filterByteCount);
		hipMemcpy(h_V_CORRECTED_Matricies[i], L_AdamOptimizer_Corrected_V_Matrix[i], filterByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_S_CORRECTED_Matricies[i], filterByteCount);
		hipMemcpy(h_S_CORRECTED_Matricies[i], L_AdamOptimizer_Corrected_S_Matrix[i], filterByteCount, hipMemcpyHostToDevice);
	}


	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_FilterPointers, h_Filters, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_FilterGradientPointers, h_FilterGradients, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_V_Matricies, h_V_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_S_Matricies, h_S_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_V_CORRECTED_Matricies, h_V_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_S_CORRECTED_Matricies, h_S_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);


	dim3 blockGrid(numberOfBlocks_X, 1, 1); ///OK
	dim3 threads(numberOfThreadsPerBlock, 1, 1); ///OK

	FilterUpdateKernel << <blockGrid, threads >> > (d_FilterPointers, d_FilterGradientPointers, d_V_Matricies, d_S_Matricies, d_V_CORRECTED_Matricies, d_S_CORRECTED_Matricies, m_FilterSize, m_HyperParam_Beta1, m_HyperParam_Beta2, m_HyperParam_T, m_HyperParam_alpha, m_HyperParam_Epsilon);
	hipDeviceSynchronize();

	float* temp = new float[filterByteCount];

	hipMemcpy(h_Filters, d_FilterPointers, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_FilterGradients, d_FilterGradientPointers, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_V_Matricies, d_V_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_S_Matricies, d_S_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_V_CORRECTED_Matricies, d_V_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_S_CORRECTED_Matricies, d_S_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {

		hipMemcpy(temp, h_Filters[i], filterByteCount, hipMemcpyDeviceToHost);
		memcpy(L_Filters[i], temp, filterByteCount);
		hipFree(h_Filters[i]);

		hipMemcpy(temp, h_V_Matricies[i], filterByteCount, hipMemcpyDeviceToHost);
		memcpy(L_AdamOptimizer_V_Matrix[i], temp, filterByteCount);
		hipFree(h_V_Matricies[i]);

		hipMemcpy(temp, h_S_Matricies[i], filterByteCount, hipMemcpyDeviceToHost);
		memcpy(L_AdamOptimizer_S_Matrix[i], temp, filterByteCount);
		hipFree(h_S_Matricies[i]);

		hipMemcpy(temp, h_V_CORRECTED_Matricies[i], filterByteCount, hipMemcpyDeviceToHost);
		memcpy(L_AdamOptimizer_Corrected_V_Matrix[i], temp, filterByteCount);
		hipFree(h_V_CORRECTED_Matricies[i]);

		hipMemcpy(temp, h_S_CORRECTED_Matricies[i], filterByteCount, hipMemcpyDeviceToHost);
		memcpy(L_AdamOptimizer_Corrected_S_Matrix[i], temp, filterByteCount);
		hipFree(h_S_CORRECTED_Matricies[i]);
	}

	hipFree(d_FilterPointers);
	hipFree(d_FilterGradientPointers);
	hipFree(d_V_Matricies);
	hipFree(d_S_Matricies);
	hipFree(d_V_CORRECTED_Matricies);
	hipFree(d_S_CORRECTED_Matricies);

	delete[] h_Filters;
	delete[] h_FilterGradients;
	delete[] h_V_Matricies;
	delete[] h_S_Matricies;
	delete[] h_V_CORRECTED_Matricies;
	delete[] h_S_CORRECTED_Matricies;
}

void Convolution::LayerBiasUpdate()
{
	int biasMatrixSize = L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH;

	size_t biasMatrixByteCount = biasMatrixSize * sizeof(float);


	int numberOfBlocks_X = L_BACKWARD_NumberOf_INPUTS;
	int numberOfThreadsPerBlock = L_BACKWARD_NumberOf_OUTPUTS;

	// create intermediate host array for storage of device row-pointers

	// create top-level device array pointer
	float** h_Biases = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_BiasGradients = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_V_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_S_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_V_CORRECTED_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_S_CORRECTED_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));


	float** d_BiasPointers;
	hipMalloc((void**)&d_BiasPointers, L_NumberOf_FILTERS * sizeof(float*));

	float** d_BiasGradientPointers;
	hipMalloc((void**)&d_BiasGradientPointers, L_NumberOf_FILTERS * sizeof(float*));

	float** d_V_Matricies;
	hipMalloc((void**)&d_V_Matricies, L_NumberOf_FILTERS * sizeof(float*));

	float** d_S_Matricies;
	hipMalloc((void**)&d_S_Matricies, L_NumberOf_FILTERS * sizeof(float*));

	float** d_V_CORRECTED_Matricies;
	hipMalloc((void**)&d_V_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*));

	float** d_S_CORRECTED_Matricies;
	hipMalloc((void**)&d_S_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*));

	for (int i = 0; i < L_BACKWARD_NumberOf_INPUTS; ++i)
	{
		hipMalloc(&h_BiasGradients[i], biasMatrixByteCount);
		hipMemcpy(h_BiasGradients[i], L_BACKWARD_Pass_INPUTS[i], biasMatrixByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Biases[i], biasMatrixByteCount);
		hipMemcpy(h_Biases[i], L_Biases[i], biasMatrixByteCount, hipMemcpyHostToDevice);

		L_PrevBiases[i] = new float[biasMatrixSize];
		memcpy(L_PrevBiases[i], L_Biases[i], biasMatrixByteCount);

		hipMalloc(&h_V_Matricies[i], biasMatrixByteCount);
		hipMemcpy(h_V_Matricies[i], L_BIAS_AdamOptimizer_V_Matrix[i], biasMatrixByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_S_Matricies[i], biasMatrixByteCount);
		hipMemcpy(h_S_Matricies[i], L_BIAS_AdamOptimizer_S_Matrix[i], biasMatrixByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_V_CORRECTED_Matricies[i], biasMatrixByteCount);
		hipMemcpy(h_V_CORRECTED_Matricies[i], L_BIAS_AdamOptimizer_Corrected_V_Matrix[i], biasMatrixByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_S_CORRECTED_Matricies[i], biasMatrixByteCount);
		hipMemcpy(h_S_CORRECTED_Matricies[i], L_BIAS_AdamOptimizer_Corrected_S_Matrix[i], biasMatrixByteCount, hipMemcpyHostToDevice);
	}


	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_BiasPointers, h_Biases, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_BiasGradientPointers, h_BiasGradients, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_V_Matricies, h_V_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_S_Matricies, h_S_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_V_CORRECTED_Matricies, h_V_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_S_CORRECTED_Matricies, h_S_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);


	dim3 blockGrid(numberOfBlocks_X, 1, 1); ///OK
	dim3 threads(numberOfThreadsPerBlock, 1, 1); ///OK

	BiasUpdateKernel << <blockGrid, threads >> > (d_BiasPointers, d_BiasGradientPointers, d_V_Matricies, d_S_Matricies, d_V_CORRECTED_Matricies, d_S_CORRECTED_Matricies, L_FORWARD_OutputLayer_HEIGHT, L_FORWARD_OutputLayer_WIDTH, m_HyperParam_Beta1, m_HyperParam_Beta2, m_HyperParam_T, m_HyperParam_alpha, m_HyperParam_Epsilon);
	hipDeviceSynchronize();

	float* temp = new float[biasMatrixByteCount];

	hipMemcpy(h_Biases, d_BiasPointers, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_BiasGradients, d_BiasGradientPointers, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_V_Matricies, d_V_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_S_Matricies, d_S_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_V_CORRECTED_Matricies, d_V_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_S_CORRECTED_Matricies, d_S_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {

		hipMemcpy(temp, h_Biases[i], biasMatrixByteCount, hipMemcpyDeviceToHost);
		memcpy(L_Biases[i], temp, biasMatrixByteCount);
		hipFree(h_Biases[i]);

		hipMemcpy(temp, h_V_Matricies[i], biasMatrixByteCount, hipMemcpyDeviceToHost);
		memcpy(L_BIAS_AdamOptimizer_V_Matrix[i], temp, biasMatrixByteCount);
		hipFree(h_V_Matricies[i]);

		hipMemcpy(temp, h_S_Matricies[i], biasMatrixByteCount, hipMemcpyDeviceToHost);
		memcpy(L_BIAS_AdamOptimizer_S_Matrix[i], temp, biasMatrixByteCount);
		hipFree(h_S_Matricies[i]);

		hipMemcpy(temp, h_V_CORRECTED_Matricies[i], biasMatrixByteCount, hipMemcpyDeviceToHost);
		memcpy(L_BIAS_AdamOptimizer_Corrected_V_Matrix[i], temp, biasMatrixByteCount);
		hipFree(h_V_CORRECTED_Matricies[i]);

		hipMemcpy(temp, h_S_CORRECTED_Matricies[i], biasMatrixByteCount, hipMemcpyDeviceToHost);
		memcpy(L_BIAS_AdamOptimizer_Corrected_S_Matrix[i], temp, biasMatrixByteCount);
		hipFree(h_S_CORRECTED_Matricies[i]);
	}

	hipFree(d_BiasPointers);
	hipFree(d_BiasGradientPointers);
	hipFree(d_V_Matricies);
	hipFree(d_S_Matricies);
	hipFree(d_V_CORRECTED_Matricies);
	hipFree(d_S_CORRECTED_Matricies);

	delete[] h_Biases;
	delete[] h_BiasGradients;
	delete[] h_V_Matricies;
	delete[] h_S_Matricies;
	delete[] h_V_CORRECTED_Matricies;
	delete[] h_S_CORRECTED_Matricies;
}



void Convolution::Dialate(float* _input, float* _output)
{

}

void Convolution::InitializeFilter()
{
	std::random_device rd{};
	std::mt19937 gen{ rd() };
	std::normal_distribution<> distribution{ 1,2 };
	m_Filter = new float[m_FilterSize * m_FilterSize];

	for (int i = 0; i < m_FilterSize * m_FilterSize; ++i)
	{
		m_Filter[i] = i + 1;// distribution(gen);  //TODO FIX INITIALIZATION
	}
}

void Convolution::LayerFilterInitialization()
{
	std::random_device rd{};
	std::mt19937 gen{ rd() };
	std::normal_distribution<> distribution{ 0,1 };

	int filterElementCount = m_FilterSize * m_FilterSize;

	for (int filterNumber = 0; filterNumber < L_NumberOf_FILTERS; ++filterNumber)
	{
		L_Filters[filterNumber] = new float[filterElementCount];

		for (int i = 0; i < filterElementCount; ++i)
		{
			L_Filters[filterNumber][i] = i + 1; //distribution(gen);
		}
	}
}

void Convolution::LayerBiasInitialization()
{
	std::random_device rd{};
	std::mt19937 gen{ rd() };
	std::normal_distribution<> distribution{ 0,1 };

	int biasElementCount = L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH;

	for (int biasNumber = 0; biasNumber < L_NumberOf_FILTERS; ++biasNumber)
	{
		L_Biases[biasNumber] = new float[biasElementCount];

		for (int i = 0; i < biasElementCount; ++i)
		{
			L_Biases[biasNumber][i] = 1; //distribution(gen);
		}
	}
}


void Convolution::FlipFilter()
{
	int filterArraySize = m_FilterSize * m_FilterSize;
	m_FlippedFilter = new float[filterArraySize];

	int k = 0;

	//Loop from back and assign value to new array
	for (int i = filterArraySize - 1; i >= 0; ) {
		m_FlippedFilter[k++] = m_Filter[i--];
	}
}

void Convolution::LayerFlipFilter()
{
	int inputSize = m_FilterSize * m_FilterSize;

	int outputSize = m_FilterSize * m_FilterSize;

	size_t inputByteCount = inputSize * sizeof(float);
	size_t outputByteCount = outputSize * sizeof(float);

	for (int filterNumber = 0; filterNumber < L_NumberOf_FILTERS; ++filterNumber)
	{
		L_FLIPPED_Filters[filterNumber] = new float[inputSize];
	}

	int numberOfBlockx_X = L_NumberOf_FILTERS;

	// create intermediate host array for storage of device row-pointers

	// create top-level device array pointer
	float** h_Inputs = new float* [L_NumberOf_FILTERS];

	float** h_Outputs = new float* [L_NumberOf_FILTERS];


	float** d_InputPointerArray;
	hipMalloc((void**)&d_InputPointerArray, L_NumberOf_FILTERS * sizeof(float*));

	CudaErrHandler("Convolution", "FlipFilterLayer - CudaMalloc for InputPointerArray");

	float** d_OutputPointerArray;
	hipMalloc((void**)&d_OutputPointerArray, L_NumberOf_FILTERS * sizeof(float*));

	CudaErrHandler("Convolution", "FlipFilterLayer - CudaMalloc for OutputPointerArray");

	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc((void**)&h_Inputs[i], inputByteCount);
		CudaErrHandler("Convolution", "FlipFilterLayer - CudaMalloc for Filters");

		hipMemcpy(h_Inputs[i], L_Filters[i], inputByteCount, hipMemcpyHostToDevice);

		CudaErrHandler("Convolution", "FlipFilterLayer - Filters copy to device");


	}

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc((void**)&h_Outputs[i], outputByteCount);
		
	}

	int pointerArraySize = L_NumberOf_FILTERS * sizeof(float*);
	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_InputPointerArray, h_Inputs, pointerArraySize, hipMemcpyHostToDevice);
	hipMemcpy(d_OutputPointerArray, h_Outputs, pointerArraySize, hipMemcpyHostToDevice);

	dim3 blockGrid(numberOfBlockx_X, 1, 1); ///OK
	dim3 threads(1, 1, 1); ///OK

	LayerFilterFlipKernel << <blockGrid, threads >> > (d_InputPointerArray, d_OutputPointerArray, m_FilterSize);
	CudaErrHandler("Convolution", "FlipFilterLayer - LayerFlipFilterKernal call");
	hipDeviceSynchronize();

	float* temp = new float[outputSize];

	hipMemcpy(h_Outputs, d_OutputPointerArray, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {

		hipMemcpy(L_FLIPPED_Filters[i], h_Outputs[i], outputByteCount, hipMemcpyDeviceToHost);
		CudaErrHandler("Convolution", "FlipFilterLayer - Output copyback");

		//memcpy(L_FLIPPED_Filters[i], temp, outputByteCount);
		hipFree(h_Outputs[i]);

		CudaErrHandler("Convolution", "FlipFilterLayer - Free h_Outputs");
	}
	hipFree(d_OutputPointerArray);
	CudaErrHandler("Convolution", "FlipFilterLayer - Free OutputPointerArray");
	delete[] h_Outputs;

	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipFree(h_Inputs[i]);
		CudaErrHandler("Convolution", "FlipFilterLayer - Free h_Outputs");
	}
	hipFree(d_InputPointerArray);
	CudaErrHandler("Convolution", "FlipFilterLayer - Free InputPointerArray");
	delete[] h_Inputs;

	delete[] temp;
}



void Convolution::SetHyperParams(float _beta1, float _beta2, float _eps, int _t, float _alpha)
{
	m_HyperParam_Beta1 = _beta1;
	m_HyperParam_Beta2 = _beta2;
	m_HyperParam_Epsilon = _eps;
	m_HyperParam_T = _t;
	m_HyperParam_alpha = _alpha;
}


void Convolution::CudaErrHandler(string responsibleModule, string function)
{
	/*hipError_t err = hipGetLastError();
	string cudaErr = hipGetErrorString(err);
	if (err != 0)
	{
		cout << "#################################################" << endl;
		cout << ">> CUDA ERROR CUDA ERROR CUDA ERROR CUDA ERROR <<" << endl;
		cout << "#################################################" << endl;
		cout << ">>> ERROR STRING >>> " << cudaErr << endl;
		cout << ">>> OFFENDING MODULE >>> " << responsibleModule << endl;
		cout << ">>> OFFENDING FUNCTION >>> " << function << endl;
		cout << endl;
		cout << endl;
	}*/
}


void Convolution::DebugPrintAll()
{
	int newLineCounter = 1;

	cout << "=====================================================" << endl;
	cout << "============ Convolution Debug Print All ============" << endl;
	cout << "=====================================================" << endl;

	cout << "Convolution: " << endl;
	cout << "Layer ID: " << layerID << endl;
	cout << "Level ID: " << levelID << endl;
	cout << "Hyper parameters: " << endl;
	cout << "Beta 1: " << m_HyperParam_Beta1 << endl;
	cout << "Beta 2: " << m_HyperParam_Beta2 << endl;
	cout << "Epsilon: " << m_HyperParam_Epsilon << endl;
	cout << "Alpha: " << m_HyperParam_alpha << endl;
	cout << "T: " << m_HyperParam_T << endl;


	cout << ">>>> Forward Inputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_FORWARD_NumberOf_INPUTS; ++inputIndex)
	{
		cout << "--- Element " << inputIndex + 1 << "---" << endl;
		for (int elementIndex = 0; elementIndex < L_FORWARD_InputLayer_HEIGHT * L_FORWARD_InputLayer_WIDTH; ++elementIndex)
		{
			cout << L_FORWARD_Pass_INPUTS[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == L_FORWARD_InputLayer_WIDTH + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Normal Filter Inputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_Filters[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}


	cout << ">>>> Flipped Filter Inputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_FLIPPED_Filters[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}


	cout << ">>>> Forward Outputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_FORWARD_NumberOf_OUTPUTS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH; ++elementIndex)
		{
			cout << L_FORWARD_Pass_OUTPUTS[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == L_FORWARD_OutputLayer_WIDTH + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Backward Inputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_BACKWARD_NumberOf_INPUTS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < L_BACKWARD_InputLayer_HEIGHT * L_BACKWARD_InputLayer_WIDTH; ++elementIndex)
		{
			cout << L_BACKWARD_Pass_INPUTS[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == L_BACKWARD_InputLayer_WIDTH + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Padded Backward Inputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_BACKWARD_NumberOf_INPUTS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < L_BACKWARD_InputLayer_PADDED_HEIGHT * L_BACKWARD_InputLayer_PADDED_WIDTH; ++elementIndex)
		{
			cout << L_BACKWARD_Pass_PADDED_INPUTS[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == L_BACKWARD_InputLayer_PADDED_WIDTH + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Backward Outputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_BACKWARD_NumberOf_OUTPUTS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < L_BACKWARD_OutputLayer_HEIGHT * L_BACKWARD_OutputLayer_WIDTH; ++elementIndex)
		{
			cout << L_BACKWARD_Pass_OUTPUTS[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == L_BACKWARD_OutputLayer_WIDTH + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Filter Backprop Outputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_Filter_BACKPROP_RESULTS[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Bias Outputs Before Update <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH; ++elementIndex)
		{
			cout << L_PrevBiases[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == L_BACKWARD_OutputLayer_WIDTH + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}


	cout << ">>>> Bias Outputs After Update <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH; ++elementIndex)
		{
			cout << L_Biases[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == L_FORWARD_OutputLayer_WIDTH + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Filter Adam Optimizer V Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_AdamOptimizer_V_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Filter Adam Optimizer >CORRECTED< V Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_AdamOptimizer_Corrected_V_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Filter Adam Optimizer S Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_AdamOptimizer_S_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Filter Adam Optimizer >CORRECTED< S Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_AdamOptimizer_Corrected_S_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> BIAS Adam Optimizer V Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_BIAS_AdamOptimizer_V_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> BIAS Adam Optimizer >CORRECTED< V Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_BIAS_AdamOptimizer_Corrected_V_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> BIAS Adam Optimizer S Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_BIAS_AdamOptimizer_S_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> BIAS Adam Optimizer >CORRECTED< S Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_BIAS_AdamOptimizer_Corrected_S_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

}

