#include "hip/hip_runtime.h"
#include <hip/hip_vector_types.h>

#ifndef __HIPCC__  
#define __HIPCC__
#endif

#ifndef  __CUDART_API_PER_THREAD_DEFAULT_STREAM
#define  __CUDART_API_PER_THREAD_DEFAULT_STREAM
#endif
#include <hip/hip_runtime_api.h>


#ifndef __CUDACC_RTC__ 
#define __CUDACC_RTC__

#endif

#include <algorithm>
#include <iostream>

#include ""
#include "Convolution.cuh"
#include <random>
#include <cmath>
using namespace std;

__global__ void ConvolutionKernel(float* d_Input, float* d_Filter, float* d_Output, int _outputWidth, int _convolutionInputWidth, int filterHeight, int filterWidth)
{
	//These define indecies for output matrix;
	int outputRowIndex = blockIdx.x;
	int outputColumnIndex = threadIdx.x;

	//Starts from "top left" of current block of pixels being processed
	int inputRowIndex = blockIdx.x;
	int inputColumnIndex = threadIdx.x;

	int outputArrayIndex = outputRowIndex * _outputWidth + outputColumnIndex;

	int inputArrayIndex = 0;

	float result = 0;
	int filterIndex = 0;
	int temp = 0;
	for (int row = 0; row < filterHeight; row++)
	{
		inputColumnIndex = threadIdx.x;

		for (int col = 0; col < filterWidth; col++)
		{
			inputArrayIndex = inputRowIndex * _convolutionInputWidth + inputColumnIndex;

			result += d_Input[inputArrayIndex] * d_Filter[filterIndex];
			filterIndex++;
			inputColumnIndex += 1;
		}
		inputRowIndex += 1;

	}

	d_Output[outputArrayIndex] = result;
};

__global__ void ConvolutionPaddingKernel(float* d_UnpaddedInput, float* d_Output, int _paddedInputWidth, int _unpaddedInputWidth, int _unpaddedInputHeight)
{
	int rowWriteIndex = (blockIdx.x + 1) * 2 + threadIdx.x;
	int columnWriteIndex = 2;

	int inputRowReadIndex = (blockIdx.x * 2) + threadIdx.x;
	int inputColumnReadIndex = 0;

	int arrayPosition = rowWriteIndex * _paddedInputWidth + columnWriteIndex;
	int inputArrayPosition = inputRowReadIndex * _unpaddedInputWidth + inputColumnReadIndex;

	int var = 0;
	for (int i = 0; i < _unpaddedInputWidth; i++)
	{
		d_Output[arrayPosition] = d_UnpaddedInput[inputArrayPosition];
		arrayPosition++;
		inputArrayPosition++;
	}
}


Convolution::Convolution(int _filterSize, int _paddingSize)
{
	m_FilterSize = _filterSize;
	m_PaddingSize = _paddingSize;
	InitializeFilter();
}


void Convolution::ForwardPass(float* forwardPassInput, int fwdPassHeight, int fwdPassWidth)
{

	m_InputMatrixHeight = fwdPassHeight;
	m_InputMatrixWidth = fwdPassWidth;

	m_OutputMatrixHeight = m_InputMatrixHeight - 2;
	m_OutputMatrixWidth = m_InputMatrixWidth - 2;

	int arrayLength = fwdPassHeight * fwdPassWidth;
	size_t inputSize = arrayLength * sizeof(float);

	m_InputMatrix = new float[arrayLength];
	m_OutputMatrix = new float[m_OutputMatrixHeight * m_OutputMatrixWidth];

	memcpy(m_InputMatrix, forwardPassInput, inputSize);


	int rowShifts = m_OutputMatrixHeight;
	int columnShifts = m_OutputMatrixWidth;


	dim3 blockGrid(rowShifts, 1, 1);
	dim3 threads(columnShifts, 1, 1);

	size_t inputElementCount = m_InputMatrixHeight * m_InputMatrixWidth;
	size_t filterMatrixElementCount = m_FilterSize * m_FilterSize;
	size_t outputElementCount = m_OutputMatrixHeight * m_OutputMatrixWidth;

	int inputByteCount = inputElementCount * sizeof(float);
	int filterByteCount = filterMatrixElementCount * sizeof(float);
	int outputByteCount = outputElementCount * sizeof(float);


	//Define pointers for deviceMemory locations
	float* d_Input;
	float* d_Filter;
	float* d_Output;


	//Allocate memory
	hipMalloc((void**)&d_Input, inputByteCount);
	hipMalloc((void**)&d_Filter, filterByteCount);
	hipMalloc((void**)&d_Output, outputByteCount);


	//Copy m_Filter into global device memory m_InputMatrix -> d_Input
	hipMemcpy(d_Input, m_InputMatrix, inputByteCount, hipMemcpyHostToDevice);
	hipMemcpy(d_Filter, m_Filter, filterByteCount, hipMemcpyHostToDevice);

	ConvolutionKernel << <blockGrid, threads >> > (d_Input, d_Filter, d_Output, m_OutputMatrixWidth, m_InputMatrixWidth,  m_FilterSize, m_FilterSize);
	hipDeviceSynchronize();

	hipMemcpy(m_OutputMatrix, d_Output, outputByteCount, hipMemcpyDeviceToHost);

}


void Convolution::BackwardPass(float* backpropInput, int backPassHeight, int backPassWidth)
{
	m_BackpropInputMatrixHeight = backPassHeight; ///OK
	m_BackpropInputMatrixWidth = backPassWidth; ///OK

	m_BackpropOutputMatrixHeight = m_BackpropInputMatrixHeight + 2; ///OK
	m_BackpropOutputMatrixWidth = m_BackpropInputMatrixWidth + 2; ///OK

	int arrayLength = backPassHeight * backPassWidth; ///OK
	size_t inputSize = arrayLength * sizeof(float); ///OK

	m_BackPropInputMatrix = new float[arrayLength]; ///OK
	m_BackpropagationOutput = new float[m_BackpropOutputMatrixHeight * m_BackpropOutputMatrixWidth]; ///OK

	memcpy(m_BackPropInputMatrix, backpropInput, inputSize); ///OK

	//Main backprop

	FlipFilter(); ///OK
	PadBackpropInput(); ///OK

	int rowShifts = m_BackpropOutputMatrixHeight;
	int columnShifts = m_BackpropOutputMatrixWidth;
	

	dim3 blockGrid(rowShifts, 1, 1); ///OK
	dim3 threads(columnShifts, 1, 1); ///OK

	size_t inputElementCount = m_PaddedInputHeight * m_PaddedInputWidth; ///OK
	size_t filterMatrixElementCount = m_FilterSize * m_FilterSize; ///OK
	size_t outputElementCount = m_BackpropOutputMatrixHeight * m_BackpropOutputMatrixWidth; ///OK

	int inputByteCount = inputElementCount * sizeof(float); ///OK
	int filterByteCount = filterMatrixElementCount * sizeof(float); ///OK
	int outputByteCount = outputElementCount * sizeof(float); ///OK
	
	//Define pointers for deviceMemory locations
	float* d_Input; ///OK
	float* d_Filter; ///OK
	float* d_Output; ///OK


	//Allocate memory
	hipMalloc((void**)&d_Input, inputByteCount); ///OK
	hipMalloc((void**)&d_Filter, filterByteCount); ///OK
	hipMalloc((void**)&d_Output, outputByteCount); ///OK


	//Copy m_Filter into global device memory m_InputMatrix -> d_Input
	hipMemcpy(d_Input, m_PaddedBackpropInput, inputByteCount, hipMemcpyHostToDevice); ///OK
	hipMemcpy(d_Filter, m_FlippedFilter, filterByteCount, hipMemcpyHostToDevice); ///OK

	ConvolutionKernel << <blockGrid, threads >> > (d_Input, d_Filter, d_Output, m_BackpropOutputMatrixWidth, m_PaddedInputWidth,  m_FilterSize, m_FilterSize);
	hipDeviceSynchronize(); ///OK


	hipMemcpy(m_BackpropagationOutput, d_Output, outputByteCount, hipMemcpyDeviceToHost); ///OK

	//Filter backprop
	FilterBackprop(backpropInput, backPassHeight, backPassWidth); ///OK


	hipFree(d_Input); ///OK
	hipFree(d_Filter); ///OK
	hipFree(d_Output); ///OK
}

void Convolution::FilterBackprop(float* backpropInput, int backPassHeight, int backPassWidth)
{
	size_t fwdInputElementCount = m_InputMatrixHeight * m_InputMatrixWidth; ///OK
	size_t filterEqivElementCount = m_OutputMatrixHeight * m_OutputMatrixWidth; ///OK
	size_t  filterOutputElementCount = m_FilterSize * m_FilterSize; ///OK

	m_FilterBackpropResult = new float[filterOutputElementCount];

	int fwdInputByteCount = fwdInputElementCount * sizeof(float); ///OK
	int filterEqivByteCount = filterEqivElementCount * sizeof(float); ///OK
	int filterOutputByteCount = filterOutputElementCount * sizeof(float); ///OK

	float* d_FwdInput; ///OK
	float* d_FilterEquiv; ///OK
	float* d_FilterOutput; ///OK

	//Allocate memory
	hipMalloc((void**)&d_FwdInput, fwdInputByteCount); ///OK
	hipMalloc((void**)&d_FilterEquiv, filterEqivByteCount); ///OK
	hipMalloc((void**)&d_FilterOutput, filterOutputByteCount); ///OK

	//Copy m_Filter into global device memory m_InputMatrix -> d_Input
	hipMemcpy(d_FwdInput, m_InputMatrix, fwdInputByteCount, hipMemcpyHostToDevice); ///OK
	hipMemcpy(d_FilterEquiv, m_OutputMatrix, filterEqivByteCount, hipMemcpyHostToDevice); ///OK

	int rowShifts = m_FilterSize; ///OK
	int columnShifts = m_FilterSize; ///OK

	dim3 blockGrid(rowShifts, 1, 1); ///OK
	dim3 threads(columnShifts, 1, 1); ///OK

	ConvolutionKernel << <blockGrid, threads >> > (d_FwdInput, d_FilterEquiv, d_FilterOutput, m_FilterSize, m_InputMatrixWidth,  m_OutputMatrixHeight, m_OutputMatrixWidth);
	hipDeviceSynchronize();

	hipMemcpy(m_FilterBackpropResult, d_FilterOutput, filterOutputByteCount, hipMemcpyDeviceToHost);
}


void Convolution::PadBackpropInput()
{
	m_PaddedInputHeight = m_BackpropInputMatrixHeight + 2 * m_PaddingSize;
	m_PaddedInputWidth = m_BackpropInputMatrixWidth + 2 * m_PaddingSize;
	int elementsInPaddedInput = m_PaddedInputHeight * m_PaddedInputWidth;

	m_PaddedBackpropInput = new float[elementsInPaddedInput];

	memset(m_PaddedBackpropInput, 0, elementsInPaddedInput * sizeof(float));

	float* d_Output;
	float* d_UnpaddedInput;

	size_t outputByteCount = elementsInPaddedInput * sizeof(float);
	size_t unpaddedInputByteCount = (m_BackpropInputMatrixHeight * m_BackpropInputMatrixWidth) * sizeof(float);

	hipMalloc((void**)&d_Output, outputByteCount);
	hipMalloc((void**)&d_UnpaddedInput, unpaddedInputByteCount);

	hipMemcpy(d_UnpaddedInput, m_BackPropInputMatrix, unpaddedInputByteCount, hipMemcpyHostToDevice);
	hipMemcpy(d_Output, m_PaddedBackpropInput, unpaddedInputByteCount, hipMemcpyHostToDevice);

	int numberOfBlocks = m_BackpropInputMatrixHeight / 2;
	cout << "Launching blocks: " << numberOfBlocks << endl;
	dim3 blockGrid(numberOfBlocks, 1, 1);
	dim3 threads(2, 1, 1);

	ConvolutionPaddingKernel << <blockGrid, threads >> > (d_UnpaddedInput, d_Output, m_PaddedInputWidth, m_BackpropInputMatrixWidth, m_BackpropInputMatrixHeight);
	hipDeviceSynchronize();

	hipMemcpy(m_PaddedBackpropInput, d_Output, outputByteCount, hipMemcpyDeviceToHost);
}

void Convolution::UpdateModule()
{
	for (int rowIndex = 0; rowIndex < m_FilterSize; ++rowIndex)
	{
		for (int columnIndex = 0; columnIndex < m_FilterSize; ++columnIndex)
		{
			int index = rowIndex * m_FilterSize + columnIndex;

			float filterBackpropValue = m_FilterBackpropResult[index];

			float newV = m_HyperParam_Beta1 * filterBackpropValue + (1 - m_HyperParam_Beta1) * filterBackpropValue;
			float newS = m_HyperParam_Beta2 * filterBackpropValue + (1 - m_HyperParam_Beta2) * filterBackpropValue;

			float newVCorrected = newV / (1 - pow(m_HyperParam_Beta1, m_HyperParam_T));
			float newSCorrected = newS / (1 - pow(m_HyperParam_Beta2, m_HyperParam_T));

			m_AdamOptimizer_VMatrix[index] = newV;
			m_AdamOptimizer_SMatrix[index] = newS;

			m_AdamOptimizer_Corrected_VMatrix[index] = newVCorrected;
			m_AdamOptimizer_Corrected_SMatrix[index] = newSCorrected;

			float oldFilterValue = m_Filter[index];
			float newF = oldFilterValue - m_HyperParam_alpha * (newVCorrected / sqrt(newSCorrected + m_HyperParam_Epsilon));

			m_Filter[index] = newF;
		}
	}
}


void Convolution::Dialate(float* _input, float* _output)
{

}

void Convolution::InitializeFilter()
{
	std::random_device rd{};
	std::mt19937 gen{ rd() };
	std::normal_distribution<> distribution{ 1,2 };
	m_Filter = new float[m_FilterSize * m_FilterSize];

	for (int i = 0; i < m_FilterSize * m_FilterSize; ++i)
	{
		m_Filter[i] = distribution(gen);  //TODO FIX INITIALIZATION
	}
}

void Convolution::FlipFilter()
{
	int filterArraySize = m_FilterSize * m_FilterSize;
	m_FlippedFilter = new float[filterArraySize];

	int k = 0;

	//Loop from back and assign value to new array
	for (int i = filterArraySize - 1; i >= 0; ) {
		m_FlippedFilter[k++] = m_Filter[i--];
	}
}




