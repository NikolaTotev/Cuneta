#include "hip/hip_runtime.h"
#include <hip/hip_vector_types.h>

#ifndef __HIPCC__  
#define __HIPCC__
#endif

#ifndef  __CUDART_API_PER_THREAD_DEFAULT_STREAM
#define  __CUDART_API_PER_THREAD_DEFAULT_STREAM
#endif
#include <hip/hip_runtime_api.h>


#ifndef __CUDACC_RTC__ 
#define __CUDACC_RTC__

#endif

#include <algorithm>
#include <iostream>

#include ""
#include "Convolution.cuh"
#include <random>
#include <cmath>
using namespace std;

__global__ void ConvolutionKernel(float* d_Input, float* d_Filter, float* d_Output, int _outputWidth, int _convolutionInputWidth, int filterHeight, int filterWidth)
{
	//These define indecies for output matrix;
	int outputRowIndex = blockIdx.x;
	int outputColumnIndex = threadIdx.x;

	//Starts from "top left" of current block of pixels being processed
	int inputRowIndex = blockIdx.x;
	int inputColumnIndex = threadIdx.x;

	int outputArrayIndex = outputRowIndex * _outputWidth + outputColumnIndex;

	int inputArrayIndex = 0;

	float result = 0;
	int filterIndex = 0;
	int temp = 0;
	for (int row = 0; row < filterHeight; row++)
	{
		inputColumnIndex = threadIdx.x;

		for (int col = 0; col < filterWidth; col++)
		{
			inputArrayIndex = inputRowIndex * _convolutionInputWidth + inputColumnIndex;

			result += d_Input[inputArrayIndex] * d_Filter[filterIndex];
			filterIndex++;
			inputColumnIndex += 1;
		}
		inputRowIndex += 1;

	}

	d_Output[outputArrayIndex] = result;
};

__global__ void ConvolutionPaddingKernel(float* d_UnpaddedInput, float* d_Output, int _paddedInputWidth, int _unpaddedInputWidth, int _unpaddedInputHeight)
{
	int rowWriteIndex = (blockIdx.x + 1) * 2 + threadIdx.x;
	int columnWriteIndex = 2;

	int inputRowReadIndex = (blockIdx.x * 2) + threadIdx.x;
	int inputColumnReadIndex = 0;

	int arrayPosition = rowWriteIndex * _paddedInputWidth + columnWriteIndex;
	int inputArrayPosition = inputRowReadIndex * _unpaddedInputWidth + inputColumnReadIndex;

	int var = 0;
	for (int i = 0; i < _unpaddedInputWidth; i++)
	{
		d_Output[arrayPosition] = d_UnpaddedInput[inputArrayPosition];
		arrayPosition++;
		inputArrayPosition++;
	}
}


Convolution::Convolution(int _filterSize, int _paddingSize)
{
	m_FilterSize = _filterSize;
	m_PaddingSize = _paddingSize;
	InitializeFilter();
	//TODO FIX INITIALIZATION ===========================================================================
	//TODO FIX INITIALIZATION ===========================================================================
	//TODO FIX INITIALIZATION ===========================================================================
	//TODO FIX INITIALIZATION ===========================================================================
	//TODO FIX INITIALIZATION ===========================================================================
}


void Convolution::ForwardPass(float* forwardPassInput, int fwdPassHeight, int fwdPassWidth)
{

	m_InputMatrixHeight = fwdPassHeight;
	m_InputMatrixWidth = fwdPassWidth;

	m_OutputMatrixHeight = m_InputMatrixHeight - 2;
	m_OutputMatrixWidth = m_InputMatrixWidth - 2;

	int arrayLength = fwdPassHeight * fwdPassWidth;
	size_t inputSize = arrayLength * sizeof(float);

	m_InputMatrix = new float[arrayLength];
	m_OutputMatrix = new float[m_OutputMatrixHeight * m_OutputMatrixWidth];

	memcpy(m_InputMatrix, forwardPassInput, inputSize);



	int rowShifts = m_OutputMatrixHeight;
	int columnShifts = m_OutputMatrixWidth;

	int elementsInInput = m_InputMatrixHeight * m_InputMatrixWidth;
	int elementsInOutput = m_OutputMatrixHeight * m_OutputMatrixWidth;
	std::cout << "Number of row shifts " << rowShifts << std::endl;
	std::cout << "Number of column shifts " << columnShifts << std::endl;

	std::cout << "Input elements " << elementsInInput << std::endl;
	std::cout << "Output elements" << elementsInOutput << std::endl;

	dim3 blockGrid(rowShifts, 1, 1);
	dim3 threads(columnShifts, 1, 1);

	size_t inputElementCount = m_InputMatrixHeight * m_InputMatrixWidth;
	size_t filterMatrixElementCount = m_FilterSize * m_FilterSize;
	size_t outputElementCount = m_OutputMatrixHeight * m_OutputMatrixWidth;

	int inputByteCount = inputElementCount * sizeof(float);
	int filterByteCount = filterMatrixElementCount * sizeof(float);
	int outputByteCount = outputElementCount * sizeof(float);
	std::cout << "Input element count " << inputElementCount << std::endl;
	std::cout << "Filter element count " << filterMatrixElementCount << std::endl;
	std::cout << "Output element count " << outputByteCount << std::endl;

	//Define pointers for deviceMemory locations
	float* d_Input;
	float* d_Filter;
	float* d_Output;


	//Allocate memory
	hipMalloc((void**)&d_Input, inputByteCount);
	hipMalloc((void**)&d_Filter, filterByteCount);
	hipMalloc((void**)&d_Output, outputByteCount);


	//Copy m_Filter into global device memory m_InputMatrix -> d_Input
	hipMemcpy(d_Input, m_InputMatrix, inputByteCount, hipMemcpyHostToDevice);
	hipMemcpy(d_Filter, m_Filter, filterByteCount, hipMemcpyHostToDevice);

	ConvolutionKernel << <blockGrid, threads >> > (d_Input, d_Filter, d_Output, m_OutputMatrixWidth, m_InputMatrixWidth,  m_FilterSize, m_FilterSize);
	hipDeviceSynchronize();

	hipMemcpy(m_OutputMatrix, d_Output, outputByteCount, hipMemcpyDeviceToHost);

}


void Convolution::BackwardPass(float* backpropInput, int backPassHeight, int backPassWidth)
{
	m_BackpropInputMatrixHeight = backPassHeight;
	m_BackpropInputMatrixWidth = backPassWidth;

	m_BackpropOutputMatrixHeight = m_BackpropInputMatrixHeight + 2;
	m_BackpropOutputMatrixWidth = m_BackpropInputMatrixWidth + 2;

	int arrayLength = backPassHeight * backPassWidth;
	size_t inputSize = arrayLength * sizeof(float);

	m_BackPropInputMatrix = new float[arrayLength];
	m_BackpropagationOutput = new float[m_BackpropOutputMatrixHeight * m_BackpropOutputMatrixWidth];

	memcpy(m_BackPropInputMatrix, backpropInput, inputSize);

	//Main backprop

	int rowShifts = m_BackpropOutputMatrixHeight;
	int columnShifts = m_BackpropOutputMatrixWidth;

	int elementsInInput = m_BackpropInputMatrixHeight * m_BackpropInputMatrixWidth;
	int elementsInOutput = m_BackpropOutputMatrixHeight * m_BackpropOutputMatrixWidth;
	std::cout << "Number of row shifts " << rowShifts << std::endl;
	std::cout << "Number of column shifts " << columnShifts << std::endl;

	std::cout << "Input elements " << elementsInInput << std::endl;
	std::cout << "Output elements" << elementsInOutput << std::endl;

	dim3 blockGrid(rowShifts, 1, 1);
	dim3 threads(columnShifts, 1, 1);

	size_t inputElementCount = m_BackpropInputMatrixHeight * m_BackpropInputMatrixWidth;
	size_t filterMatrixElementCount = m_FilterSize * m_FilterSize;
	size_t outputElementCount = m_BackpropOutputMatrixHeight * m_BackpropOutputMatrixWidth;

	int inputByteCount = inputElementCount * sizeof(float);
	int filterByteCount = filterMatrixElementCount * sizeof(float);
	int outputByteCount = outputElementCount * sizeof(float);

	std::cout << "Input element count " << inputElementCount << std::endl;
	std::cout << "Filter element count " << filterMatrixElementCount << std::endl;
	std::cout << "Output element count " << outputByteCount << std::endl;

	//Define pointers for deviceMemory locations
	float* d_Input;
	float* d_Filter;
	float* d_Output;


	//Allocate memory
	hipMalloc((void**)&d_Input, inputByteCount);
	hipMalloc((void**)&d_Filter, filterByteCount);
	hipMalloc((void**)&d_Output, outputByteCount);


	//Copy m_Filter into global device memory m_InputMatrix -> d_Input
	hipMemcpy(d_Input, m_BackPropInputMatrix, inputByteCount, hipMemcpyHostToDevice);
	hipMemcpy(d_Filter, m_FlippedFilter, filterByteCount, hipMemcpyHostToDevice);

	ConvolutionKernel << <blockGrid, threads >> > (d_Input, d_Filter, d_Output, m_BackpropOutputMatrixWidth, m_BackpropInputMatrixWidth,  m_FilterSize, m_FilterSize);
	hipDeviceSynchronize();

	hipFree(d_Input);
	hipFree(d_Filter);
	hipFree(d_Output);

	hipMemcpy(m_BackpropagationOutput, d_Output, outputByteCount, hipMemcpyDeviceToHost);

	//Filter backprop
	FilterBackprop(backpropInput, backPassHeight, backPassWidth);
}

void Convolution::FilterBackprop(float* backpropInput, int backPassHeight, int backPassWidth)
{
	size_t fwdInputElementCount = m_InputMatrixHeight * m_InputMatrixWidth;
	size_t filterEqivElementCount = m_OutputMatrixHeight * m_OutputMatrixWidth;
	size_t  filterOutputElementCount = m_FilterSize * m_FilterSize;

	int fwdInputByteCount = fwdInputElementCount * sizeof(float);
	int filterEqivByteCount = filterEqivElementCount * sizeof(float);
	int filterOutputByteCount = filterOutputElementCount * sizeof(float);

	float* d_FwdInput;
	float* d_FilterEquiv;
	float* d_FilterOutput;

	//Allocate memory
	hipMalloc((void**)&d_FwdInput, fwdInputByteCount);
	hipMalloc((void**)&d_FilterEquiv, filterEqivByteCount);
	hipMalloc((void**)&d_FilterOutput, filterOutputByteCount);

	//Copy m_Filter into global device memory m_InputMatrix -> d_Input
	hipMemcpy(d_FwdInput, m_InputMatrix, fwdInputByteCount, hipMemcpyHostToDevice);
	hipMemcpy(d_FilterEquiv, m_OutputMatrix, filterEqivByteCount, hipMemcpyHostToDevice);

	int rowShifts = m_FilterSize;
	int columnShifts = m_FilterSize;

	dim3 blockGrid(rowShifts, 1, 1);
	dim3 threads(columnShifts, 1, 1);

	ConvolutionKernel << <blockGrid, threads >> > (d_FwdInput, d_FilterEquiv, d_FilterOutput, m_FilterSize, m_InputMatrixWidth,  m_OutputMatrixHeight, m_OutputMatrixWidth);
	hipDeviceSynchronize();

	hipMemcpy(m_FilterBackpropResult, d_FilterOutput, filterOutputByteCount, hipMemcpyDeviceToHost);
}


void Convolution::PadBackpropInput()
{
	m_PaddedInputHeight = m_BackpropInputMatrixHeight + 2 * m_PaddingSize;
	m_PaddedInputWidth = m_BackpropInputMatrixWidth + 2 * m_PaddingSize;
	int elementsInPaddedInput = m_PaddedInputHeight * m_PaddedInputWidth;

	m_PaddedBackpropInput = new float[elementsInPaddedInput];

	memset(m_PaddedBackpropInput, 0, elementsInPaddedInput * sizeof(float));

	float* d_Output;
	float* d_UnpaddedInput;

	size_t outputByteCount = elementsInPaddedInput * sizeof(float);
	size_t unpaddedInputByteCount = (m_BackpropInputMatrixHeight * m_BackpropInputMatrixWidth) * sizeof(float);

	hipMalloc((void**)&d_Output, outputByteCount);
	hipMalloc((void**)&d_UnpaddedInput, unpaddedInputByteCount);

	hipMemcpy(d_UnpaddedInput, m_BackPropInputMatrix, unpaddedInputByteCount, hipMemcpyHostToDevice);
	hipMemcpy(d_Output, m_PaddedBackpropInput, unpaddedInputByteCount, hipMemcpyHostToDevice);

	int numberOfBlocks = m_BackpropInputMatrixHeight / 2;
	cout << "Launching blocks: " << numberOfBlocks << endl;
	dim3 blockGrid(numberOfBlocks, 1, 1);
	dim3 threads(2, 1, 1);

	ConvolutionPaddingKernel << <blockGrid, threads >> > (d_UnpaddedInput, d_Output, m_PaddedInputWidth, m_BackpropInputMatrixWidth, m_BackpropInputMatrixHeight);
	hipDeviceSynchronize();

	hipMemcpy(m_PaddedBackpropInput, d_Output, outputByteCount, hipMemcpyDeviceToHost);
}

void Convolution::UpdateModule()
{
	for (int rowIndex = 0; rowIndex < m_FilterSize; ++rowIndex)
	{
		for (int columnIndex = 0; columnIndex < m_FilterSize; ++columnIndex)
		{
			int index = rowIndex * m_FilterSize + columnIndex;

			float filterBackpropValue = m_FilterBackpropResult[index];

			float newV = m_HyperParam_Beta1 * filterBackpropValue + (1 - m_HyperParam_Beta1) * filterBackpropValue;
			float newS = m_HyperParam_Beta2 * filterBackpropValue + (1 - m_HyperParam_Beta2) * filterBackpropValue;

			float newVCorrected = newV / (1 - pow(m_HyperParam_Beta1, m_HyperParam_T));
			float newSCorrected = newS / (1 - pow(m_HyperParam_Beta2, m_HyperParam_T));

			m_AdamOptimizer_VMatrix[index] = newV;
			m_AdamOptimizer_SMatrix[index] = newS;

			m_AdamOptimizer_Corrected_VMatrix[index] = newVCorrected;
			m_AdamOptimizer_Corrected_SMatrix[index] = newSCorrected;

			float oldFilterValue = m_Filter[index];
			float newF = oldFilterValue - m_HyperParam_alpha * (newVCorrected / sqrt(newSCorrected + m_HyperParam_Epsilon));

			m_Filter[index] = newF;
		}
	}
}


void Convolution::Dialate(float* _input, float* _output)
{

}

void Convolution::InitializeFilter()
{
	std::random_device rd{};
	std::mt19937 gen{ rd() };
	std::normal_distribution<> distribution{ 1.42,2 };
	m_Filter = new float[m_FilterSize * m_FilterSize];

	for (int i = 0; i < m_FilterSize * m_FilterSize; ++i)
	{
		m_Filter[i] = i + 1;//distribution(gen);  //TODO FIX INITIALIZATION
	}
}

void Convolution::FlipFilter()
{
	int filterArraySize = m_FilterSize * m_FilterSize;
	m_FlippedFilter = new float[filterArraySize];

	int k = 0;

	//Loop from back and assign value to new array
	for (int i = filterArraySize - 1; i >= 0; ) {
		m_FlippedFilter[k++] = m_Filter[i--];
	}
}




