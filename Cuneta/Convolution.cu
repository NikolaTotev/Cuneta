#include "hip/hip_runtime.h"
#include <hip/hip_vector_types.h>

#ifndef __HIPCC__  
#define __HIPCC__
#endif

#ifndef  __CUDART_API_PER_THREAD_DEFAULT_STREAM
#define  __CUDART_API_PER_THREAD_DEFAULT_STREAM
#endif
#include <hip/hip_runtime_api.h>


#ifndef __CUDACC_RTC__ 
#define __CUDACC_RTC__

#endif

#include <algorithm>
#include <iostream>

#include ""
#include "Convolution.cuh"
#include <random>
#include <cmath>

#ifndef __HIPCC__  
#define __HIPCC__
#endif

#ifndef __CUDACC_RTC__ 
#define __CUDACC_RTC__
#endif

#include <hip/device_functions.h>
using namespace std;

__global__ void ConvolutionKernel(float* d_Input, float* d_Filter, float* d_Output, int _outputWidth, int _convolutionInputWidth, int filterHeight, int filterWidth)
{
	//These define indecies for output matrix;
	int outputRowIndex = blockIdx.x;
	int outputColumnIndex = threadIdx.x;

	//Starts from "top left" of current block of pixels being processed
	int inputRowIndex = blockIdx.x;
	int inputColumnIndex = threadIdx.x;

	int outputArrayIndex = outputRowIndex * _outputWidth + outputColumnIndex;

	int inputArrayIndex = 0;

	float result = 0;
	int filterIndex = 0;
	int temp = 0;
	for (int row = 0; row < filterHeight; row++)
	{
		inputColumnIndex = threadIdx.x;

		for (int col = 0; col < filterWidth; col++)
		{
			inputArrayIndex = inputRowIndex * _convolutionInputWidth + inputColumnIndex;

			result += d_Input[inputArrayIndex] * d_Filter[filterIndex];
			filterIndex++;
			inputColumnIndex += 1;
		}
		inputRowIndex += 1;

	}

	d_Output[outputArrayIndex] = result;
}

__global__ void LayerConvolutionKernal(float** _inputs, float** _filters, float** _outputs, float* biases, int _numberOfInputs, int _outputWidth, int _inputWidth, int _filterSize)
{
	int inputSelectionIndex = threadIdx.x;
	int filterSelectionIndex = blockIdx.z* _numberOfInputs + threadIdx.x;
	int outputSelectionIndex = blockIdx.z;

	float* selectedInput = _inputs[inputSelectionIndex];
	float* selectedFilter = _filters[filterSelectionIndex];
	float* selectedOutput = _outputs[outputSelectionIndex];

	int inputStartReadRowIndex = blockIdx.x;
	int inputStartReadColumnIndex = blockIdx.y;

	int outputWriteRowIndex = blockIdx.x;
	int outputWriteColumnIndex = blockIdx.y;

	int inputArrayIndex = 0;

	int outputArrayIndex = outputWriteRowIndex * _outputWidth + outputWriteColumnIndex;

	float result = 0;
	int filterIndex = 0;
	int temp = 0;
	for (int row = 0; row < _filterSize; row++)
	{
		inputStartReadColumnIndex = blockIdx.y;

		for (int col = 0; col < _filterSize; col++)
		{
			inputArrayIndex = inputStartReadRowIndex * _inputWidth + inputStartReadColumnIndex;

			result += selectedInput[inputArrayIndex] * selectedFilter[filterIndex];
			filterIndex++;
			inputStartReadColumnIndex += 1;
		}
		inputStartReadRowIndex += 1;
	}
	
	//selectedOutput[outputArrayIndex] = filterSelectionIndex;
	atomicAdd(&selectedOutput[outputArrayIndex], result);
};


__global__ void ConvolutionPaddingKernel(float* d_UnpaddedInput, float* d_Output, int _paddedInputWidth, int _unpaddedInputWidth, int _unpaddedInputHeight)
{
	int rowWriteIndex = (blockIdx.x + 1) * 2 + threadIdx.x;
	int columnWriteIndex = 2;

	int inputRowReadIndex = (blockIdx.x * 2) + threadIdx.x;
	int inputColumnReadIndex = 0;

	int arrayPosition = rowWriteIndex * _paddedInputWidth + columnWriteIndex;
	int inputArrayPosition = inputRowReadIndex * _unpaddedInputWidth + inputColumnReadIndex;

	int var = 0;
	for (int i = 0; i < _unpaddedInputWidth; i++)
	{
		d_Output[arrayPosition] = d_UnpaddedInput[inputArrayPosition];
		arrayPosition++;
		inputArrayPosition++;
	}
}


Convolution::Convolution(int _filterSize, int _paddingSize, int _numberOfInputs, int _numberOfOutputs, int _inputHeight, int _inputWidth)
{
	m_FilterSize = _filterSize;
	m_PaddingSize = _paddingSize;
	m_AdamOptimizer_VMatrix = new float[m_FilterSize * m_FilterSize];
	m_AdamOptimizer_SMatrix = new float[m_FilterSize * m_FilterSize];

	m_AdamOptimizer_Corrected_VMatrix = new float[m_FilterSize * m_FilterSize];
	m_AdamOptimizer_Corrected_SMatrix = new float[m_FilterSize * m_FilterSize];

	memset(m_AdamOptimizer_VMatrix, 0, m_FilterSize * m_FilterSize * sizeof(float));
	memset(m_AdamOptimizer_SMatrix, 0, m_FilterSize * m_FilterSize * sizeof(float));

	L_FORWARD_NumberOf_INPUTS = _numberOfInputs;
	L_FORWARD_NumerOf_OUTPUTS = _numberOfOutputs;

	L_BACKWARD_NumberOf_INPUTS = L_FORWARD_NumerOf_OUTPUTS;
	L_BACKWARD_NumberOf_OUTPUTS = L_FORWARD_NumberOf_INPUTS;

	L_FORWARD_InputLayer_HEIGHT = _inputHeight;
	L_FORWARD_InputLayer_WIDTH = _inputWidth;

	L_FORWARD_OutputLayer_HEIGHT = _inputHeight / 2;
	L_FORWARD_OutputLayer_WIDTH = _inputWidth / 2;

	L_BACKWARD_InputLayer_HEIGHT = L_FORWARD_OutputLayer_HEIGHT;
	L_BACKWARD_InputLayer_WIDTH = L_FORWARD_OutputLayer_WIDTH;

	L_BACKWARD_OutputLayer_HEIGHT = L_FORWARD_InputLayer_HEIGHT;
	L_BACKWARD_OutputLayer_WIDTH = L_FORWARD_InputLayer_WIDTH;

	L_FORWARD_Pass_INPUTS = new float* [L_FORWARD_NumberOf_INPUTS];
	L_FORWARD_Pass_OUTPUTS = new float* [L_FORWARD_NumerOf_OUTPUTS];

	L_BACKWARD_Pass_INPUTS = new float* [L_FORWARD_NumerOf_OUTPUTS];
	L_BACKWARD_Pass_OUTPUTS = new float* [L_FORWARD_NumberOf_INPUTS];

	L_NumberOf_FILTERS = L_FORWARD_NumberOf_INPUTS * L_FORWARD_NumerOf_OUTPUTS;

	L_Filters = new float* [L_NumberOf_FILTERS];

	L_Baises = new float[L_NumberOf_FILTERS];

	InitializeFilter();
}


void Convolution::ForwardPass(float* forwardPassInput, int fwdPassHeight, int fwdPassWidth)
{

	m_InputMatrixHeight = fwdPassHeight;
	m_InputMatrixWidth = fwdPassWidth;

	m_OutputMatrixHeight = m_InputMatrixHeight - 2;
	m_OutputMatrixWidth = m_InputMatrixWidth - 2;

	int arrayLength = fwdPassHeight * fwdPassWidth;
	size_t inputSize = arrayLength * sizeof(float);

	m_InputMatrix = new float[arrayLength];
	m_OutputMatrix = new float[m_OutputMatrixHeight * m_OutputMatrixWidth];

	memcpy(m_InputMatrix, forwardPassInput, inputSize);


	int rowShifts = m_OutputMatrixHeight;
	int columnShifts = m_OutputMatrixWidth;


	dim3 blockGrid(rowShifts, 1, 1);
	dim3 threads(columnShifts, 1, 1);

	size_t inputElementCount = m_InputMatrixHeight * m_InputMatrixWidth;
	size_t filterMatrixElementCount = m_FilterSize * m_FilterSize;
	size_t outputElementCount = m_OutputMatrixHeight * m_OutputMatrixWidth;

	int inputByteCount = inputElementCount * sizeof(float);
	int filterByteCount = filterMatrixElementCount * sizeof(float);
	int outputByteCount = outputElementCount * sizeof(float);


	//Define pointers for deviceMemory locations
	float* d_Input;
	float* d_Filter;
	float* d_Output;


	//Allocate memory
	hipMalloc((void**)&d_Input, inputByteCount);
	hipMalloc((void**)&d_Filter, filterByteCount);
	hipMalloc((void**)&d_Output, outputByteCount);


	//Copy m_Filter into global device memory m_InputMatrix -> d_Input
	hipMemcpy(d_Input, m_InputMatrix, inputByteCount, hipMemcpyHostToDevice);
	hipMemcpy(d_Filter, m_Filter, filterByteCount, hipMemcpyHostToDevice);

	ConvolutionKernel << <blockGrid, threads >> > (d_Input, d_Filter, d_Output, m_OutputMatrixWidth, m_InputMatrixWidth, m_FilterSize, m_FilterSize);
	hipDeviceSynchronize();

	hipMemcpy(m_OutputMatrix, d_Output, outputByteCount, hipMemcpyDeviceToHost);

}


void Convolution::BackwardPass(float* backpropInput, int backPassHeight, int backPassWidth)
{
	m_BackpropInputMatrixHeight = backPassHeight; ///OK
	m_BackpropInputMatrixWidth = backPassWidth; ///OK

	m_BackpropOutputMatrixHeight = m_BackpropInputMatrixHeight + 2; ///OK
	m_BackpropOutputMatrixWidth = m_BackpropInputMatrixWidth + 2; ///OK

	int arrayLength = backPassHeight * backPassWidth; ///OK
	size_t inputSize = arrayLength * sizeof(float); ///OK

	m_BackPropInputMatrix = new float[arrayLength]; ///OK
	m_BackpropagationOutput = new float[m_BackpropOutputMatrixHeight * m_BackpropOutputMatrixWidth]; ///OK

	memcpy(m_BackPropInputMatrix, backpropInput, inputSize); ///OK

	//Main backprop

	FlipFilter(); ///OK
	PadBackpropInput(); ///OK

	int rowShifts = m_BackpropOutputMatrixHeight;
	int columnShifts = m_BackpropOutputMatrixWidth;


	dim3 blockGrid(rowShifts, 1, 1); ///OK
	dim3 threads(columnShifts, 1, 1); ///OK

	size_t inputElementCount = m_PaddedInputHeight * m_PaddedInputWidth; ///OK
	size_t filterMatrixElementCount = m_FilterSize * m_FilterSize; ///OK
	size_t outputElementCount = m_BackpropOutputMatrixHeight * m_BackpropOutputMatrixWidth; ///OK

	int inputByteCount = inputElementCount * sizeof(float); ///OK
	int filterByteCount = filterMatrixElementCount * sizeof(float); ///OK
	int outputByteCount = outputElementCount * sizeof(float); ///OK

	//Define pointers for deviceMemory locations
	float* d_Input; ///OK
	float* d_Filter; ///OK
	float* d_Output; ///OK


	//Allocate memory
	hipMalloc((void**)&d_Input, inputByteCount); ///OK
	hipMalloc((void**)&d_Filter, filterByteCount); ///OK
	hipMalloc((void**)&d_Output, outputByteCount); ///OK


	//Copy m_Filter into global device memory m_InputMatrix -> d_Input
	hipMemcpy(d_Input, m_PaddedBackpropInput, inputByteCount, hipMemcpyHostToDevice); ///OK
	hipMemcpy(d_Filter, m_FlippedFilter, filterByteCount, hipMemcpyHostToDevice); ///OK

	ConvolutionKernel << <blockGrid, threads >> > (d_Input, d_Filter, d_Output, m_BackpropOutputMatrixWidth, m_PaddedInputWidth, m_FilterSize, m_FilterSize);
	hipDeviceSynchronize(); ///OK


	hipMemcpy(m_BackpropagationOutput, d_Output, outputByteCount, hipMemcpyDeviceToHost); ///OK

	//Filter backprop
	FilterBackprop(backpropInput, backPassHeight, backPassWidth); ///OK


	hipFree(d_Input); ///OK
	hipFree(d_Filter); ///OK
	hipFree(d_Output); ///OK
}

void Convolution::FilterBackprop(float* backpropInput, int backPassHeight, int backPassWidth)
{
	size_t fwdInputElementCount = m_InputMatrixHeight * m_InputMatrixWidth; ///OK
	size_t filterEqivElementCount = m_OutputMatrixHeight * m_OutputMatrixWidth; ///OK
	size_t  filterOutputElementCount = m_FilterSize * m_FilterSize; ///OK

	m_FilterBackpropResult = new float[filterOutputElementCount];

	int fwdInputByteCount = fwdInputElementCount * sizeof(float); ///OK
	int filterEqivByteCount = filterEqivElementCount * sizeof(float); ///OK
	int filterOutputByteCount = filterOutputElementCount * sizeof(float); ///OK

	float* d_FwdInput; ///OK
	float* d_FilterEquiv; ///OK
	float* d_FilterOutput; ///OK

	//Allocate memory
	hipMalloc((void**)&d_FwdInput, fwdInputByteCount); ///OK
	hipMalloc((void**)&d_FilterEquiv, filterEqivByteCount); ///OK
	hipMalloc((void**)&d_FilterOutput, filterOutputByteCount); ///OK

	//Copy m_Filter into global device memory m_InputMatrix -> d_Input
	hipMemcpy(d_FwdInput, m_InputMatrix, fwdInputByteCount, hipMemcpyHostToDevice); ///OK
	hipMemcpy(d_FilterEquiv, m_BackPropInputMatrix, filterEqivByteCount, hipMemcpyHostToDevice); ///OK

	int rowShifts = m_FilterSize; ///OK
	int columnShifts = m_FilterSize; ///OK

	dim3 blockGrid(rowShifts, 1, 1); ///OK
	dim3 threads(columnShifts, 1, 1); ///OK

	ConvolutionKernel << <blockGrid, threads >> > (d_FwdInput, d_FilterEquiv, d_FilterOutput, m_FilterSize, m_InputMatrixWidth, m_OutputMatrixHeight, m_OutputMatrixWidth);
	hipDeviceSynchronize();

	hipMemcpy(m_FilterBackpropResult, d_FilterOutput, filterOutputByteCount, hipMemcpyDeviceToHost);
}


void Convolution::LayerForwardPass(float** _inputs)
{
	int inputSize = L_FORWARD_InputLayer_HEIGHT * L_FORWARD_InputLayer_WIDTH;

	int filterSize = m_FilterSize * m_FilterSize;

	int outputSize = L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH;

	size_t inputByteCount = inputSize * sizeof(float);
	size_t filterByteCount = filterSize * sizeof(float);
	size_t outputByteCount = outputSize * sizeof(float);

	for (int inputNumber = 0; inputNumber < L_FORWARD_NumberOf_INPUTS; ++inputNumber)
	{
		L_FORWARD_Pass_INPUTS[inputNumber] = new float[inputSize];
		memcpy(L_FORWARD_Pass_INPUTS[inputNumber], _inputs[inputNumber], inputByteCount);
	}

	for (int outputNumber = 0; outputNumber < L_FORWARD_NumerOf_OUTPUTS; ++outputNumber)
	{
		L_FORWARD_Pass_OUTPUTS[outputNumber] = new float[outputSize];
	}

	int numberOfBlockx_X = L_FORWARD_OutputLayer_HEIGHT;
	int numberOfBlocks_Y = L_FORWARD_OutputLayer_WIDTH;
	int numberOfBlocks_Z = L_FORWARD_NumerOf_OUTPUTS;
	int numberOfThreadsPerBlock = L_FORWARD_NumberOf_INPUTS;

	// create intermediate host array for storage of device row-pointers

	// create top-level device array pointer
	float** h_Inputs = new float* [L_FORWARD_NumberOf_INPUTS];  //(float**)malloc(L_FORWARD_NumberOf_INPUTS * sizeof(int*));
	float** h_Filters = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(int*));
	float** h_Outputs = new float* [L_FORWARD_NumerOf_OUTPUTS]; //(float**)malloc(L_FORWARD_NumerOf_OUTPUTS * sizeof(int*));

	float* d_Biases;
	hipMalloc((void**)&d_Biases, L_NumberOf_FILTERS * sizeof(float));


	float** d_InputPointerArray;
	hipMalloc((void**)&d_InputPointerArray, L_FORWARD_NumberOf_INPUTS * sizeof(int*));

	float** d_FilterPointerArray;
	hipMalloc((void**)&d_FilterPointerArray, L_NumberOf_FILTERS * sizeof(int*));

	float** d_OutputPointerArray;
	hipMalloc((void**)&d_OutputPointerArray, L_FORWARD_NumerOf_OUTPUTS * sizeof(int*));


	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_FORWARD_NumberOf_INPUTS; i++) {
		hipMalloc(&h_Inputs[i], inputByteCount);
		hipMemcpy(h_Inputs[i], L_FORWARD_Pass_INPUTS[i], inputByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Filters[i], filterByteCount);
		hipMemcpy(h_Filters[i], L_Filters[i], filterByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_FORWARD_NumerOf_OUTPUTS; i++) {
		hipMalloc(&h_Outputs[i], outputByteCount);
		hipMemset(&h_Outputs[i], 0, outputByteCount);
	}

	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_InputPointerArray, h_Inputs, L_FORWARD_NumberOf_INPUTS * sizeof(int*), hipMemcpyHostToDevice);
	hipMemcpy(d_FilterPointerArray, h_Filters, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyHostToDevice);
	hipMemcpy(d_OutputPointerArray, h_Outputs, L_FORWARD_NumerOf_OUTPUTS * sizeof(float*), hipMemcpyHostToDevice);

	dim3 blockGrid(numberOfBlockx_X, numberOfBlocks_Y, numberOfBlocks_Z); ///OK
	dim3 threads(numberOfThreadsPerBlock, 1, 1); ///OK

	LayerConvolutionKernal << <blockGrid, threads >> > (d_InputPointerArray, d_FilterPointerArray, d_OutputPointerArray, d_Biases, L_FORWARD_NumberOf_INPUTS, L_FORWARD_OutputLayer_WIDTH, L_FORWARD_InputLayer_WIDTH, m_FilterSize);
	hipDeviceSynchronize();
	testIt = new float[outputSize];
	hipMemcpy(h_Outputs, d_OutputPointerArray, L_FORWARD_NumerOf_OUTPUTS * sizeof(int*), hipMemcpyDeviceToHost);
	for (size_t i = 0; i < L_FORWARD_NumerOf_OUTPUTS; i++) {

		hipMemcpy(testIt, h_Outputs[i], outputByteCount, hipMemcpyDeviceToHost);
		memcpy(L_FORWARD_Pass_OUTPUTS[i], testIt, outputByteCount);
		//hipFree(h_Outputs[i]);
	}
	//hipFree(d_OutputPointerArray);
	//delete[] h_Outputs;

	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_FORWARD_NumberOf_INPUTS; i++) {
		hipFree(&d_InputPointerArray[i]);
	}
	hipFree(d_InputPointerArray);
	delete[] h_Inputs;

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipFree(&d_FilterPointerArray[i]);
	}
	hipFree(d_FilterPointerArray);
	delete[] h_Filters;
}


void Convolution::LayerBackwardPass(float** _backpropInput)
{

}



void Convolution::PadBackpropInput()
{
	m_PaddedInputHeight = m_BackpropInputMatrixHeight + 2 * m_PaddingSize;
	m_PaddedInputWidth = m_BackpropInputMatrixWidth + 2 * m_PaddingSize;
	int elementsInPaddedInput = m_PaddedInputHeight * m_PaddedInputWidth;

	m_PaddedBackpropInput = new float[elementsInPaddedInput];

	memset(m_PaddedBackpropInput, 0, elementsInPaddedInput * sizeof(float));

	float* d_Output;
	float* d_UnpaddedInput;

	size_t outputByteCount = elementsInPaddedInput * sizeof(float);
	size_t unpaddedInputByteCount = (m_BackpropInputMatrixHeight * m_BackpropInputMatrixWidth) * sizeof(float);

	hipMalloc((void**)&d_Output, outputByteCount);
	hipMalloc((void**)&d_UnpaddedInput, unpaddedInputByteCount);

	hipMemcpy(d_UnpaddedInput, m_BackPropInputMatrix, unpaddedInputByteCount, hipMemcpyHostToDevice);
	hipMemcpy(d_Output, m_PaddedBackpropInput, unpaddedInputByteCount, hipMemcpyHostToDevice);

	int numberOfBlocks = m_BackpropInputMatrixHeight / 2;
	cout << "Launching blocks: " << numberOfBlocks << endl;
	dim3 blockGrid(numberOfBlocks, 1, 1);
	dim3 threads(2, 1, 1);

	ConvolutionPaddingKernel << <blockGrid, threads >> > (d_UnpaddedInput, d_Output, m_PaddedInputWidth, m_BackpropInputMatrixWidth, m_BackpropInputMatrixHeight);
	hipDeviceSynchronize();

	hipMemcpy(m_PaddedBackpropInput, d_Output, outputByteCount, hipMemcpyDeviceToHost);
}

void Convolution::UpdateModule()
{
	for (int rowIndex = 0; rowIndex < m_FilterSize; ++rowIndex)
	{
		for (int columnIndex = 0; columnIndex < m_FilterSize; ++columnIndex)
		{
			int index = rowIndex * m_FilterSize + columnIndex;

			float filterBackpropValue = m_FilterBackpropResult[index];
			float oldV = m_AdamOptimizer_VMatrix[index];
			float oldS = m_AdamOptimizer_SMatrix[index];

			float newV = m_HyperParam_Beta1 * oldV + (1 - m_HyperParam_Beta1) * filterBackpropValue;
			float newS = m_HyperParam_Beta2 * oldS + (1 - m_HyperParam_Beta2) * filterBackpropValue;

			float newVCorrected = newV / (1 - pow(m_HyperParam_Beta1, m_HyperParam_T));
			float newSCorrected = newS / (1 - pow(m_HyperParam_Beta2, m_HyperParam_T));

			m_AdamOptimizer_VMatrix[index] = newV;
			m_AdamOptimizer_SMatrix[index] = newS;

			m_AdamOptimizer_Corrected_VMatrix[index] = newVCorrected;
			m_AdamOptimizer_Corrected_SMatrix[index] = newSCorrected;

			float oldFilterValue = m_Filter[index];
			float newF = oldFilterValue - m_HyperParam_alpha * (newVCorrected / sqrt(newSCorrected + m_HyperParam_Epsilon));

			m_Filter[index] = newF;
		}
	}
}


void Convolution::Dialate(float* _input, float* _output)
{

}

void Convolution::InitializeFilter()
{
	std::random_device rd{};
	std::mt19937 gen{ rd() };
	std::normal_distribution<> distribution{ 1,2 };
	m_Filter = new float[m_FilterSize * m_FilterSize];

	for (int i = 0; i < m_FilterSize * m_FilterSize; ++i)
	{
		m_Filter[i] = distribution(gen);  //TODO FIX INITIALIZATION
	}
}

void Convolution::LayerFilterInitialization()
{
	std::random_device rd{};
	std::mt19937 gen{ rd() };
	std::normal_distribution<> distribution{ 1,2 };

	int filterElementCount = m_FilterSize * m_FilterSize;

	for (int filterNumber = 0; filterNumber < L_NumberOf_FILTERS; ++filterNumber)
	{
		L_Filters[filterNumber] = new float[filterElementCount];

		for (int i = 0; i < filterElementCount; ++i)
		{
			L_Filters[filterNumber][i] = filterNumber; //distribution(gen);
		}
	}
}


void Convolution::FlipFilter()
{
	int filterArraySize = m_FilterSize * m_FilterSize;
	m_FlippedFilter = new float[filterArraySize];

	int k = 0;

	//Loop from back and assign value to new array
	for (int i = filterArraySize - 1; i >= 0; ) {
		m_FlippedFilter[k++] = m_Filter[i--];
	}
}


void Convolution::SetHyperParams(float _beta1, float _beta2, float _eps, int _t, float _alpha)
{
	m_HyperParam_Beta1 = _beta1;
	m_HyperParam_Beta2 = _beta2;
	m_HyperParam_Epsilon = _eps;
	m_HyperParam_T = _t;
	m_HyperParam_alpha = _alpha;
}


