#include "hip/hip_runtime.h"
#include <hip/hip_vector_types.h>

#ifndef __HIPCC__  
#define __HIPCC__
#endif

#ifndef  __CUDART_API_PER_THREAD_DEFAULT_STREAM
#define  __CUDART_API_PER_THREAD_DEFAULT_STREAM
#endif
#include <hip/hip_runtime_api.h>


#ifndef __CUDACC_RTC__ 
#define __CUDACC_RTC__

#endif

#include <algorithm>
#include <iostream>

#include ""
#include "Convolution.cuh"
#include <random>
#include <cmath>
using namespace std;

__global__ void ConvolutionKernel(float* d_Input, float* d_Toeplitz, float* d_Output, int _toeplitzHeight, int _toeplitzWidth, int _outputHeight, int _outputWidth, int _convolutionInputWidth, int _columnShiftsPerBlock)
{
	int largerSide = fmaxf(_toeplitzHeight, _toeplitzWidth);
	int smallerOutputSide = fminf(_outputHeight, _outputWidth);
	int initialBlockOffset = blockIdx.x * largerSide * _columnShiftsPerBlock;
	int rowIndexWithinBlock = initialBlockOffset + (threadIdx.x * largerSide);
	int outputArrayIndex = blockIdx.x * smallerOutputSide + threadIdx.x;

	/*if (blockIdx.x == 1 && threadIdx.x == 0)
	{
		d_Output[0] = blockIdx.x;
		d_Output[1] = threadIdx.x;
		d_Output[2] = initialBlockOffset;
		d_Output[3] = rowIndexWithinBlock;
		d_Output[4] = outputArrayIndex;
	}*/

	int result = 0;
	for (int i = 0; i < largerSide; ++i)
	{
		result += d_Input[i] * d_Toeplitz[rowIndexWithinBlock];
		rowIndexWithinBlock++;
	}

	d_Output[outputArrayIndex] = result;
};

__global__ void ToeplitzKernel(float* d_Filter, float* d_Output, int _toeplitzHeight, int _toeplitzWidth, int _filterHeight, int _filterWidth, int _convolutionInputWidth, int _columnShiftsPerBlock)
{

	int spacing = _convolutionInputWidth - _filterWidth;
	int largerSide = fmaxf(_toeplitzHeight, _toeplitzWidth);
	int initialBlockOffset = blockIdx.x * largerSide * _columnShiftsPerBlock;
	int rowIndexWithinBlock = initialBlockOffset + (threadIdx.x * largerSide);
	int threadWriteIndex = rowIndexWithinBlock + threadIdx.x + (blockIdx.x * _convolutionInputWidth);

	/*if (blockIdx.x ==0 && threadIdx.x == 0)
	{
		d_Output[0] = blockIdx.x;
		d_Output[1] = threadIdx.x;
		d_Output[2] = initialBlockOffset;
		d_Output[3] = rowIndexWithinBlock;
		d_Output[4] = threadWriteIndex;
	}*/

	int counter = 1;
	int filterReadIndex = 0;
	int skippedPositions = 0;
	bool shouldSkip = false;
	for (int j = 0; j < (_filterHeight * _filterWidth) + (2 * spacing); ++j)
	{
		if (counter == 4 || shouldSkip)
		{
			shouldSkip = true;
			skippedPositions++;
			if (skippedPositions == spacing)
			{
				shouldSkip = false;
				skippedPositions = 0;
			}
			counter = 1;

		}
		else
		{
			d_Output[threadWriteIndex] = d_Filter[filterReadIndex];
			filterReadIndex++;
			counter++;
		}
		threadWriteIndex++;
	}
};

Convolution::Convolution(float* _inputMatrix, int _inputHeight, int _inputWidth, int _filterSize)
{
	m_InputMatrix = _inputMatrix;
	m_InputMatrixHeight = _inputHeight;
	m_InputMatrixWidth = _inputWidth;

	m_OutputMatrixHeight = _inputHeight - 2;
	m_OutputMatrixWidth = _inputWidth - 2;

	filterSize = _filterSize;

	m_OutputMatrix = new float[m_OutputMatrixHeight * m_OutputMatrixWidth];
	InitilizeFilter();
	FilterToToeplitzMatrix();
}

void Convolution::ForwardPass()
{
	int rowShifts = m_OutputMatrixHeight;
	int columnShifts = m_OutputMatrixWidth;

	int elementsInInput = m_InputMatrixHeight * m_InputMatrixWidth;
	int elementsInOutput = m_OutputMatrixHeight * m_OutputMatrixWidth;
	std::cout << "Number of row shifts " << rowShifts << std::endl;
	std::cout << "Number of column shifts " << columnShifts << std::endl;

	std::cout << "Input elements " << elementsInInput << std::endl;
	std::cout << "Output elements" << elementsInOutput << std::endl;

	dim3 blockGrid(rowShifts, 1, 1);
	dim3 threads(columnShifts, 1, 1);

	size_t inputElementCount = m_InputMatrixHeight * m_InputMatrixWidth;
	size_t toeplitzMatrixElementCount = elementsInInput * elementsInOutput;
	size_t outputElementCount = m_OutputMatrixHeight * m_OutputMatrixWidth;

	int inputByteCount = inputElementCount * sizeof(float);
	int toeplitzByteCount = toeplitzMatrixElementCount * sizeof(float);
	int outputByteCount = outputElementCount * sizeof(float);
	std::cout << " Input element count " << inputElementCount << std::endl;
	std::cout << "Toeplitz element count " << toeplitzMatrixElementCount << std::endl;
	std::cout << "Output element count " << outputByteCount << std::endl;

	//Define pointers for deviceMemory locations
	float* d_Input;
	float* d_Toeplitz;
	float* d_Output;


	//Allocate memory
	hipMalloc((void**)&d_Input, inputByteCount);
	hipMalloc((void**)&d_Toeplitz, toeplitzByteCount);
	hipMalloc((void**)&d_Output, outputByteCount);


	//Copy filter into global device memory m_InputMatrix -> d_Input
	cout << "Filter [0]" << filter[0] << endl;
	hipMemcpy(d_Input, m_InputMatrix, inputByteCount, hipMemcpyHostToDevice);
	hipMemcpy(d_Toeplitz, toeplitzMatrix, toeplitzByteCount, hipMemcpyHostToDevice);

	ConvolutionKernel << <blockGrid, threads >> > (d_Input, d_Toeplitz, d_Output, elementsInInput, elementsInOutput, m_OutputMatrixHeight, m_OutputMatrixWidth, m_InputMatrixWidth, columnShifts);
	hipDeviceSynchronize();

	hipMemcpy(m_OutputMatrix, d_Output, outputByteCount, hipMemcpyDeviceToHost);

}


void Convolution::BackwardPass()
{

}


void Convolution::Dialate(float* _input, float* _output)
{

}


void Convolution::FilterToToeplitzMatrix()
{
	int columnShifts = m_OutputMatrixWidth;
	int rowShifts = m_OutputMatrixHeight;
	int elementsInInput = m_InputMatrixHeight * m_InputMatrixWidth;
	int elementsInOutput = m_OutputMatrixHeight * m_OutputMatrixWidth;
	toeplitzMatrix = new float[elementsInInput * elementsInOutput];
	std::cout << "Number of row shifts " << rowShifts << std::endl;
	std::cout << "Number of column shifts " << columnShifts << std::endl;

	std::cout << "Input elements " << elementsInInput << std::endl;
	std::cout << "Output elements" << elementsInOutput << std::endl;

	dim3 blockGrid(rowShifts, 1, 1);
	dim3 threads(columnShifts, 1, 1);

	size_t toeplitzMatrixElementCount = elementsInInput * elementsInOutput;
	size_t filterElementCount = filterSize * filterSize;
	int toeplitzByteCount = toeplitzMatrixElementCount * sizeof(float);
	int filterByteCount = filterElementCount * sizeof(float);
	std::cout << "Toeplitz element count " << toeplitzMatrixElementCount << std::endl;
	std::cout << "Filter element count " << filterElementCount << std::endl;

	//Define pointers for deviceMemory locations
	float* d_Output;
	float* d_Filter;

	//Allocate memory
	hipMalloc((void**)&d_Output, toeplitzByteCount);
	hipMemset((void*)d_Output, 0, toeplitzByteCount);

	hipMalloc((void**)&d_Filter, filterByteCount);

	//Copy filter into global device memory m_InputMatrix -> d_Input
	cout << "Filter [0]" << filter[0] << endl;
	hipMemcpy(d_Filter, filter, filterByteCount, hipMemcpyHostToDevice);
	cout << "_convolutionInputWidth" << m_InputMatrixWidth << endl;
	ToeplitzKernel << <blockGrid, threads >> > (d_Filter, d_Output, elementsInInput, elementsInOutput, filterSize, filterSize, m_InputMatrixWidth, columnShifts);
	hipDeviceSynchronize();

	hipMemcpy(toeplitzMatrix, d_Output, toeplitzByteCount, hipMemcpyDeviceToHost);
}

void Convolution::InitilizeFilter()
{
	std::random_device rd{};
	std::mt19937 gen{ rd() };
	std::normal_distribution<> distribution{ 1.42,2 };
	filter = new float[filterSize * filterSize];

	for (int i = 0; i < filterSize * filterSize; ++i)
	{
		filter[i] = i + 1;//distribution(gen);
	}
}


void Convolution::UpdateModule()
{

}


