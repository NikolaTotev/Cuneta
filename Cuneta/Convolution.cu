#include "hip/hip_runtime.h"
#include <hip/hip_vector_types.h>

#ifndef __HIPCC__  
#define __HIPCC__
#endif

#ifndef  __CUDART_API_PER_THREAD_DEFAULT_STREAM
#define  __CUDART_API_PER_THREAD_DEFAULT_STREAM
#endif
#include <hip/hip_runtime_api.h>


#ifndef __CUDACC_RTC__ 
#define __CUDACC_RTC__

#endif

#include <algorithm>
#include <iostream>

#include ""
#include "Convolution.cuh"
#include <random>
#include <cmath>

__global__ void ConvolutionKernel(float* d_Input, float* d_Output, int _inputHeight, int _inputWidth, int _outputHeight, int _outputWidth)
{
	//These define indecies for output matrix;
	int outputRowIndex = blockIdx.x;
	int outputColumnIndex = threadIdx.x;

	//Starts from "top left" of current block of pixels being processed
	int inputRowIndex = blockIdx.x * 2;
	int inputColumnIndex = threadIdx.x * 2;


	int smallerInputSide = fminf(_inputHeight, _inputWidth);
	int smallerOutputSide = fminf(_outputHeight, _outputWidth);

	int outputArrayIndex = outputRowIndex * smallerOutputSide + outputColumnIndex;

	int inputArrayIndex = inputRowIndex * smallerInputSide + inputColumnIndex;
	int initialTopLeftRow = inputArrayIndex;

	float currentMax = d_Input[inputArrayIndex];
	float currentPixel = 555;
	int var = 0;

	for (int row = 0; row < 2; row++)
	{
		inputColumnIndex = threadIdx.x * 2;
		inputRowIndex += row;

		for (int col = 0; col < 2; col++)
		{
			inputColumnIndex += col;

			inputArrayIndex = inputRowIndex * smallerInputSide + inputColumnIndex;

			currentPixel = d_Input[inputArrayIndex];

			//NOTE the case currentPixel >= currentMax
			if (currentPixel > currentMax)
			{
				currentMax = currentPixel;
			}
		}

	}

	d_Output[outputArrayIndex] = currentMax;

};

__global__ void ToeplitzKernel(float* d_Filter, float* d_Output, int _toeplitzHeight, int _toeplitzWidth, int _filterHeight, int _filterWidth, int _convolutionInputWidth)
{

	int initialBlockOffset = blockIdx.x * _convolutionInputWidth;
	int largerSide = fmaxf(_toeplitzHeight, _toeplitzWidth);
	int currentRowStart = (blockIdx.x + threadIdx.x)*largerSide;
	int writeIndex = currentRowStart + initialBlockOffset + threadIdx.x;

	
	int counter = 1;

	for (int j = 0; j < _filterHeight * _filterWidth; ++j)
	{
		if (counter == 3)
		{
			counter = 1;
		}
		else
		{
			//d_Output[writeIndex] = d_Filter[j];
			counter++;
		}
		writeIndex++;
	}
};

Convolution::Convolution(float* _inputMatrix, int _inputHeight, int _inputWidth, int _filterSize)
{
	m_InputMatrix = _inputMatrix;
	m_InputMatrixHeight = _inputHeight;
	m_InputMatrixWidth = _inputWidth;

	m_OutputMatrixHeight = _inputHeight - 2;
	m_OutputMatrixWidth = _inputWidth - 2;

	filterSize = _filterSize;

	m_OutputMatrix = new float[m_OutputMatrixHeight * m_OutputMatrixWidth];
	InitilizeFilter();
	FilterToToeplitzMatrix();
}

void Convolution::ForwardPass()
{

}


void Convolution::BackwardPass()
{

}


void Convolution::Dialate(float* _input, float* _output)
{

}


void Convolution::FilterToToeplitzMatrix()
{
	int columnShifts = m_OutputMatrixWidth;
	int rowShifts = m_OutputMatrixHeight;
	int elementsInInput = m_InputMatrixHeight * m_InputMatrixWidth;
	int elementsInOutput = m_OutputMatrixHeight * m_OutputMatrixWidth;
	toeplitzMatrix = new float[elementsInInput * elementsInOutput];

	dim3 blockGrid(rowShifts, 1, 1);
	dim3 threads(columnShifts, 1, 1);

	size_t toeplitzMatrixElementCount = filterSize * filterSize;
	size_t filterElementCount = elementsInInput * elementsInOutput;
	int toeplitzByteCount = toeplitzMatrixElementCount * sizeof(float);
	int filterByteCount = filterElementCount * sizeof(float);
	std::cout << "Toeplitz element count " << toeplitzMatrixElementCount;
	std::cout << "Filter element count " << filterElementCount;

	//Define pointers for deviceMemory locations
	float* d_Output;
	float* d_Filter;

	//Allocate memory
	hipMalloc((void**)&d_Output, toeplitzByteCount);
	hipMemset((void*)d_Output, 0, toeplitzByteCount);

	hipMalloc((void**)&d_Filter, filterSize);

	//Copy filter into global device memory m_InputMatrix -> d_Input
	hipMemcpy(d_Filter, filter, filterByteCount, hipMemcpyHostToDevice);

	ToeplitzKernel << <blockGrid, threads >> > (d_Filter, d_Output, elementsInInput, elementsInOutput, filterSize, filterSize, m_InputMatrixWidth);
	hipDeviceSynchronize();

	hipMemcpy(toeplitzMatrix, d_Output, toeplitzByteCount, hipMemcpyDeviceToHost);
}

void Convolution::InitilizeFilter()
{
	std::random_device rd{};
	std::mt19937 gen{ rd() };
	std::normal_distribution<> distribution{ 1.42,2 };
	filter = new float[filterSize * filterSize];

	for (int i = 0; i < filterSize * filterSize; ++i)
	{
		filter[i] = distribution(gen);
	}
}


void Convolution::UpdateModule()
{

}


