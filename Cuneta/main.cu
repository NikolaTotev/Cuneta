#include "hip/hip_runtime.h"
﻿
#include <algorithm>
#include <cstdlib>
#include <iostream>

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "Test_Utils.cuh"

int main()
{
	int matrixHeight = 5;
	int matrixWidth = 4;
	int vectorizedMatrixSize = matrixWidth * matrixHeight;
	float* input = new float[vectorizedMatrixSize];
	float* output = new float[vectorizedMatrixSize];

	int max = 20;
	int min = -5;
	int range = max - min + 1;
	
	for (int i = 0; i < matrixHeight*matrixWidth; ++i)
	{
		input[i] = rand() % range + min;
		std::cout << input[i] << std::endl;
	}

	//TestReLU(input, output, matrixWidth, matrixHeight);

	//TestMaxPool(input, output, matrixWidth, matrixHeight);

	TestConvolution(input, matrixHeight, matrixWidth, 3);

	return 0;
}
