#include "hip/hip_runtime.h"
﻿
#include <algorithm>
#include <cstdlib>
#include <iostream>

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "Test_Utils.cuh"

int main()
{
	int matrixHeight = 10;
	int matrixWidth = 5;
	int vectorizedMatrixSize = matrixWidth * matrixHeight;
	float* input = new float[vectorizedMatrixSize];
	float* output = new float[vectorizedMatrixSize];

	int max = 20;
	int min = -5;
	int range = max - min + 1;
	
	for (int i = 0; i < matrixHeight*matrixWidth; ++i)
	{
		input[i] = rand() % range + min;
		//std::cout << input[i] << std::endl;
	}
	
	//TestReLU(input, output, matrixWidth, matrixHeight);

	//std::cout << input[0] << std::endl;
	//std::cout << input[1] << std::endl;
	//std::cout << input[matrixWidth] << std::endl;
	//std::cout << input[matrixWidth + 1] << std::endl;
	TestMaxPool(input, output, matrixWidth, matrixHeight, false);

	std::cout << input[0] << std::endl;
	std::cout << input[1] << std::endl;
	std::cout << input[2] << std::endl;
	std::cout << input[matrixWidth] << std::endl;
	std::cout << input[matrixWidth + 1] << std::endl;
	std::cout << input[matrixWidth + 2] << std::endl;

	TestConvolution(input, matrixHeight, matrixWidth, 3,true);

	return 0;
}
