#include "hip/hip_runtime.h"
﻿
#include <algorithm>
#include <cstdlib>
#include <iostream>

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "Test_Utils.cuh"
using namespace std;
int main()
{

	cout << "Cuneta is starting..." << endl;
	int matrixHeight = 6;
	int matrixWidth = 4;
	int vectorizedMatrixSize = matrixWidth * matrixHeight;
	float* input = new float[vectorizedMatrixSize];
	float* groundTruth = new float[vectorizedMatrixSize];
	float* output = new float[vectorizedMatrixSize];

	int max = 10;
	int min = 0;
	int range = max - min + 1;

	for (int i = 0; i < matrixHeight * matrixWidth; ++i)
	{
		input[i] = rand() % range + min;
		//std::cout << input[i] << std::endl;
	}

	max = 1;
	min = 0;
	range = max - min + 1;

	for (int i = 0; i < matrixHeight * matrixWidth; ++i)
	{
		groundTruth[i] = rand() % range + min;
		//std::cout << input[i] << std::endl;
	}

	//TestReLU(input, output, matrixWidth, matrixHeight);

	/*std::cout << input[0] << std::endl;
	std::cout << input[1] << std::endl;
	std::cout << input[matrixWidth] << std::endl;
	std::cout << input[matrixWidth + 1] << std::endl;
	TestMaxPool(input, output, matrixWidth, matrixHeight, false);*/

	/*std::cout << input[0] << std::endl;
	std::cout << input[1] << std::endl;
	std::cout << input[2] << std::endl;
	std::cout << input[matrixWidth] << std::endl;
	std::cout << input[matrixWidth + 1] << std::endl;
	std::cout << input[matrixWidth + 2] << std::endl;*/

	//TestConvolution(input, matrixHeight, matrixWidth, 3, true);

	//TestTransposeConvolution(input, matrixHeight, matrixWidth, 3, true);
	TestErrorCalcModule(input, groundTruth, matrixHeight, matrixWidth, true);

	return 0;
}
