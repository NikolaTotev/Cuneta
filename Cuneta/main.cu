#include "hip/hip_runtime.h"
﻿
#include <algorithm>
#include <cstdlib>

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "ReLU_Test.cuh"

int main()
{
	int matrixHeight = 20;
	int matrixWidth = 10;
	int vectorizedMatrixSize = matrixWidth * matrixHeight;
	float* input = new float[vectorizedMatrixSize];
	float* output = new float[vectorizedMatrixSize];

	int max = 20;
	int min = -20;
	int range = max - min + 1;
	
	for (int i = 0; i < matrixHeight*matrixWidth; ++i)
	{
		input[i] = rand() % range + min;
	}

	TestReLU(input, output, matrixWidth, matrixHeight);

	return 0;
}
