#include "hip/hip_runtime.h"
#include "ReLU.cuh"

#include <hip/hip_vector_types.h>

#ifndef __HIPCC__  
#define __HIPCC__
#endif

#ifndef  __CUDART_API_PER_THREAD_DEFAULT_STREAM
#define  __CUDART_API_PER_THREAD_DEFAULT_STREAM
#endif
#include <hip/hip_runtime_api.h>


#ifndef __CUDACC_RTC__ 
#define __CUDACC_RTC__

#endif

#include <algorithm>
#include <iostream>

#include ""



//Input and output will be in global memory. d_ shows in which memory the variables are stored.
__global__ void ReLUKernel(float* d_Input, float* d_Output, int matrixWidth)
{
	int rowIndex = blockIdx.x;
	int columnIndex = threadIdx.x;
	int arrayIndex = rowIndex * matrixWidth + columnIndex;
	float pixel = d_Input[arrayIndex];

	float ReLUResult = fmaxf(0, pixel);

	d_Output[arrayIndex] = ReLUResult;
}


__global__ void BackpropReLUKernel(float* d_BackpropInput, float* d_FwdInput, float* d_BackpropOutput, int matrixWidth)
{
	int rowIndex = blockIdx.x; ///OK
	int columnIndex = threadIdx.x;///OK
	int arrayIndex = rowIndex * matrixWidth + columnIndex; ///OK
	float fwdInputPixel = d_FwdInput[arrayIndex];///OK
	float backpropInputPixel = d_BackpropInput[arrayIndex];///OK

	float ReLUResult = 0; ///OK
	if (fwdInputPixel > 0)
	{
		ReLUResult = 1 * backpropInputPixel;
	}

	d_BackpropOutput[arrayIndex] = ReLUResult;///OK
}

ReLU::ReLU(int _numberOfInputs, int _numberOfOutputs, int _IOHeight, int _IOWidth)
{
	L_FORWARD_NumberOf_INPUTS = _numberOfInputs;
	L_FORWARD_NumerOf_OUTPUTS = _numberOfOutputs;

	L_BACKWARD_NumberOf_INPUTS = L_FORWARD_NumerOf_OUTPUTS;
	L_BACKWARD_NumberOf_OUTPUTS = L_FORWARD_NumberOf_INPUTS;

	L_FORWARD_InputLayer_HEIGHT = _IOHeight;
	L_FORWARD_InputLayer_WIDTH = _IOWidth;

	L_FORWARD_OutputLayer_HEIGHT = _IOHeight;
	L_FORWARD_OutputLayer_WIDTH = _IOWidth;

	L_BACKWARD_InputLayer_HEIGHT = L_FORWARD_OutputLayer_HEIGHT;
	L_BACKWARD_InputLayer_WIDTH = L_FORWARD_OutputLayer_WIDTH;

	L_BACKWARD_OutputLayer_HEIGHT = L_FORWARD_InputLayer_HEIGHT;
	L_BACKWARD_OutputLayer_WIDTH = L_FORWARD_InputLayer_WIDTH;

	L_FORWARD_Pass_INPUTS = new float* [L_FORWARD_NumberOf_INPUTS];
	L_FORWARD_Pass_OUTPUTS = new float* [L_FORWARD_NumerOf_OUTPUTS];

	L_BACKWARD_Pass_INPUTS = new float* [L_FORWARD_NumerOf_OUTPUTS];
	L_BACKWARD_Pass_OUTPUTS = new float* [L_FORWARD_NumberOf_INPUTS];
}


void ReLU::ForwardPass(float* forwardPassInput, int fwdPassHeight, int fwdPassWidth)
{
	int arrayLength = fwdPassHeight * fwdPassWidth;
	size_t inputSize = arrayLength * sizeof(float);

	m_InputMatrixHeight = fwdPassHeight;
	m_InputMatrixWidth = fwdPassWidth;

	m_OutputMatrixHeight = m_InputMatrixHeight;
	m_OutputMatrixWidth = m_InputMatrixWidth;

	m_InputMatrix = new float[arrayLength];
	m_OutputMatrix = new float[arrayLength];

	memcpy(m_InputMatrix, forwardPassInput, inputSize);

	size_t totalPixelCount = m_InputMatrixHeight * m_InputMatrixWidth;
	int byteCount = totalPixelCount * sizeof(float);
	std::cout << "Pixel count " << totalPixelCount;

	//Define pointers for deviceMemory locations
	float* d_Input;
	float* d_Output;

	//Allocate memory
	hipMalloc((void**)&d_Input, byteCount);
	hipMalloc((void**)&d_Output, byteCount);

	//Copy memory into global device memory m_InputMatrix -> d_Input
	hipMemcpy(d_Input, m_InputMatrix, byteCount, hipMemcpyHostToDevice);

	//Define block size and threads per block.
	dim3 blockGrid(m_InputMatrixHeight, 1, 1);
	dim3 threadGrid(m_InputMatrixWidth, 1, 1);

	ReLUKernel << <blockGrid, threadGrid >> > (d_Input, d_Output, m_InputMatrixWidth);
	hipDeviceSynchronize();

	//Copy back result into host memory d_Output -> m_OutputMatrix
	hipMemcpy(m_OutputMatrix, d_Output, byteCount, hipMemcpyDeviceToHost);

}

void ReLU::BackwardPass(float* backpropInput, int backPassHeight, int backPassWidth)
{

	int arrayLength = backPassHeight * backPassWidth; ///OK
	size_t inputSize = arrayLength * sizeof(float); ///OK

	m_BackpropInputMatrixHeight = backPassHeight; ///OK
	m_BackpropInputMatrixWidth = backPassWidth; ///OK

	m_BackpropOutputMatrixHeight = m_BackpropInputMatrixHeight;///OK
	m_BackpropOutputMatrixWidth = m_BackpropInputMatrixWidth;///OK

	m_BackPropInputMatrix = new float[arrayLength];///OK
	m_BackpropagationOutput = new float[arrayLength];///OK

	memcpy(m_BackPropInputMatrix, backpropInput, inputSize); ///OK

	size_t totalPixelCount = m_BackpropInputMatrixHeight * m_BackpropInputMatrixWidth; ///OK
	int byteCount = totalPixelCount * sizeof(float); ///OK

	//Define pointers for deviceMemory locations
	float* d_FwdInput; ///OK
	float* d_BackpropInput; ///OK
	float* d_Output; ///OK

	//Allocate memory
	hipMalloc((void**)&d_FwdInput, byteCount); ///OK
	hipMalloc((void**)&d_BackpropInput, byteCount);///OK
	hipMalloc((void**)&d_Output, byteCount);///OK

	//Copy memory into global device memory m_InputMatrix -> d_Input
	hipMemcpy(d_BackpropInput, m_BackPropInputMatrix, byteCount, hipMemcpyHostToDevice); ///OK
	hipMemcpy(d_FwdInput, m_InputMatrix, byteCount, hipMemcpyHostToDevice); ///OK

	//Define block size and threads per block.
	dim3 blockGrid(m_BackpropInputMatrixHeight, 1, 1); ///OK
	dim3 threadGrid(m_BackpropInputMatrixWidth, 1, 1);///OK

	BackpropReLUKernel << <blockGrid, threadGrid >> > (d_BackpropInput, d_FwdInput, d_Output, m_InputMatrixWidth); ///OK
	hipDeviceSynchronize();///OK

	//Copy back result into host memory d_Output -> m_OutputMatrix
	hipMemcpy(m_BackpropagationOutput, d_Output, byteCount, hipMemcpyDeviceToHost);///OK
}


void ReLU::LayerForwardPass(float** _inputs)
{
	int inputSize = L_FORWARD_InputLayer_HEIGHT * L_FORWARD_InputLayer_WIDTH;
	int outputSize = L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH;
	size_t inputByteCount = inputSize * sizeof(float);
	size_t outputByteCount = inputSize * sizeof(float);

	for (int inputNumber = 0; inputNumber < L_FORWARD_NumberOf_INPUTS; ++inputNumber)
	{
		L_FORWARD_Pass_INPUTS[inputNumber] = new float[inputSize];
		L_FORWARD_Pass_OUTPUTS[inputNumber] = new float[outputSize];

		memcpy(L_FORWARD_Pass_INPUTS[inputNumber], _inputs[inputNumber], inputSize);

		//Define pointers for deviceMemory locations
		float* d_Input;
		float* d_Output;

		//Allocate memory
		hipMalloc((void**)&d_Input, inputByteCount);
		hipMalloc((void**)&d_Output, inputByteCount);

		//Copy memory into global device memory m_InputMatrix -> d_Input
		hipMemcpy(d_Input, L_FORWARD_Pass_INPUTS[inputNumber], inputByteCount, hipMemcpyHostToDevice);

		//Define block size and threads per block.
		dim3 blockGrid(L_FORWARD_InputLayer_HEIGHT, 1, 1);
		dim3 threadGrid(L_FORWARD_InputLayer_WIDTH, 1, 1);

		ReLUKernel << <blockGrid, threadGrid >> > (d_Input, d_Output, L_FORWARD_InputLayer_WIDTH);
		hipDeviceSynchronize();

		//Copy back result into host memory d_Output -> m_OutputMatrix
		hipMemcpy(L_FORWARD_Pass_OUTPUTS[inputNumber], d_Output, outputByteCount, hipMemcpyDeviceToHost);
	}
}


void ReLU::LayerBackwardPass(float** _backpropInput)
{
	for (int inputNumber = 0; inputNumber < L_FORWARD_NumerOf_OUTPUTS; ++inputNumber)
	{
		int forwardInputSize = L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH;

		int backwardInputSize = L_BACKWARD_InputLayer_HEIGHT* L_BACKWARD_InputLayer_WIDTH;

		int backwardOutputSize = L_BACKWARD_OutputLayer_HEIGHT* L_BACKWARD_OutputLayer_WIDTH;

		size_t forwardInputByteCount = forwardInputSize * sizeof(float);

		size_t backwardInputByteCount = backwardInputSize * sizeof(float);

		size_t backwardOutputByteCount = backwardOutputSize * sizeof(float);


		L_BACKWARD_Pass_INPUTS[inputNumber] = new float[backwardOutputSize];
		L_BACKWARD_Pass_OUTPUTS[inputNumber] = new float[backwardInputSize];

		memcpy(L_BACKWARD_Pass_INPUTS[inputNumber], _backpropInput[inputNumber], backwardInputByteCount);

		//Define pointers for deviceMemory locations
		float* d_FwdInput; ///OK
		float* d_BackpropInput; ///OK
		float* d_BackwardOutput; ///OK

		//Allocate memory
		hipMalloc((void**)&d_FwdInput, forwardInputByteCount); ///OK
		hipMalloc((void**)&d_BackpropInput, backwardInputByteCount);///OK
		hipMalloc((void**)&d_BackwardOutput, backwardOutputByteCount);///OK

		//Copy memory into global device memory m_InputMatrix -> d_Input
		hipMemcpy(d_BackpropInput, L_BACKWARD_Pass_INPUTS[inputNumber], backwardInputByteCount, hipMemcpyHostToDevice); ///OK
		hipMemcpy(d_FwdInput, L_FORWARD_Pass_INPUTS[inputNumber], forwardInputByteCount, hipMemcpyHostToDevice); ///OK

		//Define block size and threads per block.
		dim3 blockGrid(m_BackpropInputMatrixHeight, 1, 1); ///OK
		dim3 threadGrid(m_BackpropInputMatrixWidth, 1, 1);///OK

		BackpropReLUKernel << <blockGrid, threadGrid >> > (d_BackpropInput, d_FwdInput, d_BackwardOutput, L_FORWARD_OutputLayer_WIDTH); ///OK
		hipDeviceSynchronize();///OK

		//Copy back result into host memory d_Output -> m_OutputMatrix
		hipMemcpy(L_BACKWARD_Pass_OUTPUTS[inputNumber], d_BackwardOutput, backwardOutputByteCount, hipMemcpyDeviceToHost);///OK
	}
}


void ReLU::UpdateModule()
{

}





