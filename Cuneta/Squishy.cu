#include "hip/hip_runtime.h"
#include <hip/hip_vector_types.h>

#ifndef __HIPCC__  
#define __HIPCC__
#endif

#ifndef  __CUDART_API_PER_THREAD_DEFAULT_STREAM
#define  __CUDART_API_PER_THREAD_DEFAULT_STREAM
#endif
#include <hip/hip_runtime_api.h>


#ifndef __CUDACC_RTC__ 
#define __CUDACC_RTC__

#endif

#include <algorithm>
#include <iostream>

#include ""
#include "Squishy.cuh"
#include <random>
#include <cmath>

#ifndef __HIPCC__  
#define __HIPCC__
#endif

#ifndef __CUDACC_RTC__ 
#define __CUDACC_RTC__
#endif

#include <hip/device_functions.h>
using namespace std;



__global__ void SquishyConvolutionKernel(float** _inputs, float** _filters, float** _outputs, float** _biases, int _numberOfInputs, int _outputWidth, int _inputWidth, int _filterSize)
{
	int inputSelectionIndex = threadIdx.x;
	int filterSelectionIndex = blockIdx.z * _numberOfInputs + threadIdx.x;
	//int biasSelectionIndex = blockIdx.z * _numberOfInputs + threadIdx.x;
	int outputSelectionIndex = blockIdx.z;

	float* selectedInput = _inputs[inputSelectionIndex];
	float* selectedFilter = _filters[filterSelectionIndex];
	//float* selectedBias = _biases[biasSelectionIndex];
	float* selectedOutput = _outputs[outputSelectionIndex];

	int inputStartReadRowIndex = blockIdx.x;
	int inputStartReadColumnIndex = blockIdx.y;

	int outputWriteRowIndex = blockIdx.x;
	int outputWriteColumnIndex = blockIdx.y;

	int inputArrayIndex = 0;

	int outputArrayIndex = outputWriteRowIndex * _outputWidth + outputWriteColumnIndex;
	inputArrayIndex = inputStartReadRowIndex * _inputWidth + inputStartReadColumnIndex;
	float result = 0;
	int filterIndex = 0;


	result += selectedInput[inputArrayIndex] * selectedFilter[filterIndex];

	//result += selectedBias[outputArrayIndex];
	atomicAdd(&selectedOutput[outputArrayIndex], result);
};

__global__ void SquishyLayerBackpropConvolutionKernel(float** _inputs, float** _filters, float** _outputs, int _numberOfOutputs, int _outputWidth, int _inputWidth, int _filterSize)
{
	int inputSelectionIndex = threadIdx.x;
	int filterSelectionIndex = blockIdx.z * _numberOfOutputs + threadIdx.x;
	int outputSelectionIndex = blockIdx.z;

	float* selectedInput = _inputs[inputSelectionIndex];
	float* selectedFilter = _filters[filterSelectionIndex];
	float* selectedOutput = _outputs[outputSelectionIndex];

	int inputStartReadRowIndex = blockIdx.x;
	int inputStartReadColumnIndex = blockIdx.y;

	int outputWriteRowIndex = blockIdx.x;
	int outputWriteColumnIndex = blockIdx.y;

	int inputArrayIndex = 0;

	int outputArrayIndex = outputWriteRowIndex * _outputWidth + outputWriteColumnIndex;
	inputArrayIndex = inputStartReadRowIndex * _inputWidth + inputStartReadColumnIndex;

	float result = 0;
	int filterIndex = 0;

	selectedOutput[outputArrayIndex] = selectedInput[inputArrayIndex] * selectedFilter[filterIndex];
}

__global__ void SquishyLayerFilterBackpropKernel(float** _inputs, float** _outputs, float** _filterEquivalents, int _inputsWidth, int _outputsWidth, int _filterEquivsHeight, int _filterEquivsWidth, int _numberOfInputs)
{
	int inputSelectionIndex = threadIdx.x;
	int filterSelectionIndex = blockIdx.z;
	int outputSelectionIndex = blockIdx.z * _numberOfInputs + threadIdx.x;

	float* selectedInput = _inputs[inputSelectionIndex];
	float* selectedFilter = _filterEquivalents[filterSelectionIndex];
	float* selectedOutput = _outputs[outputSelectionIndex];

	int inputStartReadRowIndex = blockIdx.x;
	int inputStartReadColumnIndex = blockIdx.y;

	int outputWriteRowIndex = blockIdx.x;
	int outputWriteColumnIndex = blockIdx.y;

	int inputArrayIndex = 0;

	int outputArrayIndex = outputWriteRowIndex * _outputsWidth + outputWriteColumnIndex;

	float result = 0;
	int filterIndex = 0;

	for (int row = 0; row < _filterEquivsHeight; row++)
	{
		inputStartReadColumnIndex = blockIdx.y;

		for (int col = 0; col < _filterEquivsWidth; col++)
		{
			inputArrayIndex = inputStartReadRowIndex * _inputsWidth + inputStartReadColumnIndex;

			result += selectedInput[inputArrayIndex] * selectedFilter[filterIndex];
			filterIndex++;
			inputStartReadColumnIndex += 1;
		}
		inputStartReadRowIndex += 1;
	}

	selectedOutput[outputArrayIndex] = result;
}

__global__ void SquishyLayerFilterFlipKernel(float** _inputFilters, float** _outputFilters, int _filterSize)
{
	float* filterToFlip = _inputFilters[blockIdx.x];
	int filterArraySize = _filterSize * _filterSize;
	float* flippedOutput = _outputFilters[blockIdx.x];
	int k = 0;

	//Loop from back and assign value to new array
	for (int i = filterArraySize - 1; i >= 0; ) {
		flippedOutput[k++] = filterToFlip[i--];
	}
}

__global__ void SquishyLayerPaddingKernel(float** __inputs, float** _outputs, int _unpaddedWidth, int _paddedWidth)
{
	int inputSelectionIndex = blockIdx.y;
	int outputSelectionIndex = blockIdx.y;

	float* selectedInput = __inputs[inputSelectionIndex];
	float* selectedOutput = _outputs[outputSelectionIndex];

	int rowWriteIndex = (blockIdx.x + 1) * 2 + threadIdx.x;
	int columnWriteIndex = 2;

	int inputRowReadIndex = (blockIdx.x * 2) + threadIdx.x;
	int inputColumnReadIndex = 0;

	int arrayPosition = rowWriteIndex * _paddedWidth + columnWriteIndex;
	int inputArrayPosition = inputRowReadIndex * _unpaddedWidth + inputColumnReadIndex;

	int var = 0;
	for (int i = 0; i < _unpaddedWidth; i++)
	{
		selectedOutput[arrayPosition] = selectedInput[inputArrayPosition];
		arrayPosition++;
		inputArrayPosition++;
	}
}


__global__ void SquishyFilterUpdateKernel(float** _currentFilters, float** _filterGradients, float** _VMatricies, float** _SMatricies, float** _V_CorrectedMatrices, float** _S_CorrectedMatricies, int _filterSize, float _HyperParam_Beta1, float _HyperParam_Beta2, float _HyperParam_T, float _HyperParam_alpha, float _HyperParam_Epsilon)
{
	float* selectedFilter = _currentFilters[blockIdx.x];
	float* selectedGradient = _filterGradients[blockIdx.x];
	float* selected_V_Matrix = _VMatricies[blockIdx.x];
	float* selected_S_Matrix = _SMatricies[blockIdx.x];
	float* selected_Corrected_V_Matrix = _V_CorrectedMatrices[blockIdx.x];
	float* selected_Corrected_S_Matrix = _S_CorrectedMatricies[blockIdx.x];

	for (int rowIndex = 0; rowIndex < _filterSize; ++rowIndex)
	{
		for (int columnIndex = 0; columnIndex < _filterSize; ++columnIndex)
		{
			int index = rowIndex * _filterSize + columnIndex;

			float filterBackpropValue = selectedGradient[index];
			float oldV = selected_V_Matrix[index];
			float oldS = selected_S_Matrix[index];

			float newV = _HyperParam_Beta1 * oldV + (1 - _HyperParam_Beta1) * filterBackpropValue;
			float newS = _HyperParam_Beta2 * oldS + (1 - _HyperParam_Beta2) * filterBackpropValue;

			float newVCorrected = newV / (1 - pow(_HyperParam_Beta1, _HyperParam_T));
			float newSCorrected = newS / (1 - pow(_HyperParam_Beta2, _HyperParam_T));

			selected_V_Matrix[index] = newV;
			selected_S_Matrix[index] = newS;

			selected_Corrected_V_Matrix[index] = newVCorrected;
			selected_Corrected_S_Matrix[index] = newSCorrected;

			float oldFilterValue = selectedFilter[index];
			float newF = oldFilterValue - _HyperParam_alpha * (newVCorrected / sqrt(newSCorrected + _HyperParam_Epsilon));

			selectedFilter[index] = _HyperParam_alpha * (newVCorrected / sqrt(newSCorrected + _HyperParam_Epsilon));
		}
	}
}

__global__ void SquishyBiasUpdateKernel(float** _currentBiasies, float** _biasGradients, float** _VMatricies, float** _SMatricies, float** _V_CorrectedMatrices, float** _S_CorrectedMatricies, int _height, int _width, int _HyperParam_Beta1, int _HyperParam_Beta2, int _HyperParam_T, int _HyperParam_alpha, int _HyperParam_Epsilon)
{
	float* selectedBias = _currentBiasies[blockIdx.x];
	float* selectedGradient = _biasGradients[0];
	float* selected_V_Matrix = _VMatricies[blockIdx.x];
	float* selected_S_Matrix = _SMatricies[blockIdx.x];
	float* selected_Corrected_V_Matrix = _V_CorrectedMatrices[blockIdx.x];
	float* selected_Corrected_S_Matrix = _S_CorrectedMatricies[blockIdx.x];

	for (int rowIndex = 0; rowIndex < _height; ++rowIndex)
	{
		for (int columnIndex = 0; columnIndex < _width; ++columnIndex)
		{
			int index = rowIndex * _width + columnIndex;

			float filterBackpropValue = selectedGradient[index];
			float oldV = selected_V_Matrix[index];
			float oldS = selected_S_Matrix[index];

			float newV = _HyperParam_Beta1 * oldV + (1 - _HyperParam_Beta1) * filterBackpropValue;
			float newS = _HyperParam_Beta2 * oldS + (1 - _HyperParam_Beta2) * filterBackpropValue;

			float newVCorrected = newV / (1 - pow(_HyperParam_Beta1, _HyperParam_T));
			float newSCorrected = newS / (1 - pow(_HyperParam_Beta2, _HyperParam_T));

			selected_V_Matrix[index] = newV;
			selected_S_Matrix[index] = newS;

			selected_Corrected_V_Matrix[index] = newVCorrected;
			selected_Corrected_S_Matrix[index] = newSCorrected;

			float oldFilterValue = selectedBias[index];
			float newF =  oldFilterValue - _HyperParam_alpha * (newVCorrected / sqrt(newSCorrected + _HyperParam_Epsilon));

			selectedBias[index] = newF;
		}
	}
}

Squishy::Squishy(){}


Squishy::Squishy(int _filterSize, int _paddingSize, int _numberOfInputs, int _numberOfOutputs, int _inputHeight, int _inputWidth, int _layerID, int _levelID)
{
	m_FilterSize = _filterSize;
	m_PaddingSize = _paddingSize;
	m_AdamOptimizer_VMatrix = new float[m_FilterSize * m_FilterSize];
	m_AdamOptimizer_SMatrix = new float[m_FilterSize * m_FilterSize];

	m_AdamOptimizer_Corrected_VMatrix = new float[m_FilterSize * m_FilterSize];
	m_AdamOptimizer_Corrected_SMatrix = new float[m_FilterSize * m_FilterSize];

	memset(m_AdamOptimizer_VMatrix, 0, m_FilterSize * m_FilterSize * sizeof(float));
	memset(m_AdamOptimizer_SMatrix, 0, m_FilterSize * m_FilterSize * sizeof(float));

	L_FORWARD_NumberOf_INPUTS = _numberOfInputs;
	L_FORWARD_NumberOf_OUTPUTS = _numberOfOutputs;

	L_BACKWARD_NumberOf_INPUTS = L_FORWARD_NumberOf_OUTPUTS;
	L_BACKWARD_NumberOf_OUTPUTS = L_FORWARD_NumberOf_INPUTS;

	L_FORWARD_InputLayer_HEIGHT = _inputHeight;
	L_FORWARD_InputLayer_WIDTH = _inputWidth;

	L_FORWARD_OutputLayer_HEIGHT = _inputHeight;
	L_FORWARD_OutputLayer_WIDTH = _inputWidth;

	L_BACKWARD_InputLayer_HEIGHT = L_FORWARD_OutputLayer_HEIGHT;
	L_BACKWARD_InputLayer_WIDTH = L_FORWARD_OutputLayer_WIDTH;

	L_BACKWARD_OutputLayer_HEIGHT = L_FORWARD_InputLayer_HEIGHT;
	L_BACKWARD_OutputLayer_WIDTH = L_FORWARD_InputLayer_WIDTH;

	L_FORWARD_Pass_INPUTS = new float* [L_FORWARD_NumberOf_INPUTS];
	L_FORWARD_Pass_OUTPUTS = new float* [L_FORWARD_NumberOf_OUTPUTS];

	L_BACKWARD_Pass_INPUTS = new float* [L_BACKWARD_NumberOf_INPUTS];
	L_BACKWARD_Pass_PADDED_INPUTS = new float* [L_BACKWARD_NumberOf_INPUTS];
	L_BACKWARD_Pass_OUTPUTS = new float* [L_BACKWARD_NumberOf_OUTPUTS];

	L_NumberOf_FILTERS = L_FORWARD_NumberOf_INPUTS * L_FORWARD_NumberOf_OUTPUTS;

	L_Filters = new float* [L_NumberOf_FILTERS];
	L_FLIPPED_Filters = new float* [L_NumberOf_FILTERS];
	L_Filter_BACKPROP_RESULTS = new float* [L_NumberOf_FILTERS];

	L_Baises = new float*[L_NumberOf_FILTERS];
	L_PrevBiases = new float*[L_NumberOf_FILTERS];


	L_AdamOptimizer_V_Matrix = new float* [L_NumberOf_FILTERS];
	L_AdamOptimizer_S_Matrix = new float* [L_NumberOf_FILTERS];
	L_AdamOptimizer_Corrected_V_Matrix = new float* [L_NumberOf_FILTERS];
	L_AdamOptimizer_Corrected_S_Matrix = new float* [L_NumberOf_FILTERS];

	L_BIAS_AdamOptimizer_V_Matrix = new float* [L_NumberOf_FILTERS];
	L_BIAS_AdamOptimizer_S_Matrix = new float* [L_NumberOf_FILTERS];
	L_BIAS_AdamOptimizer_Corrected_V_Matrix = new float* [L_NumberOf_FILTERS];
	L_BIAS_AdamOptimizer_Corrected_S_Matrix = new float* [L_NumberOf_FILTERS];

	for (int i = 0; i < L_NumberOf_FILTERS; ++i)
	{
		L_AdamOptimizer_V_Matrix[i] = new float[m_FilterSize * m_FilterSize];
		L_AdamOptimizer_S_Matrix[i] = new float[m_FilterSize * m_FilterSize];
		L_AdamOptimizer_Corrected_V_Matrix[i] = new float[m_FilterSize * m_FilterSize];
		L_AdamOptimizer_Corrected_S_Matrix[i] = new float[m_FilterSize * m_FilterSize];
		
		size_t byteCount = m_FilterSize * m_FilterSize * sizeof(float);
		memset(L_AdamOptimizer_V_Matrix[i], 0, byteCount);
		memset(L_AdamOptimizer_S_Matrix[i], 0, byteCount);
		memset(L_AdamOptimizer_Corrected_V_Matrix[i], 0, byteCount);
		memset(L_AdamOptimizer_Corrected_S_Matrix[i], 0, byteCount);

		L_Biases = new float* [L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH];
		L_BIAS_AdamOptimizer_V_Matrix[i] = new float[L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH];
		L_BIAS_AdamOptimizer_S_Matrix[i] = new float[L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH];
		L_BIAS_AdamOptimizer_Corrected_V_Matrix[i] = new float[L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH];
		L_BIAS_AdamOptimizer_Corrected_S_Matrix[i] = new float[L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH];

		size_t biasByteCount = L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH * sizeof(float);
		memset(L_BIAS_AdamOptimizer_V_Matrix[i], 0, biasByteCount);
		memset(L_BIAS_AdamOptimizer_S_Matrix[i], 0, biasByteCount);
		memset(L_BIAS_AdamOptimizer_Corrected_V_Matrix[i], 0, biasByteCount);
		memset(L_BIAS_AdamOptimizer_Corrected_S_Matrix[i], 0, biasByteCount);
	}

	levelID = _levelID;
	layerID = _layerID;

	LayerFilterInitialization();
	LayerBiasInitialization();
}



void Squishy::LayerFilterBackprop()
{
	int inputSize = L_FORWARD_InputLayer_HEIGHT * L_FORWARD_InputLayer_WIDTH;

	int filterEquivalentSize = L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH;

	int outputSize = m_FilterSize * m_FilterSize;

	size_t inputByteCount = inputSize * sizeof(float);
	size_t filterEquivalentByteCount = filterEquivalentSize * sizeof(float);
	size_t outputByteCount = outputSize * sizeof(float);


	for (int outputNumber = 0; outputNumber < L_NumberOf_FILTERS; ++outputNumber)
	{
		L_Filter_BACKPROP_RESULTS[outputNumber] = new float[outputSize];
	}


	int numberOfBlockx_X = m_FilterSize;
	int numberOfBlocks_Y = m_FilterSize;
	int numberOfBlocks_Z = L_BACKWARD_NumberOf_INPUTS;
	int numberOfThreadsPerBlock = L_FORWARD_NumberOf_INPUTS;

	// create intermediate host array for storage of device row-pointers

	// create top-level device array pointer
	float** h_Inputs = new float* [L_FORWARD_NumberOf_INPUTS];  //(float**)malloc(L_FORWARD_NumberOf_INPUTS * sizeof(float*));
	float** h_Filters = new float* [L_BACKWARD_NumberOf_INPUTS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_Outputs = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_FORWARD_NumberOf_OUTPUTS * sizeof(float*));



	float** d_InputPointerArray;
	hipMalloc((void**)&d_InputPointerArray, L_FORWARD_NumberOf_INPUTS * sizeof(float*));

	float** d_FilterPointerArray;
	hipMalloc((void**)&d_FilterPointerArray, L_BACKWARD_NumberOf_INPUTS * sizeof(float*));

	float** d_OutputPointerArray;
	hipMalloc((void**)&d_OutputPointerArray, L_NumberOf_FILTERS * sizeof(float*));


	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_FORWARD_NumberOf_INPUTS; i++) {
		hipMalloc(&h_Inputs[i], inputByteCount);
		hipMemcpy(h_Inputs[i], L_FORWARD_Pass_INPUTS[i], inputByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_BACKWARD_NumberOf_INPUTS; i++) {
		hipMalloc(&h_Filters[i], filterEquivalentByteCount);
		hipMemcpy(h_Filters[i], L_BACKWARD_Pass_INPUTS[i], filterEquivalentByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Outputs[i], outputByteCount);
		hipMemset(h_Outputs[i], 0, outputByteCount);
	}

	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_InputPointerArray, h_Inputs, L_FORWARD_NumberOf_INPUTS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_FilterPointerArray, h_Filters, L_BACKWARD_NumberOf_INPUTS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_OutputPointerArray, h_Outputs, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);

	dim3 blockGrid(numberOfBlockx_X, numberOfBlocks_Y, numberOfBlocks_Z); ///OK
	dim3 threads(numberOfThreadsPerBlock, 1, 1); ///OK

	SquishyLayerFilterBackpropKernel << <blockGrid, threads >> > (d_InputPointerArray, d_OutputPointerArray, d_FilterPointerArray, L_FORWARD_InputLayer_WIDTH, m_FilterSize, L_BACKWARD_InputLayer_HEIGHT, L_BACKWARD_InputLayer_WIDTH, L_FORWARD_NumberOf_INPUTS);
	hipDeviceSynchronize();

	float* temp = new float[outputSize];
	hipMemcpy(h_Outputs, d_OutputPointerArray, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {

		hipMemcpy(temp, h_Outputs[i], outputByteCount, hipMemcpyDeviceToHost);
		memcpy(L_Filter_BACKPROP_RESULTS[i], temp, outputByteCount);
		hipFree(h_Outputs[i]);
	}
	hipFree(d_OutputPointerArray);
	delete[] h_Outputs;

	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_FORWARD_NumberOf_INPUTS; i++) {
		hipFree(h_Inputs[i]);
	}
	hipFree(d_InputPointerArray);
	delete[] h_Inputs;

	for (size_t i = 0; i < L_BACKWARD_NumberOf_INPUTS; i++) {
		hipFree(h_Filters[i]);
	}
	hipFree(d_FilterPointerArray);
	delete[] h_Filters;
}


void Squishy::LayerForwardPass(float** _inputs)
{
	int inputSize = L_FORWARD_InputLayer_HEIGHT * L_FORWARD_InputLayer_WIDTH;

	int filterSize = m_FilterSize * m_FilterSize;

	int biasSize = L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH;

	int outputSize = L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH;

	size_t inputByteCount = inputSize * sizeof(float);
	size_t filterByteCount = filterSize * sizeof(float);
	size_t biasByteCount = biasSize * sizeof(float);
	size_t outputByteCount = outputSize * sizeof(float);

	for (int inputNumber = 0; inputNumber < L_FORWARD_NumberOf_INPUTS; ++inputNumber)
	{
		L_FORWARD_Pass_INPUTS[inputNumber] = new float[inputSize];
		memcpy(L_FORWARD_Pass_INPUTS[inputNumber], _inputs[inputNumber], inputByteCount);
	}

	for (int outputNumber = 0; outputNumber < L_FORWARD_NumberOf_OUTPUTS; ++outputNumber)
	{
		L_FORWARD_Pass_OUTPUTS[outputNumber] = new float[outputSize];
	}


	int numberOfBlockx_X = L_FORWARD_OutputLayer_HEIGHT;
	int numberOfBlocks_Y = L_FORWARD_OutputLayer_WIDTH;
	int numberOfBlocks_Z = L_FORWARD_NumberOf_OUTPUTS;
	int numberOfThreadsPerBlock = L_FORWARD_NumberOf_INPUTS;

	// create intermediate host array for storage of device row-pointers

	// create top-level device array pointer
	float** h_Inputs = new float* [L_FORWARD_NumberOf_INPUTS];  //(float**)malloc(L_FORWARD_NumberOf_INPUTS * sizeof(float*));
	float** h_Filters = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_Biases = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_Outputs = new float* [L_FORWARD_NumberOf_OUTPUTS]; //(float**)malloc(L_FORWARD_NumberOf_OUTPUTS * sizeof(float*));


	float** d_InputPointerArray;
	hipMalloc((void**)&d_InputPointerArray, L_FORWARD_NumberOf_INPUTS * sizeof(float*));

	float** d_BiasesPointerArray;
	hipMalloc((void**)&d_BiasesPointerArray, L_NumberOf_FILTERS * sizeof(float*));

	float** d_FilterPointerArray;
	hipMalloc((void**)&d_FilterPointerArray, L_NumberOf_FILTERS * sizeof(float*));

	float** d_OutputPointerArray;
	hipMalloc((void**)&d_OutputPointerArray, L_FORWARD_NumberOf_OUTPUTS * sizeof(float*));


	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_FORWARD_NumberOf_INPUTS; i++) {
		hipMalloc(&h_Inputs[i], inputByteCount);
		hipMemcpy(h_Inputs[i], L_FORWARD_Pass_INPUTS[i], inputByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Filters[i], filterByteCount);
		hipMemcpy(h_Filters[i], L_Filters[i], filterByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Biases[i], biasByteCount);
		hipMemcpy(h_Biases[i], L_Biases[i], biasByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_FORWARD_NumberOf_OUTPUTS; i++) {
		hipMalloc(&h_Outputs[i], outputByteCount);
		hipMemset(h_Outputs[i], 0, outputByteCount);
	}
	int something = L_NumberOf_FILTERS * sizeof(float*);
	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_InputPointerArray, h_Inputs, L_FORWARD_NumberOf_INPUTS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_FilterPointerArray, h_Filters, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_BiasesPointerArray, h_Biases, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_OutputPointerArray, h_Outputs, L_FORWARD_NumberOf_OUTPUTS * sizeof(float*), hipMemcpyHostToDevice);

	dim3 blockGrid(numberOfBlockx_X, numberOfBlocks_Y, numberOfBlocks_Z); ///OK
	dim3 threads(numberOfThreadsPerBlock, 1, 1); ///OK

	SquishyConvolutionKernel << <blockGrid, threads >> > (d_InputPointerArray, d_FilterPointerArray, d_OutputPointerArray, d_BiasesPointerArray, L_FORWARD_NumberOf_INPUTS, L_FORWARD_OutputLayer_WIDTH, L_FORWARD_InputLayer_WIDTH, m_FilterSize);
	hipDeviceSynchronize();
	float* temp = new float[outputSize];
	hipMemcpy(h_Outputs, d_OutputPointerArray, L_FORWARD_NumberOf_OUTPUTS * sizeof(float*), hipMemcpyDeviceToHost);
	for (size_t i = 0; i < L_FORWARD_NumberOf_OUTPUTS; i++) {

		hipMemcpy(temp, h_Outputs[i], outputByteCount, hipMemcpyDeviceToHost);
		memcpy(L_FORWARD_Pass_OUTPUTS[i], temp, outputByteCount);
		hipFree(h_Outputs[i]);
	}
	hipFree(d_OutputPointerArray);
	delete[] h_Outputs;

	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_FORWARD_NumberOf_INPUTS; i++) {
		hipFree(h_Inputs[i]);
	}
	hipFree(d_InputPointerArray);
	delete[] h_Inputs;

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipFree(h_Filters[i]);
	}
	hipFree(d_FilterPointerArray);
	delete[] h_Filters;
}


void Squishy::LayerBackwardPass(float** _backpropInput)
{
	int inputSize = L_BACKWARD_InputLayer_HEIGHT * L_BACKWARD_InputLayer_WIDTH;


	int filterSize = m_FilterSize * m_FilterSize;

	int outputSize = L_BACKWARD_OutputLayer_HEIGHT * L_BACKWARD_OutputLayer_WIDTH;

	size_t inputByteCount = inputSize * sizeof(float);


	size_t filterByteCount = filterSize * sizeof(float);
	size_t outputByteCount = outputSize * sizeof(float);

	for (int inputNumber = 0; inputNumber < L_BACKWARD_NumberOf_INPUTS; ++inputNumber)
	{
		L_BACKWARD_Pass_INPUTS[inputNumber] = new float[inputSize];
		memcpy(L_BACKWARD_Pass_INPUTS[inputNumber], _backpropInput[inputNumber], inputByteCount);
	}

	for (int outputNumber = 0; outputNumber < L_BACKWARD_NumberOf_OUTPUTS; ++outputNumber)
	{
		L_BACKWARD_Pass_OUTPUTS[outputNumber] = new float[outputSize];
	}

	LayerFlipFilter();

	int numberOfBlockx_X = L_BACKWARD_OutputLayer_HEIGHT;//L_FORWARD_OutputLayer_HEIGHT;
	int numberOfBlocks_Y = L_BACKWARD_OutputLayer_WIDTH;//L_FORWARD_OutputLayer_WIDTH;
	int numberOfBlocks_Z = L_BACKWARD_NumberOf_OUTPUTS;//L_FORWARD_NumberOf_OUTPUTS;
	int numberOfThreadsPerBlock = L_BACKWARD_NumberOf_INPUTS;//L_FORWARD_NumberOf_INPUTS;

	float** h_Inputs = new float* [L_BACKWARD_NumberOf_INPUTS];  //(float**)malloc(L_FORWARD_NumberOf_INPUTS * sizeof(float*));
	float** h_Filters = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_Outputs = new float* [L_BACKWARD_NumberOf_OUTPUTS]; //(float**)malloc(L_FORWARD_NumberOf_OUTPUTS * sizeof(float*));


	float** d_InputPointerArray;
	hipMalloc((void**)&d_InputPointerArray, L_BACKWARD_NumberOf_INPUTS * sizeof(float*));

	float** d_FilterPointerArray;
	hipMalloc((void**)&d_FilterPointerArray, L_NumberOf_FILTERS * sizeof(float*));

	float** d_OutputPointerArray;
	hipMalloc((void**)&d_OutputPointerArray, L_BACKWARD_NumberOf_OUTPUTS * sizeof(float*));

	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_BACKWARD_NumberOf_INPUTS; i++) {
		hipMalloc(&h_Inputs[i], inputByteCount);
		hipMemcpy(h_Inputs[i], L_BACKWARD_Pass_INPUTS[i], inputByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Filters[i], filterByteCount);
		hipMemcpy(h_Filters[i], L_FLIPPED_Filters[i], filterByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_BACKWARD_NumberOf_OUTPUTS; i++) {
		hipMalloc(&h_Outputs[i], outputByteCount);
		hipMemset(h_Outputs[i], 0, outputByteCount);
	}

	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_InputPointerArray, h_Inputs, L_BACKWARD_NumberOf_INPUTS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_FilterPointerArray, h_Filters, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_OutputPointerArray, h_Outputs, L_BACKWARD_NumberOf_OUTPUTS * sizeof(float*), hipMemcpyHostToDevice);

	dim3 blockGrid(numberOfBlockx_X, numberOfBlocks_Y, numberOfBlocks_Z); ///OK
	dim3 threads(numberOfThreadsPerBlock, 1, 1); ///OK

	SquishyLayerBackpropConvolutionKernel << <blockGrid, threads >> > (d_InputPointerArray, d_FilterPointerArray, d_OutputPointerArray, L_BACKWARD_NumberOf_INPUTS, L_BACKWARD_OutputLayer_WIDTH, L_BACKWARD_InputLayer_WIDTH, m_FilterSize);
	hipDeviceSynchronize();
	float* temp = new float[outputSize];
	hipMemcpy(h_Outputs, d_OutputPointerArray, L_BACKWARD_NumberOf_OUTPUTS * sizeof(float*), hipMemcpyDeviceToHost);
	for (size_t i = 0; i < L_BACKWARD_NumberOf_OUTPUTS; i++) {

		hipMemcpy(temp, h_Outputs[i], outputByteCount, hipMemcpyDeviceToHost);
		memcpy(L_BACKWARD_Pass_OUTPUTS[i], temp, outputByteCount);
		hipFree(h_Outputs[i]);
	}
	hipFree(d_OutputPointerArray);
	delete[] h_Outputs;

	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_BACKWARD_NumberOf_INPUTS; i++) {
		hipFree(h_Inputs[i]);
	}
	hipFree(d_InputPointerArray);
	delete[] h_Inputs;

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipFree(h_Filters[i]);
	}
	hipFree(d_FilterPointerArray);
	delete[] h_Filters;

	LayerFilterBackprop();
	LayerUpdate();
	LayerBiasUpdate();
}


void Squishy::LayerPadBackpropInput()
{
	L_BACKWARD_InputLayer_PADDED_HEIGHT = L_BACKWARD_InputLayer_HEIGHT + 2 * m_PaddingSize;
	L_BACKWARD_InputLayer_PADDED_WIDTH = L_BACKWARD_InputLayer_WIDTH + 2 * m_PaddingSize;

	int inputSize = L_BACKWARD_InputLayer_HEIGHT * L_BACKWARD_InputLayer_WIDTH;

	int outputSize = L_BACKWARD_InputLayer_PADDED_HEIGHT * L_BACKWARD_InputLayer_PADDED_WIDTH;

	size_t inputByteCount = inputSize * sizeof(float);
	size_t outputByteCount = outputSize * sizeof(float);


	for (int inputNumber = 0; inputNumber < L_BACKWARD_NumberOf_INPUTS; ++inputNumber)
	{
		L_BACKWARD_Pass_PADDED_INPUTS[inputNumber] = new float[outputSize];
	}

	int numberOfBlockx_X = L_BACKWARD_InputLayer_HEIGHT / 2;
	int numberOfBlocks_Y = L_BACKWARD_NumberOf_INPUTS;
	int numberOfThreadsPerBlock = 2;

	// create intermediate host array for storage of device row-pointers

	// create top-level device array pointer
	float** h_Inputs = new float* [L_BACKWARD_NumberOf_INPUTS];  //(float**)malloc(L_FORWARD_NumberOf_INPUTS * sizeof(float*));
	float** h_Outputs = new float* [L_BACKWARD_NumberOf_INPUTS]; //(float**)malloc(L_FORWARD_NumberOf_OUTPUTS * sizeof(float*));


	float** d_InputPointerArray;
	hipMalloc((void**)&d_InputPointerArray, L_BACKWARD_NumberOf_INPUTS * sizeof(float*));

	float** d_OutputPointerArray;
	hipMalloc((void**)&d_OutputPointerArray, L_BACKWARD_NumberOf_INPUTS * sizeof(float*));


	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_BACKWARD_NumberOf_INPUTS; i++) {
		hipMalloc(&h_Inputs[i], inputByteCount);
		hipMemcpy(h_Inputs[i], L_BACKWARD_Pass_INPUTS[i], inputByteCount, hipMemcpyHostToDevice);
	}


	for (size_t i = 0; i < L_BACKWARD_NumberOf_INPUTS; i++) {
		hipMalloc(&h_Outputs[i], outputByteCount);
		hipMemset(&h_Outputs[i], 0, outputByteCount);
	}

	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_InputPointerArray, h_Inputs, L_BACKWARD_NumberOf_INPUTS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_OutputPointerArray, h_Outputs, L_BACKWARD_NumberOf_INPUTS * sizeof(float*), hipMemcpyHostToDevice);

	dim3 blockGrid(numberOfBlockx_X, numberOfBlocks_Y, 1); ///OK
	dim3 threads(numberOfThreadsPerBlock, 1, 1); ///OK

	SquishyLayerPaddingKernel << <blockGrid, threads >> > (d_InputPointerArray, d_OutputPointerArray, L_BACKWARD_InputLayer_WIDTH, L_BACKWARD_InputLayer_PADDED_WIDTH);
	hipDeviceSynchronize();

	float* temp = new float[outputSize];
	hipMemcpy(h_Outputs, d_OutputPointerArray, L_BACKWARD_NumberOf_INPUTS * sizeof(float*), hipMemcpyDeviceToHost);
	for (size_t i = 0; i < L_BACKWARD_NumberOf_INPUTS; i++) {

		hipMemcpy(temp, h_Outputs[i], outputByteCount, hipMemcpyDeviceToHost);
		memcpy(L_BACKWARD_Pass_PADDED_INPUTS[i], temp, outputByteCount);
		hipFree(h_Outputs[i]);
	}
	hipFree(d_OutputPointerArray);
	delete[] h_Outputs;

	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_BACKWARD_NumberOf_INPUTS; i++) {
		hipFree(h_Inputs[i]);
	}
	hipFree(d_InputPointerArray);
	delete h_Inputs;
}


void Squishy::LayerUpdate()
{
	int filterSize = m_FilterSize * m_FilterSize;

	size_t filterByteCount = filterSize * sizeof(float);


	int numberOfBlocks_X = L_NumberOf_FILTERS;
	int numberOfThreadsPerBlock = 1;

	// create intermediate host array for storage of device row-pointers

	// create top-level device array pointer
	float** h_Filters = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_FilterGradients = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_V_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_S_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_V_CORRECTED_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_S_CORRECTED_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));


	float** d_FilterPointers;
	hipMalloc((void**)&d_FilterPointers, L_NumberOf_FILTERS * sizeof(float*));

	float** d_FilterGradientPointers;
	hipMalloc((void**)&d_FilterGradientPointers, L_NumberOf_FILTERS * sizeof(float*));

	float** d_V_Matricies;
	hipMalloc((void**)&d_V_Matricies, L_NumberOf_FILTERS * sizeof(float*));

	float** d_S_Matricies;
	hipMalloc((void**)&d_S_Matricies, L_NumberOf_FILTERS * sizeof(float*));

	float** d_V_CORRECTED_Matricies;
	hipMalloc((void**)&d_V_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*));

	float** d_S_CORRECTED_Matricies;
	hipMalloc((void**)&d_S_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*));



	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Filters[i], filterByteCount);
		hipMemcpy(h_Filters[i], L_Filters[i], filterByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_FilterGradients[i], filterByteCount);
		hipMemcpy(h_FilterGradients[i], L_Filter_BACKPROP_RESULTS[i], filterByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_V_Matricies[i], filterByteCount);
		hipMemcpy(h_V_Matricies[i], L_AdamOptimizer_V_Matrix[i], filterByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_S_Matricies[i], filterByteCount);
		hipMemcpy(h_S_Matricies[i], L_AdamOptimizer_S_Matrix[i], filterByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_V_CORRECTED_Matricies[i], filterByteCount);
		hipMemcpy(h_V_CORRECTED_Matricies[i], L_AdamOptimizer_Corrected_V_Matrix[i], filterByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_S_CORRECTED_Matricies[i], filterByteCount);
		hipMemcpy(h_S_CORRECTED_Matricies[i], L_AdamOptimizer_Corrected_S_Matrix[i], filterByteCount, hipMemcpyHostToDevice);
	}


	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_FilterPointers, h_Filters, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_FilterGradientPointers, h_FilterGradients, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_V_Matricies, h_V_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_S_Matricies, h_S_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_V_CORRECTED_Matricies, h_V_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_S_CORRECTED_Matricies, h_S_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);


	dim3 blockGrid(numberOfBlocks_X, 1, 1); ///OK
	dim3 threads(numberOfThreadsPerBlock, 1, 1); ///OK

	SquishyFilterUpdateKernel << <blockGrid, threads >> > (d_FilterPointers, d_FilterGradientPointers, d_V_Matricies, d_S_Matricies, d_V_CORRECTED_Matricies, d_S_CORRECTED_Matricies, m_FilterSize, m_HyperParam_Beta1, m_HyperParam_Beta2, m_HyperParam_T, m_HyperParam_alpha, m_HyperParam_Epsilon);
	hipDeviceSynchronize();

	float* temp = new float[filterByteCount];

	hipMemcpy(h_Filters, d_FilterPointers, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_FilterGradients, d_FilterGradientPointers, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_V_Matricies, d_V_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_S_Matricies, d_S_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_V_CORRECTED_Matricies, d_V_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_S_CORRECTED_Matricies, d_S_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {

		hipMemcpy(temp, h_Filters[i], filterByteCount, hipMemcpyDeviceToHost);
		memcpy(L_Filters[i], temp, filterByteCount);
		hipFree(h_Filters[i]);

		hipMemcpy(temp, h_V_Matricies[i], filterByteCount, hipMemcpyDeviceToHost);
		memcpy(L_AdamOptimizer_V_Matrix[i], temp, filterByteCount);
		hipFree(h_V_Matricies[i]);

		hipMemcpy(temp, h_S_Matricies[i], filterByteCount, hipMemcpyDeviceToHost);
		memcpy(L_AdamOptimizer_S_Matrix[i], temp, filterByteCount);
		hipFree(h_S_Matricies[i]);

		hipMemcpy(temp, h_V_CORRECTED_Matricies[i], filterByteCount, hipMemcpyDeviceToHost);
		memcpy(L_AdamOptimizer_Corrected_V_Matrix[i], temp, filterByteCount);
		hipFree(h_V_CORRECTED_Matricies[i]);

		hipMemcpy(temp, h_S_CORRECTED_Matricies[i], filterByteCount, hipMemcpyDeviceToHost);
		memcpy(L_AdamOptimizer_Corrected_S_Matrix[i], temp, filterByteCount);
		hipFree(h_S_CORRECTED_Matricies[i]);
	}

	hipFree(d_FilterPointers);
	hipFree(d_FilterGradientPointers);
	hipFree(d_V_Matricies);
	hipFree(d_S_Matricies);
	hipFree(d_V_CORRECTED_Matricies);
	hipFree(d_S_CORRECTED_Matricies);

	delete[] h_Filters;
	delete[] h_FilterGradients;
	delete[] h_V_Matricies;
	delete[] h_S_Matricies;
	delete[] h_V_CORRECTED_Matricies;
	delete[] h_S_CORRECTED_Matricies;
}

void Squishy::LayerBiasUpdate()
{
	int biasMatrixSize = L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH;

	size_t biasMatrixByteCount = biasMatrixSize * sizeof(float);


	int numberOfBlocks_X = L_NumberOf_FILTERS;
	int numberOfThreadsPerBlock = 1;

	// create intermediate host array for storage of device row-pointers

	// create top-level device array pointer
	float** h_Biases = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_BiasGradients = new float* [L_BACKWARD_NumberOf_INPUTS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_V_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_S_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_V_CORRECTED_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_S_CORRECTED_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));


	float** d_BiasPointers;
	hipMalloc((void**)&d_BiasPointers, L_NumberOf_FILTERS * sizeof(float*));

	float** d_BiasGradientPointers;
	hipMalloc((void**)&d_BiasGradientPointers, L_BACKWARD_NumberOf_INPUTS * sizeof(float*));

	float** d_V_Matricies;
	hipMalloc((void**)&d_V_Matricies, L_NumberOf_FILTERS * sizeof(float*));

	float** d_S_Matricies;
	hipMalloc((void**)&d_S_Matricies, L_NumberOf_FILTERS * sizeof(float*));

	float** d_V_CORRECTED_Matricies;
	hipMalloc((void**)&d_V_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*));

	float** d_S_CORRECTED_Matricies;
	hipMalloc((void**)&d_S_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*));


	for (int i = 0; i < L_BACKWARD_NumberOf_INPUTS; ++i)
	{
		hipMalloc(&h_BiasGradients[i], biasMatrixByteCount);
		hipMemcpy(h_BiasGradients[i], L_BACKWARD_Pass_INPUTS[i], biasMatrixByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Biases[i], biasMatrixByteCount);
		hipMemcpy(h_Biases[i], L_Biases[i], biasMatrixByteCount, hipMemcpyHostToDevice);

		L_PrevBiases[i] = new float[biasMatrixSize];
		memcpy(L_PrevBiases[i], L_Biases[i], biasMatrixByteCount);

		hipMalloc(&h_V_Matricies[i], biasMatrixByteCount);
		hipMemcpy(h_V_Matricies[i], L_BIAS_AdamOptimizer_V_Matrix[i], biasMatrixByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_S_Matricies[i], biasMatrixByteCount);
		hipMemcpy(h_S_Matricies[i], L_BIAS_AdamOptimizer_S_Matrix[i], biasMatrixByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_V_CORRECTED_Matricies[i], biasMatrixByteCount);
		hipMemcpy(h_V_CORRECTED_Matricies[i], L_BIAS_AdamOptimizer_Corrected_V_Matrix[i], biasMatrixByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_S_CORRECTED_Matricies[i], biasMatrixByteCount);
		hipMemcpy(h_S_CORRECTED_Matricies[i], L_BIAS_AdamOptimizer_Corrected_S_Matrix[i], biasMatrixByteCount, hipMemcpyHostToDevice);
	}


	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_BiasPointers, h_Biases, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_BiasGradientPointers, h_BiasGradients, L_BACKWARD_NumberOf_INPUTS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_V_Matricies, h_V_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_S_Matricies, h_S_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_V_CORRECTED_Matricies, h_V_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_S_CORRECTED_Matricies, h_S_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);


	dim3 blockGrid(numberOfBlocks_X, 1, 1); ///OK
	dim3 threads(numberOfThreadsPerBlock, 1, 1); ///OK

	SquishyBiasUpdateKernel << <blockGrid, threads >> > (d_BiasPointers, d_BiasGradientPointers, d_V_Matricies, d_S_Matricies, d_V_CORRECTED_Matricies, d_S_CORRECTED_Matricies, L_BACKWARD_InputLayer_HEIGHT, L_BACKWARD_InputLayer_WIDTH, m_HyperParam_Beta1, m_HyperParam_Beta2, m_HyperParam_T, m_HyperParam_alpha, m_HyperParam_Epsilon);
	hipDeviceSynchronize();

	float* temp = new float[biasMatrixByteCount];

	hipMemcpy(h_Biases, d_BiasPointers, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_BiasGradients, d_BiasGradientPointers, L_BACKWARD_NumberOf_INPUTS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_V_Matricies, d_V_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_S_Matricies, d_S_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_V_CORRECTED_Matricies, d_V_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_S_CORRECTED_Matricies, d_S_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {

		hipMemcpy(temp, h_Biases[i], biasMatrixByteCount, hipMemcpyDeviceToHost);
		memcpy(L_Biases[i], temp, biasMatrixByteCount);
		hipFree(h_Biases[i]);

		hipMemcpy(temp, h_V_Matricies[i], biasMatrixByteCount, hipMemcpyDeviceToHost);
		memcpy(L_BIAS_AdamOptimizer_V_Matrix[i], temp, biasMatrixByteCount);
		hipFree(h_V_Matricies[i]);

		hipMemcpy(temp, h_S_Matricies[i], biasMatrixByteCount, hipMemcpyDeviceToHost);
		memcpy(L_BIAS_AdamOptimizer_S_Matrix[i], temp, biasMatrixByteCount);
		hipFree(h_S_Matricies[i]);

		hipMemcpy(temp, h_V_CORRECTED_Matricies[i], biasMatrixByteCount, hipMemcpyDeviceToHost);
		memcpy(L_BIAS_AdamOptimizer_Corrected_V_Matrix[i], temp, biasMatrixByteCount);
		hipFree(h_V_CORRECTED_Matricies[i]);

		hipMemcpy(temp, h_S_CORRECTED_Matricies[i], biasMatrixByteCount, hipMemcpyDeviceToHost);
		memcpy(L_BIAS_AdamOptimizer_Corrected_S_Matrix[i], temp, biasMatrixByteCount);
		hipFree(h_S_CORRECTED_Matricies[i]);
	}

	hipFree(d_BiasPointers);
	hipFree(d_BiasGradientPointers);
	hipFree(d_V_Matricies);
	hipFree(d_S_Matricies);
	hipFree(d_V_CORRECTED_Matricies);
	hipFree(d_S_CORRECTED_Matricies);

	delete[] h_Biases;
	delete[] h_BiasGradients;
	delete[] h_V_Matricies;
	delete[] h_S_Matricies;
	delete[] h_V_CORRECTED_Matricies;
	delete[] h_S_CORRECTED_Matricies;
}

void Squishy::LayerFilterInitialization()
{
	std::random_device rd{};
	std::mt19937 gen{ rd() };
	std::normal_distribution<> distribution{ 0,1 };

	int filterElementCount = m_FilterSize * m_FilterSize;

	for (int filterNumber = 0; filterNumber < L_NumberOf_FILTERS; ++filterNumber)
	{
		L_Filters[filterNumber] = new float[filterElementCount];

		for (int i = 0; i < filterElementCount; ++i)
		{
			L_Filters[filterNumber][i] = i + 1; //distribution(gen);
		}
	}
}

void Squishy::LayerBiasInitialization()
{
	std::random_device rd{};
	std::mt19937 gen{ rd() };
	std::normal_distribution<> distribution{ 0,1 };

	int biasElementCount = L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH;

	for (int biasNumber = 0; biasNumber < L_NumberOf_FILTERS; ++biasNumber)
	{
		L_Biases[biasNumber] = new float[biasElementCount];

		for (int i = 0; i < biasElementCount; ++i)
		{
			L_Biases[biasNumber][i] = 2; //distribution(gen);
		}
	}

}


void Squishy::LayerFlipFilter()
{
	int inputSize = m_FilterSize * m_FilterSize;

	int outputSize = m_FilterSize * m_FilterSize;

	size_t inputByteCount = inputSize * sizeof(float);
	size_t outputByteCount = outputSize * sizeof(float);

	for (int filterNumber = 0; filterNumber < L_NumberOf_FILTERS; ++filterNumber)
	{
		L_FLIPPED_Filters[filterNumber] = new float[inputSize];
	}

	int numberOfBlockx_X = L_NumberOf_FILTERS;

	// create intermediate host array for storage of device row-pointers

	// create top-level device array pointer
	float** h_Inputs = new float* [L_NumberOf_FILTERS];

	float** h_Outputs = new float* [L_NumberOf_FILTERS];

	float** d_InputPointerArray;
	hipMalloc((void**)&d_InputPointerArray, L_NumberOf_FILTERS * sizeof(float*));


	float** d_OutputPointerArray;
	hipMalloc((void**)&d_OutputPointerArray, L_NumberOf_FILTERS * sizeof(float*));


	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Inputs[i], inputByteCount);
		hipMemcpy(h_Inputs[i], L_Filters[i], inputByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Outputs[i], outputByteCount);
	}

	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_InputPointerArray, h_Inputs, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_OutputPointerArray, h_Outputs, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);

	dim3 blockGrid(numberOfBlockx_X, 1, 1); ///OK
	dim3 threads(1, 1, 1); ///OK

	SquishyLayerFilterFlipKernel << <blockGrid, threads >> > (d_InputPointerArray, d_OutputPointerArray, m_FilterSize);
	hipDeviceSynchronize();

	float* temp = new float[outputSize];

	hipMemcpy(h_Outputs, d_OutputPointerArray, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {

		hipMemcpy(temp, h_Outputs[i], outputByteCount, hipMemcpyDeviceToHost);
		memcpy(L_FLIPPED_Filters[i], temp, outputByteCount);
		hipFree(h_Outputs[i]);
	}
	hipFree(d_OutputPointerArray);
	delete[] h_Outputs;

	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipFree(h_Inputs[i]);
	}
	hipFree(d_InputPointerArray);
	delete[] h_Inputs;

	delete[] temp;
}



void Squishy::SetHyperParams(float _beta1, float _beta2, float _eps, int _t, float _alpha)
{
	m_HyperParam_Beta1 = _beta1;
	m_HyperParam_Beta2 = _beta2;
	m_HyperParam_Epsilon = _eps;
	m_HyperParam_T = _t;
	m_HyperParam_alpha = _alpha;
}

void Squishy::PrintLayerParams()
{
	cout << "====================================================" << endl;
	cout << "====== Squishy Convolution Layer Parameters ======" << endl;
	cout << "====================================================" << endl;
	cout << "Squishy: Layer " << layerID << " " << "Level " << levelID << endl;

	cout << endl;

	cout << "-- Forward Dimensions --" << endl;
	cout << "Forward Input Height: " << L_FORWARD_InputLayer_HEIGHT << " || Forward Output Height: " << L_FORWARD_OutputLayer_HEIGHT << endl;
	cout << "Forward Input Width: " << L_FORWARD_InputLayer_WIDTH << " || Forward Output Width: " << L_FORWARD_OutputLayer_WIDTH << endl;

	cout << endl;

	cout << "-- Backward Dimensions --" << endl;
	cout << "Backward Input Height: " << L_BACKWARD_InputLayer_HEIGHT << " || Forward Output Height: " << L_BACKWARD_OutputLayer_HEIGHT << endl;
	cout << "Backward Input Width: " << L_BACKWARD_InputLayer_WIDTH << " || Forward Output Width: " << L_BACKWARD_OutputLayer_WIDTH << endl;

	cout << endl;

	cout << "-- Feature map count --" << endl;
	cout << "Forward Input Count: " << L_FORWARD_NumberOf_INPUTS << " || Backward Input Count: " << L_BACKWARD_NumberOf_INPUTS << endl;
	cout << "Forward Output Count: " << L_FORWARD_NumberOf_OUTPUTS << " || Backward Output Count: " << L_BACKWARD_NumberOf_OUTPUTS << endl;

	cout << "===================================" << endl;
}

void Squishy::DebugPrintAll()
{
	int newLineCounter = 1;

	cout << "=================================================" << endl;
	cout << "============ Squishy Debug Print All ============" << endl;
	cout << "=================================================" << endl;

	cout << "Squishy: " << endl;
	cout << "Layer ID: " << layerID << endl;
	cout << "Level ID: " << levelID << endl;
	cout << "Hyper parameters: " << endl;
	cout << "Beta 1: " << m_HyperParam_Beta1 << endl;
	cout << "Beta 2: " << m_HyperParam_Beta2 << endl;
	cout << "Epsilon: " << m_HyperParam_Epsilon << endl;
	cout << "Alpha: " << m_HyperParam_alpha << endl;
	cout << "T: " << m_HyperParam_T << endl;


	cout << ">>>> Forward Inputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_FORWARD_NumberOf_INPUTS; ++inputIndex)
	{
		cout << "--- Element " << inputIndex + 1 << "---" << endl;
		for (int elementIndex = 0; elementIndex < L_FORWARD_InputLayer_HEIGHT * L_FORWARD_InputLayer_WIDTH; ++elementIndex)
		{
			cout << L_FORWARD_Pass_INPUTS[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == L_FORWARD_InputLayer_WIDTH + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Normal Filter Inputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_Filters[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}


	cout << ">>>> Flipped Filter Inputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_FLIPPED_Filters[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}


	cout << ">>>> Forward Outputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_FORWARD_NumberOf_OUTPUTS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH; ++elementIndex)
		{
			cout << L_FORWARD_Pass_OUTPUTS[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == L_FORWARD_OutputLayer_WIDTH + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Backward Inputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_BACKWARD_NumberOf_INPUTS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < L_BACKWARD_InputLayer_HEIGHT * L_BACKWARD_InputLayer_WIDTH; ++elementIndex)
		{
			cout << L_BACKWARD_Pass_INPUTS[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == L_BACKWARD_InputLayer_WIDTH + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}


	cout << ">>>> Backward Outputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_BACKWARD_NumberOf_OUTPUTS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < L_BACKWARD_OutputLayer_HEIGHT * L_BACKWARD_OutputLayer_WIDTH; ++elementIndex)
		{
			cout << L_BACKWARD_Pass_OUTPUTS[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == L_BACKWARD_OutputLayer_WIDTH + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Filter Backprop Outputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_Filter_BACKPROP_RESULTS[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Bias Outputs Before Update <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH; ++elementIndex)
		{
			cout << L_PrevBiases[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == L_BACKWARD_OutputLayer_WIDTH + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}


	cout << ">>>> Bias Outputs After Update <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH; ++elementIndex)
		{
			cout << L_Biases[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == L_FORWARD_OutputLayer_WIDTH + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Filter Adam Optimizer V Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_AdamOptimizer_V_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Filter Adam Optimizer >CORRECTED< V Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_AdamOptimizer_Corrected_V_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Filter Adam Optimizer S Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_AdamOptimizer_S_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Filter Adam Optimizer >CORRECTED< S Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_AdamOptimizer_Corrected_S_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> BIAS Adam Optimizer V Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_BIAS_AdamOptimizer_V_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> BIAS Adam Optimizer >CORRECTED< V Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_BIAS_AdamOptimizer_Corrected_V_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> BIAS Adam Optimizer S Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_BIAS_AdamOptimizer_S_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> BIAS Adam Optimizer >CORRECTED< S Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_BIAS_AdamOptimizer_Corrected_S_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}
}


void Squishy::ForwardPass(float* forwardPassInput, int fwdPassHeight, int fwdPassWidth)
{

}

void Squishy::BackwardPass(float* backpropInput, int backPassHeight, int backPassWidth)
{

}

void Squishy::UpdateModule()
{

}


