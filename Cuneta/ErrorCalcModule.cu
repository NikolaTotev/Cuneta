#include "hip/hip_runtime.h"
#include "ErrorCalcModule.cuh"

#include <hip/hip_vector_types.h>

#ifndef __HIPCC__  
#define __HIPCC__
#endif

#ifndef  __CUDART_API_PER_THREAD_DEFAULT_STREAM
#define  __CUDART_API_PER_THREAD_DEFAULT_STREAM
#endif
#include <hip/hip_runtime_api.h>


#ifndef __CUDACC_RTC__ 
#define __CUDACC_RTC__

#endif

#include <algorithm>
#include <iostream>

#include ""
using namespace std;

__global__ void PixelWiseSigmoidKernel(float* d_Input, float* d_Output, int matrixWidth)
{
	int rowIndex = blockIdx.x;
	int columnIndex = threadIdx.x;
	int arrayIndex = rowIndex * matrixWidth + columnIndex;
	float pixel = d_Input[arrayIndex];

	float SigmoidResult = 1 / 1 + exp(pixel);

	d_Output[arrayIndex] = SigmoidResult;
};

__global__ void PixelWiseCrossEntropyKernel(float* d_Input, float* d_Output, float* d_GroundTruthMatrix, int matrixWidth)
{
	int rowIndex = blockIdx.x;
	int columnIndex = threadIdx.x;
	int arrayIndex = rowIndex * matrixWidth + columnIndex;

	float predictedPixel = d_Input[arrayIndex];
	float groundTruthClass = d_Input[arrayIndex];

	float correctedPixel = predictedPixel;
	if (groundTruthClass == 0)
	{
		correctedPixel = 1 - predictedPixel;
	}


	d_Output[arrayIndex] = -log(correctedPixel);
};

__global__ void LevelCrossEntropySumKernel(float* d_Input, float* d_Output, int matrixWidth)
{
	int blockStart = (blockIdx.x + 1) * 2;
	int rowNumber = blockStart + threadIdx.x;
	int arrayIndex = rowNumber * matrixWidth;

	float sum = 0;

	for (int i = 0; i < matrixWidth; ++i)
	{
		sum += d_Input[arrayIndex];
		arrayIndex++;
	}

	d_Output[arrayIndex] = sum;
};
__global__ void GradientGenerationKernel(float* d_Input, float* d_Output) {};


ErrorCalcModule::ErrorCalcModule(float* _inputMatrix, float* _groundTruth, int _inputHeight, int _inputWidth)
{
	m_InputMatrix = _inputMatrix;
	groundTruthMatrix = _groundTruth;
	m_InputMatrixHeight = _inputHeight;
	m_InputMatrixWidth = _inputWidth;

	m_OutputMatrixHeight = _inputHeight;
	m_OutputMatrixWidth = _inputWidth;

	m_OutputMatrix = new float[m_OutputMatrixHeight * m_OutputMatrixWidth];
}

void ErrorCalcModule::ForwardPass()
{
	PixelWiseSoftMax();
	PixelWiseCrossEntropy();
	CrossEntropySum();
}

void ErrorCalcModule::PixelWiseSoftMax()
{
	size_t totalPixelCount = m_InputMatrixHeight * m_InputMatrixWidth;
	int byteCount = totalPixelCount * sizeof(float);
	std::cout << "Pixel count " << totalPixelCount;

	sigmoidResultMatrix = new float[totalPixelCount];

	//Define pointers for deviceMemory locations
	float* d_Input;
	float* d_Output;

	//Allocate memory
	hipMalloc((void**)&d_Input, byteCount);
	hipMalloc((void**)&d_Output, byteCount);

	//Copy memory into global device memory m_InputMatrix -> d_Input
	hipMemcpy(d_Input, m_InputMatrix, byteCount, hipMemcpyHostToDevice);

	//Define block size and threads per block.
	dim3 blockGrid(m_InputMatrixHeight, 1, 1);
	dim3 threadGrid(m_InputMatrixWidth, 1, 1);

	PixelWiseSigmoidKernel << <blockGrid, threadGrid >> > (d_Input, d_Output, m_InputMatrixWidth);
	hipDeviceSynchronize();

	//Copy back result into host memory d_Output -> m_OutputMatrix
	hipMemcpy(sigmoidResultMatrix, d_Output, byteCount, hipMemcpyDeviceToHost);
}

void ErrorCalcModule::PixelWiseCrossEntropy()
{
	size_t totalPixelCount = m_InputMatrixHeight * m_InputMatrixWidth;
	int byteCount = totalPixelCount * sizeof(float);
	std::cout << "Pixel count " << totalPixelCount;

	crossEntropyResultMatrix = new float[totalPixelCount];

	//Define pointers for deviceMemory locations
	float* d_SigmoidInput;
	float* d_Output;
	float* d_GroundTruthMatrix;


	//Allocate memory
	hipMalloc((void**)&d_SigmoidInput, byteCount);
	hipMalloc((void**)&d_GroundTruthMatrix, byteCount);
	hipMalloc((void**)&d_Output, byteCount);


	//Copy memory into global device memory m_InputMatrix -> d_Input
	hipMemcpy(d_SigmoidInput, sigmoidResultMatrix, byteCount, hipMemcpyHostToDevice);
	hipMemcpy(d_GroundTruthMatrix, groundTruthMatrix, byteCount, hipMemcpyHostToDevice);


	//Define block size and threads per block.
	dim3 blockGrid(m_InputMatrixHeight, 1, 1);
	dim3 threadGrid(m_InputMatrixWidth, 1, 1);

	PixelWiseCrossEntropyKernel << <blockGrid, threadGrid >> > (d_SigmoidInput, d_Output, d_GroundTruthMatrix, m_InputMatrixWidth);
	hipDeviceSynchronize();

	//Copy back result into host memory d_Output -> m_OutputMatrix
	hipMemcpy(crossEntropyResultMatrix, d_Output, byteCount, hipMemcpyDeviceToHost);
}


void ErrorCalcModule::CrossEntropySum()
{
	int blockCount = m_InputMatrixHeight / 2;
	
	float* intermediateSum = new float[m_InputMatrixHeight];


	size_t inputPixelCount = m_InputMatrixHeight * m_InputMatrixWidth;
	int inputByteCount = inputPixelCount * sizeof(float);
	int outputByteCount = m_InputMatrixHeight * sizeof(float);
	std::cout << "Pixel count " << inputPixelCount ;

	float* d_Input;
	float* d_Output;

	//Allocate memory
	hipMalloc((void**)&d_Input, inputByteCount);
	hipMalloc((void**)&d_Output, outputByteCount);
	hipMemcpy(d_Input, crossEntropyResultMatrix, inputByteCount, hipMemcpyHostToDevice);

	//Define block size and threads per block.
	dim3 blockGrid(blockCount, 1, 1);
	dim3 threadGrid(m_InputMatrixWidth, 1, 1);

	LevelCrossEntropySumKernel<< <blockGrid, threadGrid >> > (d_Input, d_Output, m_InputMatrixWidth);
	hipDeviceSynchronize();

	hipMemcpy(intermediateSum, d_Output, outputByteCount, hipMemcpyDeviceToHost);

	networkError = 0;

	for (int i = 0; i < m_InputMatrixHeight; ++i)
	{
		networkError += intermediateSum[i];
	}
	cout << "Network error " << networkError << endl;
}




void ErrorCalcModule::BackwardPass()
{

}

void ErrorCalcModule::UpdateModule()
{

}


