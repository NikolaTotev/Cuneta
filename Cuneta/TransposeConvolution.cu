#include "hip/hip_runtime.h"
#include <hip/hip_vector_types.h>

#ifndef __HIPCC__  
#define __HIPCC__
#endif

#ifndef  __CUDART_API_PER_THREAD_DEFAULT_STREAM
#define  __CUDART_API_PER_THREAD_DEFAULT_STREAM
#endif
#include <hip/hip_runtime_api.h>


#ifndef __CUDACC_RTC__ 
#define __CUDACC_RTC__

#endif

#include <algorithm>
#include <iostream>

#include ""
#include "TransposeConvolution.cuh"
#include <random>
#include <cmath>

#ifndef __HIPCC__  
#define __HIPCC__
#endif

#ifndef __CUDACC_RTC__ 
#define __CUDACC_RTC__
#endif

#include <hip/device_functions.h>
using namespace std;

__global__ void TransposeConvolutionKernel(float* d_Input, float* d_Filter, float* d_Output, int _outputWidth, int _convolutionInputWidth, int filterHeight, int filterWidth)
{
	//These define indecies for output matrix;
	int outputRowIndex = blockIdx.x;
	int outputColumnIndex = threadIdx.x;

	//Starts from "top left" of current block of pixels being processed
	int inputRowIndex = blockIdx.x;
	int inputColumnIndex = threadIdx.x;

	int outputArrayIndex = outputRowIndex * _outputWidth + outputColumnIndex;

	int inputArrayIndex = 0;

	float result = 0;
	int filterIndex = 0;
	int temp = 0;
	for (int row = 0; row < filterHeight; row++)
	{
		inputColumnIndex = threadIdx.x;


		for (int col = 0; col < filterWidth; col++)
		{

			inputArrayIndex = inputRowIndex * _convolutionInputWidth + inputColumnIndex;

			result += d_Input[inputArrayIndex] * d_Filter[filterIndex];
			filterIndex++;
			inputColumnIndex += 1;
		}
		inputRowIndex += 1;
	}

	d_Output[outputArrayIndex] = result;
};

__global__ void LayerTransposeConvolutionKernel(float** _inputs, float** _filters, float** _outputs, float** _biases, int _numberOfInputs, int _outputWidth, int _inputWidth, int _filterSize)
{
	int inputSelectionIndex = threadIdx.x;
	int filterSelectionIndex = blockIdx.z * _numberOfInputs + threadIdx.x;
	int biasSelectionIndex = blockIdx.z * _numberOfInputs + threadIdx.x;
	int outputSelectionIndex = blockIdx.z;

	float* selectedInput = _inputs[inputSelectionIndex];
	float* selectedFilter = _filters[filterSelectionIndex];
	float* selectedBias = _biases[filterSelectionIndex];
	float* selectedOutput = _outputs[outputSelectionIndex];

	int inputStartReadRowIndex = blockIdx.x;
	int inputStartReadColumnIndex = blockIdx.y;

	int outputWriteRowIndex = blockIdx.x*2;
	int outputWriteColumnIndex = blockIdx.y*2;

	int inputArrayIndex = inputStartReadRowIndex * _inputWidth + inputStartReadColumnIndex;

	int outputArrayIndex = outputWriteRowIndex * _outputWidth + outputWriteColumnIndex;

	float result = 0;
	int filterIndex = 0;
	int temp = 0;
	for (int row = 0; row < _filterSize; row++)
	{
		outputWriteColumnIndex = blockIdx.y * 2;

		for (int col = 0; col < _filterSize; col++)
		{
			outputArrayIndex = outputWriteRowIndex* _outputWidth + outputWriteColumnIndex;
			result =  selectedInput[inputArrayIndex] * selectedFilter[filterIndex];
			atomicAdd(&selectedOutput[outputArrayIndex], result);
			filterIndex++;
			outputWriteColumnIndex++;
		}
		outputWriteRowIndex++;
	}
};

__global__ void LayerTransposeConvolutionBackPropKernel(float** _inputs, float** _filters, float** _outputs, int _numberOfOutputs, int _outputWidth, int _inputWidth, int _filterSize)
{
	int inputSelectionIndex = threadIdx.x;
	int filterSelectionIndex = blockIdx.z * _numberOfOutputs + threadIdx.x;
	int outputSelectionIndex = blockIdx.z;

	float* selectedInput = _inputs[inputSelectionIndex];
	float* selectedFilter = _filters[filterSelectionIndex];
	float* selectedOutput = _outputs[outputSelectionIndex];

	int inputStartReadRowIndex = blockIdx.x*2;
	int inputStartReadColumnIndex = blockIdx.y*2;

	int outputWriteRowIndex = blockIdx.x;
	int outputWriteColumnIndex = blockIdx.y;

	int inputArrayIndex = 0;

	int outputArrayIndex = outputWriteRowIndex * _outputWidth + outputWriteColumnIndex;

	float result = 0;
	int filterIndex = 0;
	int temp = 0;
	for (int row = 0; row < _filterSize; row++)
	{
		inputStartReadColumnIndex = blockIdx.y*2;

		for (int col = 0; col < _filterSize; col++)
		{
			inputArrayIndex = inputStartReadRowIndex * _inputWidth + inputStartReadColumnIndex;

			result += selectedInput[inputArrayIndex] * selectedFilter[filterIndex];
			filterIndex++;
			inputStartReadColumnIndex += 1;
		}
		inputStartReadRowIndex += 1;
	}

	atomicAdd(&selectedOutput[outputArrayIndex], result);
}

__global__ void PaddingKernel(float* d_UnpaddedInput, float* d_Output, int _paddedInputWidth, int _unpaddedInputWidth, int _unpaddedInputHeight)
{
	int rowWriteIndex = (blockIdx.x + 1) * 2 + threadIdx.x;
	int columnWriteIndex = 2;

	int inputRowReadIndex = (blockIdx.x * 2) + threadIdx.x;
	int inputColumnReadIndex = 0;

	int arrayPosition = rowWriteIndex * _paddedInputWidth + columnWriteIndex;
	int inputArrayPosition = inputRowReadIndex * _unpaddedInputWidth + inputColumnReadIndex;

	int var = 0;
	for (int i = 0; i < _unpaddedInputWidth; i++)
	{
		d_Output[arrayPosition] = d_UnpaddedInput[inputArrayPosition];
		arrayPosition++;
		inputArrayPosition++;
	}
}

__global__ void LayerTransposeInputPaddingKernel(float** __inputs, float** _outputs, int _unpaddedWidth, int _paddedWidth)
{
	int inputSelectionIndex = blockIdx.y;
	int outputSelectionIndex = blockIdx.y;

	float* selectedInput = __inputs[inputSelectionIndex];
	float* selectedOutput = _outputs[outputSelectionIndex];

	int rowWriteIndex = (blockIdx.x + 1) * 2 + threadIdx.x;
	int columnWriteIndex = 2;

	int inputRowReadIndex = (blockIdx.x * 2) + threadIdx.x;
	int inputColumnReadIndex = 0;

	int arrayPosition = rowWriteIndex * _paddedWidth + columnWriteIndex;
	int inputArrayPosition = inputRowReadIndex * _unpaddedWidth + inputColumnReadIndex;

	int var = 0;
	for (int i = 0; i < _unpaddedWidth; i++)
	{
		selectedOutput[arrayPosition] = selectedInput[inputArrayPosition];
		arrayPosition++;
		inputArrayPosition++;
	}
}

__global__ void LayerTransposeConvFilterFlipKernel(float** _inputFilters, float** _outputFilters, int _filterSize)
{
	float* filterToFlip = _inputFilters[blockIdx.x];
	int filterArraySize = _filterSize * _filterSize;
	float* flippedOutput = _outputFilters[blockIdx.x];
	int k = 0;

	//Loop from back and assign value to new array
	for (int i = filterArraySize - 1; i >= 0; ) {
		flippedOutput[k++] = filterToFlip[i--];
	}
}

__global__ void LayerTransposeConvFilterBackpropKernel(float** _forwardInputs, float** _outputs, float** _backpropInputs, int _inputsWidth, int _outputsWidth, int _backpropInputHeight, int _backpropInputWidth, int _numberOfInputs)
{
	int forwardInputSelectionIndex = threadIdx.x;
	int backpropInputSelectionIndex = blockIdx.z;
	int outputSelectionIndex = blockIdx.z * _numberOfInputs + threadIdx.x;

	float* selectedForwardInput = _forwardInputs[forwardInputSelectionIndex];
	float* selectedBackpropInput = _backpropInputs[backpropInputSelectionIndex];
	float* selectedOutput = _outputs[outputSelectionIndex];

	int inputStartReadRowIndex = blockIdx.x;
	int inputStartReadColumnIndex = blockIdx.y;

	int backpropInputStartReadRowIndex = inputStartReadRowIndex * 2;
	int backpropInputStartReadColumnIndex = inputStartReadColumnIndex * 2;

	int backpropArrayIndex = backpropInputStartReadRowIndex * _backpropInputWidth + backpropInputStartReadColumnIndex;
	
	int outputWriteIndex = 0;

	int inputArrayIndex = inputStartReadRowIndex * _inputsWidth + inputStartReadColumnIndex;

	
	float result = 0;
	int filterIndex = 0;

	for (int row = 0; row < _outputsWidth; row++)
	{
		backpropInputStartReadColumnIndex = inputStartReadColumnIndex * 2;
		for (int col = 0; col < _outputsWidth; col++)
		{
			backpropArrayIndex = backpropInputStartReadRowIndex * _backpropInputWidth + backpropInputStartReadColumnIndex;
			result = selectedForwardInput[inputArrayIndex] * selectedBackpropInput[backpropArrayIndex];
			atomicAdd(&selectedOutput[outputWriteIndex], result);
			backpropInputStartReadColumnIndex ++;
			outputWriteIndex++;
		}
		backpropInputStartReadRowIndex++;
	}
}

__global__ void TransposeConvFilterUpdateKernel(float** _currentFilters, float** _filterGradients, float** _VMatricies, float** _SMatricies, float** _V_CorrectedMatrices, float** _S_CorrectedMatricies, int _filterSize, int _HyperParam_Beta1, int _HyperParam_Beta2, int _HyperParam_T, int _HyperParam_alpha, int _HyperParam_Epsilon)
{
	float* selectedFilter = _currentFilters[blockIdx.x];
	float* selectedGradient = _filterGradients[blockIdx.x];
	float* selected_V_Matrix = _VMatricies[blockIdx.x];
	float* selected_S_Matrix = _SMatricies[blockIdx.x];
	float* selected_Corrected_V_Matrix = _V_CorrectedMatrices[blockIdx.x];
	float* selected_Corrected_S_Matrix = _S_CorrectedMatricies[blockIdx.x];

	for (int rowIndex = 0; rowIndex < _filterSize; ++rowIndex)
	{
		for (int columnIndex = 0; columnIndex < _filterSize; ++columnIndex)
		{
			int index = rowIndex * _filterSize + columnIndex;

			float filterBackpropValue = selectedGradient[index];
			float oldV = selected_V_Matrix[index];
			float oldS = selected_S_Matrix[index];

			float newV = _HyperParam_Beta1 * oldV + (1 - _HyperParam_Beta1) * filterBackpropValue;
			float newS = _HyperParam_Beta2 * oldS + (1 - _HyperParam_Beta2) * filterBackpropValue;

			float newVCorrected = newV / (1 - pow(_HyperParam_Beta1, _HyperParam_T));
			float newSCorrected = newS / (1 - pow(_HyperParam_Beta2, _HyperParam_T));

			selected_V_Matrix[index] = newV;
			selected_S_Matrix[index] = newS;

			selected_Corrected_V_Matrix[index] = newVCorrected;
			selected_Corrected_S_Matrix[index] = newSCorrected;

			float oldFilterValue = selectedFilter[index];
			float newF = oldFilterValue - _HyperParam_alpha * (newVCorrected / sqrt(newSCorrected + _HyperParam_Epsilon));

			selectedFilter[index] = newF;
		}
	}
}

__global__ void TransposeConvBiasUpdateKernel(float** _currentFilters, float** _filterGradients, float** _VMatricies, float** _SMatricies, float** _V_CorrectedMatrices, float** _S_CorrectedMatricies, int _height, int _width, int _HyperParam_Beta1, int _HyperParam_Beta2, int _HyperParam_T, int _HyperParam_alpha, int _HyperParam_Epsilon)
{
	int filterSelectionIndex = blockIdx.x * blockDim.x + threadIdx.x;

	float* selectedFilter = _currentFilters[filterSelectionIndex];
	float* selectedGradient = _filterGradients[blockIdx.x];
	float* selected_V_Matrix = _VMatricies[filterSelectionIndex];
	float* selected_S_Matrix = _SMatricies[filterSelectionIndex];
	float* selected_Corrected_V_Matrix = _V_CorrectedMatrices[filterSelectionIndex];
	float* selected_Corrected_S_Matrix = _S_CorrectedMatricies[filterSelectionIndex];

	for (int rowIndex = 0; rowIndex < _height; ++rowIndex)
	{
		for (int columnIndex = 0; columnIndex < _width; ++columnIndex)
		{
			int index = rowIndex * _width + columnIndex;

			float filterBackpropValue = selectedGradient[index];
			float oldV = selected_V_Matrix[index];
			float oldS = selected_S_Matrix[index];

			float newV = _HyperParam_Beta1 * oldV + (1 - _HyperParam_Beta1) * filterBackpropValue;
			float newS = _HyperParam_Beta2 * oldS + (1 - _HyperParam_Beta2) * filterBackpropValue;

			float newVCorrected = newV / (1 - pow(_HyperParam_Beta1, _HyperParam_T));
			float newSCorrected = newS / (1 - pow(_HyperParam_Beta2, _HyperParam_T));

			selected_V_Matrix[index] = newV;
			selected_S_Matrix[index] = newS;

			selected_Corrected_V_Matrix[index] = newVCorrected;
			selected_Corrected_S_Matrix[index] = newSCorrected;

			float oldFilterValue = selectedFilter[index];
			float newF = oldFilterValue - _HyperParam_alpha * (newVCorrected / sqrt(newSCorrected + _HyperParam_Epsilon));

			selectedFilter[index] = newF;
		}
	}
}

TransposeConvolution::TransposeConvolution()
{
	
}

TransposeConvolution::TransposeConvolution(int _filterSize, int _paddingSize, int _numberOfInputs, int _numberOfOutputs, int _inputHeight, int _inputWidth, int _layerID, int _levelID)
{
	layerID = _layerID;
	levelID = _levelID;
	m_FilterSize = 2;
	m_PaddingSize = _paddingSize;

	L_FORWARD_NumberOf_INPUTS = _numberOfInputs;
	L_FORWARD_NumberOf_OUTPUTS = _numberOfOutputs;

	L_BACKWARD_NumberOf_INPUTS = L_FORWARD_NumberOf_OUTPUTS;
	L_BACKWARD_NumberOf_OUTPUTS = L_FORWARD_NumberOf_INPUTS;

	L_FORWARD_InputLayer_HEIGHT = _inputHeight;
	L_FORWARD_InputLayer_WIDTH = _inputWidth;

	L_FORWARD_OutputLayer_HEIGHT = _inputHeight * 2;
	L_FORWARD_OutputLayer_WIDTH = _inputWidth * 2;

	L_BACKWARD_InputLayer_HEIGHT = L_FORWARD_OutputLayer_HEIGHT;
	L_BACKWARD_InputLayer_WIDTH = L_FORWARD_OutputLayer_WIDTH;

	L_BACKWARD_OutputLayer_HEIGHT = L_FORWARD_InputLayer_HEIGHT;
	L_BACKWARD_OutputLayer_WIDTH = L_FORWARD_InputLayer_WIDTH;

	L_FORWARD_Pass_INPUTS = new float* [L_FORWARD_NumberOf_INPUTS];
	L_FORWARD_Pass_OUTPUTS = new float* [L_FORWARD_NumberOf_OUTPUTS];

	L_BACKWARD_Pass_INPUTS = new float* [L_BACKWARD_NumberOf_INPUTS];
	L_BACKWARD_Pass_OUTPUTS = new float* [L_BACKWARD_NumberOf_OUTPUTS];

	L_NumberOf_FILTERS = L_FORWARD_NumberOf_INPUTS * L_FORWARD_NumberOf_OUTPUTS;

	L_Filters = new float* [L_NumberOf_FILTERS];
	L_FLIPPED_Filters = new float* [L_NumberOf_FILTERS];
	L_Filter_BACKPROP_RESULTS = new float* [L_NumberOf_FILTERS];

	L_PrevBiases = new float* [L_NumberOf_FILTERS];

	L_AdamOptimizer_V_Matrix = new float* [L_NumberOf_FILTERS];
	L_AdamOptimizer_S_Matrix = new float* [L_NumberOf_FILTERS];
	L_AdamOptimizer_Corrected_V_Matrix = new float* [L_NumberOf_FILTERS];
	L_AdamOptimizer_Corrected_S_Matrix = new float* [L_NumberOf_FILTERS];

	L_BIAS_AdamOptimizer_V_Matrix = new float* [L_NumberOf_FILTERS];
	L_BIAS_AdamOptimizer_S_Matrix = new float* [L_NumberOf_FILTERS];
	L_BIAS_AdamOptimizer_Corrected_V_Matrix = new float* [L_NumberOf_FILTERS];
	L_BIAS_AdamOptimizer_Corrected_S_Matrix = new float* [L_NumberOf_FILTERS];


	for (int i = 0; i < L_NumberOf_FILTERS; ++i)
	{
		L_AdamOptimizer_V_Matrix[i] = new float[m_FilterSize * m_FilterSize];
		L_AdamOptimizer_S_Matrix[i] = new float[m_FilterSize * m_FilterSize];
		L_AdamOptimizer_Corrected_V_Matrix[i] = new float[m_FilterSize * m_FilterSize];
		L_AdamOptimizer_Corrected_S_Matrix[i] = new float[m_FilterSize * m_FilterSize];

		size_t byteCount = m_FilterSize * m_FilterSize * sizeof(float);
		memset(L_AdamOptimizer_V_Matrix[i], 0, byteCount);
		memset(L_AdamOptimizer_S_Matrix[i], 0, byteCount);
		memset(L_AdamOptimizer_Corrected_V_Matrix[i], 0, byteCount);
		memset(L_AdamOptimizer_Corrected_S_Matrix[i], 0, byteCount);

		L_BIAS_AdamOptimizer_V_Matrix[i] = new float[L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH];
		L_BIAS_AdamOptimizer_S_Matrix[i] = new float[L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH];
		L_BIAS_AdamOptimizer_Corrected_V_Matrix[i] = new float[L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH];
		L_BIAS_AdamOptimizer_Corrected_S_Matrix[i] = new float[L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH];

		size_t biasByteCount = L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH * sizeof(float);
		memset(L_BIAS_AdamOptimizer_V_Matrix[i], 0, biasByteCount);
		memset(L_BIAS_AdamOptimizer_S_Matrix[i], 0, biasByteCount);
		memset(L_BIAS_AdamOptimizer_Corrected_V_Matrix[i], 0, biasByteCount);
		memset(L_BIAS_AdamOptimizer_Corrected_S_Matrix[i], 0, biasByteCount);
	}

	LayerFilterInitialization();
	LayerBiasInitialization();
}


void TransposeConvolution::LayerForwardPass(float** _inputs)
{
	int inputSize = L_FORWARD_InputLayer_HEIGHT * L_FORWARD_InputLayer_WIDTH;

	int filterSize = m_FilterSize * m_FilterSize;

	int biasSize = L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH;

	int outputSize = L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH;

	size_t inputByteCount = inputSize * sizeof(float);
	size_t filterByteCount = filterSize * sizeof(float);
	size_t biasByteCount = biasSize * sizeof(float);
	size_t outputByteCount = outputSize * sizeof(float);

	for (int inputNumber = 0; inputNumber < L_FORWARD_NumberOf_INPUTS; ++inputNumber)
	{
		L_FORWARD_Pass_INPUTS[inputNumber] = new float[inputSize];
		memcpy(L_FORWARD_Pass_INPUTS[inputNumber], _inputs[inputNumber], inputByteCount);
	}

	for (int outputNumber = 0; outputNumber < L_FORWARD_NumberOf_OUTPUTS; ++outputNumber)
	{
		L_FORWARD_Pass_OUTPUTS[outputNumber] = new float[outputSize];
	}


	int numberOfBlockx_X = L_FORWARD_InputLayer_HEIGHT;//L_FORWARD_OutputLayer_HEIGHT;
	int numberOfBlocks_Y = L_FORWARD_InputLayer_WIDTH;//L_FORWARD_OutputLayer_WIDTH;
	int numberOfBlocks_Z = L_FORWARD_NumberOf_OUTPUTS;//L_FORWARD_NumberOf_OUTPUTS;
	int numberOfThreadsPerBlock = L_FORWARD_NumberOf_INPUTS;//L_FORWARD_NumberOf_INPUTS;

	float** h_Inputs = new float* [L_FORWARD_NumberOf_INPUTS];  //(float**)malloc(L_FORWARD_NumberOf_INPUTS * sizeof(float*));
	float** h_Filters = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_Biases = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_Outputs = new float* [L_FORWARD_NumberOf_OUTPUTS]; //(float**)malloc(L_FORWARD_NumberOf_OUTPUTS * sizeof(float*));

	float** d_BiasesPointerArray;
	hipMalloc((void**)&d_BiasesPointerArray, L_NumberOf_FILTERS * sizeof(float*));

	float** d_InputPointerArray;
	hipMalloc((void**)&d_InputPointerArray, L_FORWARD_NumberOf_INPUTS * sizeof(float*));

	float** d_FilterPointerArray;
	hipMalloc((void**)&d_FilterPointerArray, L_NumberOf_FILTERS * sizeof(float*));

	float** d_OutputPointerArray;
	hipMalloc((void**)&d_OutputPointerArray, L_FORWARD_NumberOf_OUTPUTS * sizeof(float*));

	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_FORWARD_NumberOf_INPUTS; i++) {
		hipMalloc(&h_Inputs[i], inputByteCount);
		hipMemcpy(h_Inputs[i], L_FORWARD_Pass_INPUTS[i], inputByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Filters[i], filterByteCount);
		hipMemcpy(h_Filters[i], L_Filters[i], filterByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Biases[i], biasByteCount);
		hipMemcpy(h_Biases[i], L_Biases[i], biasByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_FORWARD_NumberOf_OUTPUTS; i++) {
		hipMalloc(&h_Outputs[i], outputByteCount);
		hipMemset(h_Outputs[i], 0, outputByteCount);
	}

	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_InputPointerArray, h_Inputs, L_FORWARD_NumberOf_INPUTS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_FilterPointerArray, h_Filters, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_BiasesPointerArray, h_Biases, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_OutputPointerArray, h_Outputs, L_FORWARD_NumberOf_OUTPUTS * sizeof(float*), hipMemcpyHostToDevice);

	dim3 blockGrid(numberOfBlockx_X, numberOfBlocks_Y, numberOfBlocks_Z); ///OK
	dim3 threads(numberOfThreadsPerBlock, 1, 1); ///OK

	LayerTransposeConvolutionKernel << <blockGrid, threads >> > (d_InputPointerArray, d_FilterPointerArray, d_OutputPointerArray, d_BiasesPointerArray, L_FORWARD_NumberOf_INPUTS, L_FORWARD_OutputLayer_WIDTH, L_FORWARD_InputLayer_WIDTH, m_FilterSize);
	hipDeviceSynchronize();
	float* temp = new float[outputSize];
	hipMemcpy(h_Outputs, d_OutputPointerArray, L_FORWARD_NumberOf_OUTPUTS * sizeof(float*), hipMemcpyDeviceToHost);
	for (size_t i = 0; i < L_FORWARD_NumberOf_OUTPUTS; i++) {

		hipMemcpy(temp, h_Outputs[i], outputByteCount, hipMemcpyDeviceToHost);
		memcpy(L_FORWARD_Pass_OUTPUTS[i], temp, outputByteCount);
		hipFree(h_Outputs[i]);
	}
	hipFree(d_OutputPointerArray);
	delete[] h_Outputs;

	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_FORWARD_NumberOf_INPUTS; i++) {
		hipFree(h_Inputs[i]);
	}
	hipFree(d_InputPointerArray);
	delete[] h_Inputs;

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipFree(h_Filters[i]);
	}
	hipFree(d_FilterPointerArray);
	delete[] h_Filters;
}


void TransposeConvolution::LayerBackwardPass(float** _backpropInput)
{
	int inputSize = L_BACKWARD_InputLayer_HEIGHT * L_BACKWARD_InputLayer_WIDTH;

	int filterSize = m_FilterSize * m_FilterSize;

	int outputSize = L_BACKWARD_OutputLayer_HEIGHT * L_BACKWARD_OutputLayer_WIDTH;

	size_t inputByteCount = inputSize * sizeof(float);
	size_t filterByteCount = filterSize * sizeof(float);
	size_t outputByteCount = outputSize * sizeof(float);

	for (int inputNumber = 0; inputNumber < L_BACKWARD_NumberOf_INPUTS; ++inputNumber)
	{
		L_BACKWARD_Pass_INPUTS[inputNumber] = new float[inputSize];
		memcpy(L_BACKWARD_Pass_INPUTS[inputNumber], _backpropInput[inputNumber], inputByteCount);
	}

	for (int outputNumber = 0; outputNumber < L_BACKWARD_NumberOf_OUTPUTS; ++outputNumber)
	{
		L_BACKWARD_Pass_OUTPUTS[outputNumber] = new float[outputSize];
	}


	int numberOfBlockx_X = L_FORWARD_InputLayer_HEIGHT;
	int numberOfBlocks_Y = L_FORWARD_InputLayer_WIDTH;
	int numberOfBlocks_Z = L_BACKWARD_NumberOf_OUTPUTS;
	int numberOfThreadsPerBlock = L_BACKWARD_NumberOf_INPUTS;

	// create intermediate host array for storage of device row-pointers

	// create top-level device array pointer
	float** h_Inputs = new float* [L_BACKWARD_NumberOf_INPUTS];  //(float**)malloc(L_FORWARD_NumberOf_INPUTS * sizeof(float*));
	float** h_Filters = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_Outputs = new float* [L_BACKWARD_NumberOf_OUTPUTS]; //(float**)malloc(L_FORWARD_NumberOf_OUTPUTS * sizeof(float*));

	float* d_Biases;
	hipMalloc((void**)&d_Biases, L_NumberOf_FILTERS * sizeof(float));

	float** d_InputPointerArray;
	hipMalloc((void**)&d_InputPointerArray, L_BACKWARD_NumberOf_INPUTS * sizeof(float*));

	float** d_FilterPointerArray;
	hipMalloc((void**)&d_FilterPointerArray, L_NumberOf_FILTERS * sizeof(float*));

	float** d_OutputPointerArray;
	hipMalloc((void**)&d_OutputPointerArray, L_BACKWARD_NumberOf_OUTPUTS * sizeof(float*));


	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_BACKWARD_NumberOf_INPUTS; i++) {
		hipMalloc(&h_Inputs[i], inputByteCount);
		hipMemcpy(h_Inputs[i], L_BACKWARD_Pass_INPUTS[i], inputByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Filters[i], filterByteCount);
		hipMemcpy(h_Filters[i], L_Filters[i], filterByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_BACKWARD_NumberOf_OUTPUTS; i++) {
		hipMalloc(&h_Outputs[i], outputByteCount);
		hipMemset(h_Outputs[i], 0, outputByteCount);
	}

	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_InputPointerArray, h_Inputs, L_BACKWARD_NumberOf_INPUTS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_FilterPointerArray, h_Filters, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_OutputPointerArray, h_Outputs, L_BACKWARD_NumberOf_OUTPUTS * sizeof(float*), hipMemcpyHostToDevice);

	dim3 blockGrid(numberOfBlockx_X, numberOfBlocks_Y, numberOfBlocks_Z); ///OK
	dim3 threads(numberOfThreadsPerBlock, 1, 1); ///OK

	LayerTransposeConvolutionBackPropKernel << <blockGrid, threads >> > (d_InputPointerArray, d_FilterPointerArray, d_OutputPointerArray, L_BACKWARD_NumberOf_INPUTS, L_BACKWARD_OutputLayer_WIDTH, L_BACKWARD_InputLayer_WIDTH, m_FilterSize);
	hipDeviceSynchronize();
	float* temp = new float[outputSize];
	hipMemcpy(h_Outputs, d_OutputPointerArray, L_BACKWARD_NumberOf_OUTPUTS * sizeof(float*), hipMemcpyDeviceToHost);
	for (size_t i = 0; i < L_BACKWARD_NumberOf_OUTPUTS; i++) {

		hipMemcpy(temp, h_Outputs[i], outputByteCount, hipMemcpyDeviceToHost);
		memcpy(L_BACKWARD_Pass_OUTPUTS[i], temp, outputByteCount);
		hipFree(h_Outputs[i]);
	}
	hipFree(d_OutputPointerArray);
	delete[] h_Outputs;

	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_BACKWARD_NumberOf_INPUTS; i++) {
		hipFree(h_Inputs[i]);
	}
	hipFree(d_InputPointerArray);
	delete[] h_Inputs;

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipFree(h_Filters[i]);
	}
	hipFree(d_FilterPointerArray);
	delete[] h_Filters;

	LayerFilterBackprop();
	//LayerUpdate();
}

void TransposeConvolution::LayerFilterBackprop()
{
	int forwardInputSize = L_FORWARD_InputLayer_HEIGHT *L_FORWARD_InputLayer_WIDTH;

	int backpropInputSize = L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH;

	int outputSize = m_FilterSize * m_FilterSize;

	size_t forwardInputByteCount = forwardInputSize * sizeof(float);
	size_t backpropInputByteCount = backpropInputSize * sizeof(float);
	size_t outputByteCount = outputSize * sizeof(float);


	for (int outputNumber = 0; outputNumber < L_NumberOf_FILTERS; ++outputNumber)
	{
		L_Filter_BACKPROP_RESULTS[outputNumber] = new float[outputSize];
	}


	int numberOfBlockx_X = L_FORWARD_InputLayer_HEIGHT;
	int numberOfBlocks_Y = L_FORWARD_InputLayer_WIDTH;
	int numberOfBlocks_Z = L_BACKWARD_NumberOf_INPUTS;
	int numberOfThreadsPerBlock = L_FORWARD_NumberOf_INPUTS;

	// create intermediate host array for storage of device row-pointers

	// create top-level device array pointer
	float** h_Inputs = new float* [L_FORWARD_NumberOf_INPUTS];  //(float**)malloc(L_FORWARD_NumberOf_INPUTS * sizeof(float*));
	float** h_Filters = new float* [L_BACKWARD_NumberOf_INPUTS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_Outputs = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_FORWARD_NumberOf_OUTPUTS * sizeof(float*));



	float** d_ForwarInputPointerArray;
	hipMalloc((void**)&d_ForwarInputPointerArray, L_FORWARD_NumberOf_INPUTS * sizeof(float*));

	float** d_BackpropInputPointerArray;
	hipMalloc((void**)&d_BackpropInputPointerArray, L_BACKWARD_NumberOf_INPUTS * sizeof(float*));

	float** d_OutputPointerArray;
	hipMalloc((void**)&d_OutputPointerArray, L_NumberOf_FILTERS * sizeof(float*));


	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_FORWARD_NumberOf_INPUTS; i++) {
		hipMalloc(&h_Inputs[i], forwardInputByteCount);
		hipMemcpy(h_Inputs[i], L_FORWARD_Pass_INPUTS[i], forwardInputByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_BACKWARD_NumberOf_INPUTS; i++) {
		hipMalloc(&h_Filters[i], backpropInputByteCount);
		hipMemcpy(h_Filters[i], L_BACKWARD_Pass_INPUTS[i], backpropInputByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Outputs[i], outputByteCount);
		hipMemset(h_Outputs[i], 0, outputByteCount);
	}

	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_ForwarInputPointerArray, h_Inputs, L_FORWARD_NumberOf_INPUTS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_BackpropInputPointerArray, h_Filters, L_BACKWARD_NumberOf_INPUTS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_OutputPointerArray, h_Outputs, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);

	dim3 blockGrid(numberOfBlockx_X, numberOfBlocks_Y, numberOfBlocks_Z); ///OK
	dim3 threads(numberOfThreadsPerBlock, 1, 1); ///OK

	LayerTransposeConvFilterBackpropKernel << <blockGrid, threads >> > (d_ForwarInputPointerArray, d_OutputPointerArray, d_BackpropInputPointerArray, L_FORWARD_InputLayer_WIDTH, m_FilterSize, L_BACKWARD_InputLayer_HEIGHT, L_BACKWARD_InputLayer_WIDTH, L_FORWARD_NumberOf_INPUTS);
	hipDeviceSynchronize();

	float* temp = new float[outputSize];
	hipMemcpy(h_Outputs, d_OutputPointerArray, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {

		hipMemcpy(temp, h_Outputs[i], outputByteCount, hipMemcpyDeviceToHost);
		memcpy(L_Filter_BACKPROP_RESULTS[i], temp, outputByteCount);
		hipFree(h_Outputs[i]);
	}
	hipFree(d_OutputPointerArray);
	delete[] h_Outputs;

	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_FORWARD_NumberOf_INPUTS; i++) {
		hipFree(h_Inputs[i]);
	}
	hipFree(d_ForwarInputPointerArray);
	delete[] h_Inputs;

	for (size_t i = 0; i < L_BACKWARD_NumberOf_INPUTS; i++) {
		hipFree(h_Filters[i]);
	}
	hipFree(d_BackpropInputPointerArray);
	delete[] h_Filters;
}

void TransposeConvolution::LayerFlipFilter()
{
	int inputSize = m_FilterSize * m_FilterSize;

	int outputSize = m_FilterSize * m_FilterSize;

	size_t inputByteCount = inputSize * sizeof(float);
	size_t outputByteCount = outputSize * sizeof(float);

	for (int filterNumber = 0; filterNumber < L_NumberOf_FILTERS; ++filterNumber)
	{
		L_FLIPPED_Filters[filterNumber] = new float[inputSize];
	}

	int numberOfBlockx_X = L_NumberOf_FILTERS;

	// create intermediate host array for storage of device row-pointers

	// create top-level device array pointer
	float** h_Inputs = new float* [L_NumberOf_FILTERS];

	float** h_Outputs = new float* [L_NumberOf_FILTERS];

	float** d_InputPointerArray;
	hipMalloc((void**)&d_InputPointerArray, L_NumberOf_FILTERS * sizeof(float*));


	float** d_OutputPointerArray;
	hipMalloc((void**)&d_OutputPointerArray, L_NumberOf_FILTERS * sizeof(float*));


	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Inputs[i], inputByteCount);
		hipMemcpy(h_Inputs[i], L_Filters[i], inputByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Outputs[i], outputByteCount);
	}

	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_InputPointerArray, h_Inputs, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_OutputPointerArray, h_Outputs, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);

	dim3 blockGrid(numberOfBlockx_X, 1, 1); ///OK
	dim3 threads(1, 1, 1); ///OK

	LayerTransposeConvFilterFlipKernel << <blockGrid, threads >> > (d_InputPointerArray, d_OutputPointerArray, m_FilterSize);
	hipDeviceSynchronize();

	float* temp = new float[outputSize];

	hipMemcpy(h_Outputs, d_OutputPointerArray, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {

		hipMemcpy(L_FLIPPED_Filters[i], h_Outputs[i], outputByteCount, hipMemcpyDeviceToHost);
		hipFree(h_Outputs[i]);
	}
	hipFree(d_OutputPointerArray);
	delete[] h_Outputs;

	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipFree(h_Inputs[i]);
	}
	hipFree(d_InputPointerArray);
	delete[] h_Inputs;

	delete[] temp;
}


void TransposeConvolution::LayerUpdate()
{
	int filterSize = m_FilterSize * m_FilterSize;

	size_t filterByteCount = filterSize * sizeof(float);


	int numberOfBlocks_X = L_NumberOf_FILTERS;
	int numberOfThreadsPerBlock = 1;

	// create intermediate host array for storage of device row-pointers

	// create top-level device array pointer
	float** h_Filters = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_FilterGradients = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_V_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_S_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_V_CORRECTED_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_S_CORRECTED_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));


	float** d_FilterPointers;
	hipMalloc((void**)&d_FilterPointers, L_NumberOf_FILTERS * sizeof(float*));

	float** d_FilterGradientPointers;
	hipMalloc((void**)&d_FilterGradientPointers, L_NumberOf_FILTERS * sizeof(float*));

	float** d_V_Matricies;
	hipMalloc((void**)&d_V_Matricies, L_NumberOf_FILTERS * sizeof(float*));

	float** d_S_Matricies;
	hipMalloc((void**)&d_S_Matricies, L_NumberOf_FILTERS * sizeof(float*));

	float** d_V_CORRECTED_Matricies;
	hipMalloc((void**)&d_V_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*));

	float** d_S_CORRECTED_Matricies;
	hipMalloc((void**)&d_S_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*));



	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Filters[i], filterByteCount);
		hipMemcpy(h_Filters[i], L_Filters[i], filterByteCount, hipMemcpyHostToDevice);


		hipMalloc(&h_FilterGradients[i], filterByteCount);
		hipMemcpy(h_FilterGradients[i], L_Filter_BACKPROP_RESULTS[i], filterByteCount, hipMemcpyHostToDevice);


		hipMalloc(&h_V_Matricies[i], filterByteCount);
		hipMemcpy(h_V_Matricies[i], L_AdamOptimizer_V_Matrix[i], filterByteCount, hipMemcpyHostToDevice);


		hipMalloc(&h_S_Matricies[i], filterByteCount);
		hipMemcpy(h_S_Matricies[i], L_AdamOptimizer_S_Matrix[i], filterByteCount, hipMemcpyHostToDevice);


		hipMalloc(&h_V_CORRECTED_Matricies[i], filterByteCount);
		hipMemcpy(h_V_CORRECTED_Matricies[i], L_AdamOptimizer_Corrected_V_Matrix[i], filterByteCount, hipMemcpyHostToDevice);


		hipMalloc(&h_S_CORRECTED_Matricies[i], filterByteCount);
		hipMemcpy(h_S_CORRECTED_Matricies[i], L_AdamOptimizer_Corrected_S_Matrix[i], filterByteCount, hipMemcpyHostToDevice);

	}


	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_FilterPointers, h_Filters, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);

	hipMemcpy(d_FilterGradientPointers, h_FilterGradients, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);

	hipMemcpy(d_V_Matricies, h_V_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);

	hipMemcpy(d_S_Matricies, h_S_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);

	hipMemcpy(d_V_CORRECTED_Matricies, h_V_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);

	hipMemcpy(d_S_CORRECTED_Matricies, h_S_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);



	dim3 blockGrid(numberOfBlocks_X, 1, 1); ///OK
	dim3 threads(numberOfThreadsPerBlock, 1, 1); ///OK

	TransposeConvFilterUpdateKernel << <blockGrid, threads >> > (d_FilterPointers, d_FilterGradientPointers, d_V_Matricies, d_S_Matricies, d_V_CORRECTED_Matricies, d_S_CORRECTED_Matricies, m_FilterSize, m_HyperParam_Beta1, m_HyperParam_Beta2, m_HyperParam_T, m_HyperParam_alpha, m_HyperParam_Epsilon);

	hipDeviceSynchronize();

	float* temp = new float[filterByteCount];

	hipMemcpy(h_Filters, d_FilterPointers, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_FilterGradients, d_FilterGradientPointers, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_V_Matricies, d_V_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_S_Matricies, d_S_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_V_CORRECTED_Matricies, d_V_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_S_CORRECTED_Matricies, d_S_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {

		hipMemcpy(temp, h_Filters[i], filterByteCount, hipMemcpyDeviceToHost);
		memcpy(L_Filters[i], temp, filterByteCount);
		hipFree(h_Filters[i]);

		hipMemcpy(temp, h_V_Matricies[i], filterByteCount, hipMemcpyDeviceToHost);
		memcpy(L_AdamOptimizer_V_Matrix[i], temp, filterByteCount);
		hipFree(h_V_Matricies[i]);

		hipMemcpy(temp, h_S_Matricies[i], filterByteCount, hipMemcpyDeviceToHost);
		memcpy(L_AdamOptimizer_S_Matrix[i], temp, filterByteCount);
		hipFree(h_S_Matricies[i]);

		hipMemcpy(temp, h_V_CORRECTED_Matricies[i], filterByteCount, hipMemcpyDeviceToHost);
		memcpy(L_AdamOptimizer_Corrected_V_Matrix[i], temp, filterByteCount);
		hipFree(h_V_CORRECTED_Matricies[i]);

		hipMemcpy(temp, h_S_CORRECTED_Matricies[i], filterByteCount, hipMemcpyDeviceToHost);
		memcpy(L_AdamOptimizer_Corrected_S_Matrix[i], temp, filterByteCount);
		hipFree(h_S_CORRECTED_Matricies[i]);
	}

	hipFree(d_FilterPointers);
	hipFree(d_FilterGradientPointers);
	hipFree(d_V_Matricies);
	hipFree(d_S_Matricies);
	hipFree(d_V_CORRECTED_Matricies);
	hipFree(d_S_CORRECTED_Matricies);

	delete[] h_Filters;
	delete[] h_FilterGradients;
	delete[] h_V_Matricies;
	delete[] h_S_Matricies;
	delete[] h_V_CORRECTED_Matricies;
	delete[] h_S_CORRECTED_Matricies;
}

void TransposeConvolution::LayerBiasUpdate()
{
	int biasMatrixSize = L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH;

	size_t biasMatrixByteCount = biasMatrixSize * sizeof(float);


	/*int numberOfBlocks_X = L_NumberOf_FILTERS;
	int numberOfThreadsPerBlock = 1;*/

	int numberOfBlocks_X = L_BACKWARD_NumberOf_INPUTS;
	int numberOfThreadsPerBlock = L_BACKWARD_NumberOf_OUTPUTS;

	// create intermediate host array for storage of device row-pointers

	// create top-level device array pointer
	float** h_Biases = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_BiasGradients = new float* [L_BACKWARD_NumberOf_INPUTS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_V_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_S_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_V_CORRECTED_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));
	float** h_S_CORRECTED_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(float*));


	float** d_BiasPointers;
	hipMalloc((void**)&d_BiasPointers, L_NumberOf_FILTERS * sizeof(float*));

	float** d_BiasGradientPointers;
	hipMalloc((void**)&d_BiasGradientPointers, L_BACKWARD_NumberOf_INPUTS * sizeof(float*));

	float** d_V_Matricies;
	hipMalloc((void**)&d_V_Matricies, L_NumberOf_FILTERS * sizeof(float*));

	float** d_S_Matricies;
	hipMalloc((void**)&d_S_Matricies, L_NumberOf_FILTERS * sizeof(float*));

	float** d_V_CORRECTED_Matricies;
	hipMalloc((void**)&d_V_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*));

	float** d_S_CORRECTED_Matricies;
	hipMalloc((void**)&d_S_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*));

	for (int i = 0; i < L_BACKWARD_NumberOf_INPUTS; ++i)
	{
		hipMalloc(&h_BiasGradients[i], biasMatrixByteCount);
		hipMemcpy(h_BiasGradients[i], L_BACKWARD_Pass_INPUTS[i], biasMatrixByteCount, hipMemcpyHostToDevice);
	}


	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Biases[i], biasMatrixByteCount);
		hipMemcpy(h_Biases[i], L_Biases[i], biasMatrixByteCount, hipMemcpyHostToDevice);

		L_PrevBiases[i] = new float[biasMatrixSize];
		memcpy(L_PrevBiases[i], L_Biases[i], biasMatrixByteCount);

		hipMalloc(&h_V_Matricies[i], biasMatrixByteCount);
		hipMemcpy(h_V_Matricies[i], L_BIAS_AdamOptimizer_V_Matrix[i], biasMatrixByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_S_Matricies[i], biasMatrixByteCount);
		hipMemcpy(h_S_Matricies[i], L_BIAS_AdamOptimizer_S_Matrix[i], biasMatrixByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_V_CORRECTED_Matricies[i], biasMatrixByteCount);
		hipMemcpy(h_V_CORRECTED_Matricies[i], L_BIAS_AdamOptimizer_Corrected_V_Matrix[i], biasMatrixByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_S_CORRECTED_Matricies[i], biasMatrixByteCount);
		hipMemcpy(h_S_CORRECTED_Matricies[i], L_BIAS_AdamOptimizer_Corrected_S_Matrix[i], biasMatrixByteCount, hipMemcpyHostToDevice);
	}


	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_BiasPointers, h_Biases, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_BiasGradientPointers, h_BiasGradients, L_BACKWARD_NumberOf_INPUTS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_V_Matricies, h_V_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_S_Matricies, h_S_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_V_CORRECTED_Matricies, h_V_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d_S_CORRECTED_Matricies, h_S_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);


	dim3 blockGrid(numberOfBlocks_X, 1, 1); ///OK
	dim3 threads(numberOfThreadsPerBlock, 1, 1); ///OK

	TransposeConvBiasUpdateKernel << <blockGrid, threads >> > (d_BiasPointers, d_BiasGradientPointers, d_V_Matricies, d_S_Matricies, d_V_CORRECTED_Matricies, d_S_CORRECTED_Matricies, L_FORWARD_OutputLayer_HEIGHT, L_FORWARD_OutputLayer_WIDTH, m_HyperParam_Beta1, m_HyperParam_Beta2, m_HyperParam_T, m_HyperParam_alpha, m_HyperParam_Epsilon);
	hipDeviceSynchronize();

	float* temp = new float[biasMatrixByteCount];

	hipMemcpy(h_Biases, d_BiasPointers, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_BiasGradients, d_BiasGradientPointers, L_BACKWARD_NumberOf_INPUTS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_V_Matricies, d_V_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_S_Matricies, d_S_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_V_CORRECTED_Matricies, d_V_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);
	hipMemcpy(h_S_CORRECTED_Matricies, d_S_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyDeviceToHost);

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {

		hipMemcpy(L_Biases[i], h_Biases[i], biasMatrixByteCount, hipMemcpyDeviceToHost);
		//memcpy(L_Biases[i], temp, biasMatrixByteCount);
		hipFree(h_Biases[i]);

		hipMemcpy(L_BIAS_AdamOptimizer_V_Matrix[i], h_V_Matricies[i], biasMatrixByteCount, hipMemcpyDeviceToHost);
		//memcpy(L_BIAS_AdamOptimizer_V_Matrix[i], temp, biasMatrixByteCount);
		hipFree(h_V_Matricies[i]);

		hipMemcpy(L_BIAS_AdamOptimizer_S_Matrix[i], h_S_Matricies[i], biasMatrixByteCount, hipMemcpyDeviceToHost);
		//memcpy(L_BIAS_AdamOptimizer_S_Matrix[i], temp, biasMatrixByteCount);
		hipFree(h_S_Matricies[i]);

		hipMemcpy(L_BIAS_AdamOptimizer_Corrected_V_Matrix[i], h_V_CORRECTED_Matricies[i], biasMatrixByteCount, hipMemcpyDeviceToHost);
		//memcpy(L_BIAS_AdamOptimizer_Corrected_V_Matrix[i], temp, biasMatrixByteCount);
		hipFree(h_V_CORRECTED_Matricies[i]);

		hipMemcpy(L_BIAS_AdamOptimizer_Corrected_S_Matrix[i], h_S_CORRECTED_Matricies[i], biasMatrixByteCount, hipMemcpyDeviceToHost);
		//memcpy(L_BIAS_AdamOptimizer_Corrected_S_Matrix[i], temp, biasMatrixByteCount);
		hipFree(h_S_CORRECTED_Matricies[i]);
	}

	hipFree(d_BiasPointers);
	hipFree(d_BiasGradientPointers);
	hipFree(d_V_Matricies);
	hipFree(d_S_Matricies);
	hipFree(d_V_CORRECTED_Matricies);
	hipFree(d_S_CORRECTED_Matricies);

	delete[] h_Biases;
	delete[] h_BiasGradients;
	delete[] h_V_Matricies;
	delete[] h_S_Matricies;
	delete[] h_V_CORRECTED_Matricies;
	delete[] h_S_CORRECTED_Matricies;
}


void TransposeConvolution::LayerFilterInitialization()
{
	std::random_device rd{};
	std::mt19937 gen{ rd() };
	std::normal_distribution<> distribution{ 1,2 };

	int filterElementCount = m_FilterSize * m_FilterSize;

	for (int filterNumber = 0; filterNumber < L_NumberOf_FILTERS; ++filterNumber)
	{
		L_Filters[filterNumber] = new float[filterElementCount];

		for (int i = 0; i < filterElementCount; ++i)
		{
			L_Filters[filterNumber][i] = i + 1; //distribution(gen);
		}
	}
}

void TransposeConvolution::LayerBiasInitialization()
{
	std::random_device rd{};
	std::mt19937 gen{ rd() };
	std::normal_distribution<> distribution{ 0,1 };

	int biasElementCount = L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH;

	L_Biases = new float* [L_NumberOf_FILTERS];

	for (int biasNumber = 0; biasNumber < L_NumberOf_FILTERS; ++biasNumber)
	{
		L_Biases[biasNumber] = new float[biasElementCount];

		for (int i = 0; i < biasElementCount; ++i)
		{
			L_Biases[biasNumber][i] = 1; //distribution(gen);
		}
	}
}


void TransposeConvolution::SetHyperParams(float _beta1, float _beta2, float _eps, int _t, float _alpha)
{
	m_HyperParam_Beta1 = _beta1;
	m_HyperParam_Beta2 = _beta2;
	m_HyperParam_Epsilon = _eps;
	m_HyperParam_T = _t;
	m_HyperParam_alpha = _alpha;
}

void TransposeConvolution::PrintLayerParams()
{
	cout << "====================================================" << endl;
	cout << "====== Transpose Convolution Layer Parameters ======" << endl;
	cout << "====================================================" << endl;
	cout << "Transpose Convolution: Layer " << layerID << " " << "Level " << levelID << endl;

	cout << endl;

	cout << "-- Forward Dimensions --" << endl;
	cout << "Forward Input Height: " << L_FORWARD_InputLayer_HEIGHT << " || Forward Output Height: " << L_FORWARD_OutputLayer_HEIGHT << endl;
	cout << "Forward Input Width: " << L_FORWARD_InputLayer_WIDTH << " || Forward Output Width: " << L_FORWARD_OutputLayer_WIDTH << endl;

	cout << endl;

	cout << "-- Backward Dimensions --" << endl;
	cout << "Backward Input Height: " << L_BACKWARD_InputLayer_HEIGHT << " || Forward Output Height: " << L_BACKWARD_OutputLayer_HEIGHT << endl;
	cout << "Backward Input Width: " << L_BACKWARD_InputLayer_WIDTH << " || Forward Output Width: " << L_BACKWARD_OutputLayer_WIDTH << endl;

	cout << endl;

	cout << "-- Feature map count --" << endl;
	cout << "Forward Input Count: " << L_FORWARD_NumberOf_INPUTS << " || Backward Input Count: " << L_BACKWARD_NumberOf_INPUTS << endl;
	cout << "Forward Output Count: " << L_FORWARD_NumberOf_OUTPUTS << " || Backward Output Count: " << L_BACKWARD_NumberOf_OUTPUTS << endl;

	cout << "===================================" << endl;
}

void TransposeConvolution::DebugPrintAll()
{
	int newLineCounter = 1;

	cout << "========================================================" << endl;
	cout << "============ Transpose Conv Debug Print All ============" << endl;
	cout << "========================================================" << endl;

	cout << "Squishy: " << endl;
	cout << "Layer ID: " << layerID << endl;
	cout << "Level ID: " << levelID << endl;
	cout << "Hyper parameters: " << endl;
	cout << "Beta 1: " << m_HyperParam_Beta1 << endl;
	cout << "Beta 2: " << m_HyperParam_Beta2 << endl;
	cout << "Epsilon: " << m_HyperParam_Epsilon << endl;
	cout << "Alpha: " << m_HyperParam_alpha << endl;
	cout << "T: " << m_HyperParam_T << endl;


	cout << ">>>> Forward Inputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_FORWARD_NumberOf_INPUTS; ++inputIndex)
	{
		cout << "--- Element " << inputIndex + 1 << "---" << endl;
		for (int elementIndex = 0; elementIndex < L_FORWARD_InputLayer_HEIGHT * L_FORWARD_InputLayer_WIDTH; ++elementIndex)
		{
			cout << L_FORWARD_Pass_INPUTS[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == L_FORWARD_InputLayer_WIDTH + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	
	cout << ">>>> Normal Filter Inputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_Filters[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}
	
	cout << ">>>> Forward Outputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_FORWARD_NumberOf_OUTPUTS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH; ++elementIndex)
		{
			cout << L_FORWARD_Pass_OUTPUTS[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == L_FORWARD_OutputLayer_WIDTH + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Backward Inputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_BACKWARD_NumberOf_INPUTS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < L_BACKWARD_InputLayer_HEIGHT * L_BACKWARD_InputLayer_WIDTH; ++elementIndex)
		{
			cout << L_BACKWARD_Pass_INPUTS[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == L_BACKWARD_InputLayer_WIDTH + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}
	
	cout << ">>>> Backward Outputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_BACKWARD_NumberOf_OUTPUTS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < L_BACKWARD_OutputLayer_HEIGHT * L_BACKWARD_OutputLayer_WIDTH; ++elementIndex)
		{
			cout << L_BACKWARD_Pass_OUTPUTS[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == L_BACKWARD_OutputLayer_WIDTH + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Filter Backprop Outputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_Filter_BACKPROP_RESULTS[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	/*
	cout << ">>>> Bias Outputs Before Update <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH; ++elementIndex)
		{
			cout << L_PrevBiases[inputIndex][0] << " ";
			newLineCounter++;
			if (newLineCounter == L_FORWARD_OutputLayer_WIDTH + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}


	cout << ">>>> Bias Outputs After Update <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH; ++elementIndex)
		{
			cout << L_Biases[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == L_FORWARD_OutputLayer_WIDTH + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Filter Adam Optimizer V Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_AdamOptimizer_V_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Filter Adam Optimizer >CORRECTED< V Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_AdamOptimizer_Corrected_V_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Filter Adam Optimizer S Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_AdamOptimizer_S_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Filter Adam Optimizer >CORRECTED< S Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_AdamOptimizer_Corrected_S_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> BIAS Adam Optimizer V Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_BIAS_AdamOptimizer_V_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> BIAS Adam Optimizer >CORRECTED< V Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_BIAS_AdamOptimizer_Corrected_V_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> BIAS Adam Optimizer S Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_BIAS_AdamOptimizer_S_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> BIAS Adam Optimizer >CORRECTED< S Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_BIAS_AdamOptimizer_Corrected_S_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}*/

}


void TransposeConvolution::BackwardPass(float* backpropInput, int backPassHeight, int backPassWidth)
{
	
}

void TransposeConvolution::ForwardPass(float* forwardPassInput, int fwdPassHeight, int fwdPassWidth)
{

}

void TransposeConvolution::UpdateModule()
{
	
}


