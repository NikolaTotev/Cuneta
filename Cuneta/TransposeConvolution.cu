#include "hip/hip_runtime.h"
#include <hip/hip_vector_types.h>

#ifndef __HIPCC__  
#define __HIPCC__
#endif

#ifndef  __CUDART_API_PER_THREAD_DEFAULT_STREAM
#define  __CUDART_API_PER_THREAD_DEFAULT_STREAM
#endif
#include <hip/hip_runtime_api.h>


#ifndef __CUDACC_RTC__ 
#define __CUDACC_RTC__

#endif

#include <algorithm>
#include <iostream>

#include ""
#include "TransposeConvolution.cuh"
#include <random>
#include <cmath>

#ifndef __HIPCC__  
#define __HIPCC__
#endif

#ifndef __CUDACC_RTC__ 
#define __CUDACC_RTC__
#endif

#include <hip/device_functions.h>
using namespace std;

__global__ void TransposeConvolutionKernel(float* d_Input, float* d_Filter, float* d_Output, int _outputWidth, int _convolutionInputWidth, int filterHeight, int filterWidth)
{
	//These define indecies for output matrix;
	int outputRowIndex = blockIdx.x;
	int outputColumnIndex = threadIdx.x;

	//Starts from "top left" of current block of pixels being processed
	int inputRowIndex = blockIdx.x;
	int inputColumnIndex = threadIdx.x;

	int outputArrayIndex = outputRowIndex * _outputWidth + outputColumnIndex;

	int inputArrayIndex = 0;

	float result = 0;
	int filterIndex = 0;
	int temp = 0;
	for (int row = 0; row < filterHeight; row++)
	{
		inputColumnIndex = threadIdx.x;


		for (int col = 0; col < filterWidth; col++)
		{

			inputArrayIndex = inputRowIndex * _convolutionInputWidth + inputColumnIndex;

			result += d_Input[inputArrayIndex] * d_Filter[filterIndex];
			filterIndex++;
			inputColumnIndex += 1;
		}
		inputRowIndex += 1;
	}

	d_Output[outputArrayIndex] = result;
};

__global__ void LayerTransposeConvolutionKernel(float** _inputs, float** _filters, float** _outputs, float** _biases, int _numberOfOutputs, int _outputWidth, int _inputWidth, int _filterSize)
{
	int inputSelectionIndex = threadIdx.x;
	int filterSelectionIndex = blockIdx.z * _numberOfOutputs + threadIdx.x;
	int biasSelectionIndex = blockIdx.z * _numberOfOutputs + threadIdx.x;
	int outputSelectionIndex = blockIdx.z;

	float* selectedInput = _inputs[inputSelectionIndex];
	float* selectedFilter = _filters[filterSelectionIndex];
	float* selectedBias = _biases[filterSelectionIndex];
	float* selectedOutput = _outputs[outputSelectionIndex];

	int inputStartReadRowIndex = blockIdx.x;
	int inputStartReadColumnIndex = blockIdx.y;

	int outputWriteRowIndex = blockIdx.x;
	int outputWriteColumnIndex = blockIdx.y;

	int inputArrayIndex = 0;

	int outputArrayIndex = outputWriteRowIndex * _outputWidth + outputWriteColumnIndex;

	float result = 0;
	int filterIndex = 0;
	int temp = 0;
	for (int row = 0; row < _filterSize; row++)
	{
		inputStartReadColumnIndex = blockIdx.y;

		for (int col = 0; col < _filterSize; col++)
		{
			inputArrayIndex = inputStartReadRowIndex * _inputWidth + inputStartReadColumnIndex;

			result += selectedInput[inputArrayIndex] * selectedFilter[filterIndex];
			filterIndex++;
			inputStartReadColumnIndex += 1;
		}
		inputStartReadRowIndex += 1;
	}

	result += selectedBias[outputArrayIndex];
	atomicAdd(&selectedOutput[outputArrayIndex], result);
};

__global__ void LayerTransposeConvolutionBackPropKernel(float** _inputs, float** _filters, float** _outputs, int _numberOfOutputs, int _outputWidth, int _inputWidth, int _filterSize)
{
	int inputSelectionIndex = threadIdx.x;
	int filterSelectionIndex = blockIdx.z * _numberOfOutputs + threadIdx.x;
	int outputSelectionIndex = blockIdx.z;

	float* selectedInput = _inputs[inputSelectionIndex];
	float* selectedFilter = _filters[filterSelectionIndex];
	float* selectedOutput = _outputs[outputSelectionIndex];

	int inputStartReadRowIndex = blockIdx.x;
	int inputStartReadColumnIndex = blockIdx.y;

	int outputWriteRowIndex = blockIdx.x;
	int outputWriteColumnIndex = blockIdx.y;

	int inputArrayIndex = 0;

	int outputArrayIndex = outputWriteRowIndex * _outputWidth + outputWriteColumnIndex;

	float result = 0;
	int filterIndex = 0;
	int temp = 0;
	for (int row = 0; row < _filterSize; row++)
	{
		inputStartReadColumnIndex = blockIdx.y;

		for (int col = 0; col < _filterSize; col++)
		{
			inputArrayIndex = inputStartReadRowIndex * _inputWidth + inputStartReadColumnIndex;

			result += selectedInput[inputArrayIndex] * selectedFilter[filterIndex];
			filterIndex++;
			inputStartReadColumnIndex += 1;
		}
		inputStartReadRowIndex += 1;
	}

	atomicAdd(&selectedOutput[outputArrayIndex], result);
}

__global__ void PaddingKernel(float* d_UnpaddedInput, float* d_Output, int _paddedInputWidth, int _unpaddedInputWidth, int _unpaddedInputHeight)
{
	int rowWriteIndex = (blockIdx.x + 1) * 2 + threadIdx.x;
	int columnWriteIndex = 2;

	int inputRowReadIndex = (blockIdx.x * 2) + threadIdx.x;
	int inputColumnReadIndex = 0;

	int arrayPosition = rowWriteIndex * _paddedInputWidth + columnWriteIndex;
	int inputArrayPosition = inputRowReadIndex * _unpaddedInputWidth + inputColumnReadIndex;

	int var = 0;
	for (int i = 0; i < _unpaddedInputWidth; i++)
	{
		d_Output[arrayPosition] = d_UnpaddedInput[inputArrayPosition];
		arrayPosition++;
		inputArrayPosition++;
	}
}

__global__ void LayerTransposeInputPaddingKernel(float** __inputs, float** _outputs, int _unpaddedWidth, int _paddedWidth)
{
	int inputSelectionIndex = blockIdx.y;
	int outputSelectionIndex = blockIdx.y;

	float* selectedInput = __inputs[inputSelectionIndex];
	float* selectedOutput = _outputs[outputSelectionIndex];

	int rowWriteIndex = (blockIdx.x + 1) * 2 + threadIdx.x;
	int columnWriteIndex = 2;

	int inputRowReadIndex = (blockIdx.x * 2) + threadIdx.x;
	int inputColumnReadIndex = 0;

	int arrayPosition = rowWriteIndex * _paddedWidth + columnWriteIndex;
	int inputArrayPosition = inputRowReadIndex * _unpaddedWidth + inputColumnReadIndex;

	int var = 0;
	for (int i = 0; i < _unpaddedWidth; i++)
	{
		selectedOutput[arrayPosition] = selectedInput[inputArrayPosition];
		arrayPosition++;
		inputArrayPosition++;
	}
}

__global__ void LayerTransposeConvFilterFlipKernel(float** _inputFilters, float** _outputFilters, int _filterSize)
{
	float* filterToFlip = _inputFilters[blockIdx.x];
	int filterArraySize = _filterSize * _filterSize;
	float* flippedOutput = _outputFilters[blockIdx.x];
	int k = 0;

	//Loop from back and assign value to new array
	for (int i = filterArraySize - 1; i >= 0; ) {
		flippedOutput[k++] = filterToFlip[i--];
	}
}

__global__ void LayerTransposeConvFilterBackpropKernel(float** _inputs, float** _outputs, float** _filterEquivalents, int _inputsWidth, int _outputsWidth, int _filterEquivsHeight, int _filterEquivsWidth, int _numberOfInputs)
{
	int inputSelectionIndex = threadIdx.x;
	int filterSelectionIndex = blockIdx.z;
	int outputSelectionIndex = blockIdx.z * _numberOfInputs + threadIdx.x;

	float* selectedInput = _inputs[inputSelectionIndex];
	float* selectedFilter = _filterEquivalents[filterSelectionIndex];
	float* selectedOutput = _outputs[outputSelectionIndex];

	int inputStartReadRowIndex = blockIdx.x;
	int inputStartReadColumnIndex = blockIdx.y;

	int outputWriteRowIndex = blockIdx.x;
	int outputWriteColumnIndex = blockIdx.y;

	int inputArrayIndex = 0;

	int outputArrayIndex = outputWriteRowIndex * _outputsWidth + outputWriteColumnIndex;

	float result = 0;
	int filterIndex = 0;

	for (int row = 0; row < _filterEquivsHeight; row++)
	{
		inputStartReadColumnIndex = blockIdx.y;

		for (int col = 0; col < _filterEquivsWidth; col++)
		{
			inputArrayIndex = inputStartReadRowIndex * _inputsWidth + inputStartReadColumnIndex;

			result += selectedInput[inputArrayIndex] * selectedFilter[filterIndex];
			filterIndex++;
			inputStartReadColumnIndex += 1;
		}
		inputStartReadRowIndex += 1;
	}

	selectedOutput[outputArrayIndex] = result;
}

__global__ void TransposeConvFilterUpdateKernel(float** _currentFilters, float** _filterGradients, float** _VMatricies, float** _SMatricies, float** _V_CorrectedMatrices, float** _S_CorrectedMatricies, int _filterSize, int _HyperParam_Beta1, int _HyperParam_Beta2, int _HyperParam_T, int _HyperParam_alpha, int _HyperParam_Epsilon)
{
	float* selectedFilter = _currentFilters[blockIdx.x];
	float* selectedGradient = _filterGradients[blockIdx.x];
	float* selected_V_Matrix = _VMatricies[blockIdx.x];
	float* selected_S_Matrix = _SMatricies[blockIdx.x];
	float* selected_Corrected_V_Matrix = _V_CorrectedMatrices[blockIdx.x];
	float* selected_Corrected_S_Matrix = _S_CorrectedMatricies[blockIdx.x];

	for (int rowIndex = 0; rowIndex < _filterSize; ++rowIndex)
	{
		for (int columnIndex = 0; columnIndex < _filterSize; ++columnIndex)
		{
			int index = rowIndex * _filterSize + columnIndex;

			float filterBackpropValue = selectedGradient[index];
			float oldV = selected_V_Matrix[index];
			float oldS = selected_S_Matrix[index];

			float newV = _HyperParam_Beta1 * oldV + (1 - _HyperParam_Beta1) * filterBackpropValue;
			float newS = _HyperParam_Beta2 * oldS + (1 - _HyperParam_Beta2) * filterBackpropValue;

			float newVCorrected = newV / (1 - pow(_HyperParam_Beta1, _HyperParam_T));
			float newSCorrected = newS / (1 - pow(_HyperParam_Beta2, _HyperParam_T));

			selected_V_Matrix[index] = newV;
			selected_S_Matrix[index] = newS;

			selected_Corrected_V_Matrix[index] = newVCorrected;
			selected_Corrected_S_Matrix[index] = newSCorrected;

			float oldFilterValue = selectedFilter[index];
			float newF = oldFilterValue - _HyperParam_alpha * (newVCorrected / sqrt(newSCorrected + _HyperParam_Epsilon));

			selectedFilter[index] = newF;
		}
	}
}

__global__ void TransposeConvBiasUpdateKernel(float** _currentFilters, float** _filterGradients, float** _VMatricies, float** _SMatricies, float** _V_CorrectedMatrices, float** _S_CorrectedMatricies, int _height, int _width, int _HyperParam_Beta1, int _HyperParam_Beta2, int _HyperParam_T, int _HyperParam_alpha, int _HyperParam_Epsilon)
{
	float* selectedFilter = _currentFilters[blockIdx.x];
	float* selectedGradient = _filterGradients[blockIdx.x];
	float* selected_V_Matrix = _VMatricies[blockIdx.x];
	float* selected_S_Matrix = _SMatricies[blockIdx.x];
	float* selected_Corrected_V_Matrix = _V_CorrectedMatrices[blockIdx.x];
	float* selected_Corrected_S_Matrix = _S_CorrectedMatricies[blockIdx.x];

	for (int rowIndex = 0; rowIndex < _height; ++rowIndex)
	{
		for (int columnIndex = 0; columnIndex < _width; ++columnIndex)
		{
			int index = rowIndex * _width + columnIndex;

			float filterBackpropValue = selectedGradient[index];
			float oldV = selected_V_Matrix[index];
			float oldS = selected_S_Matrix[index];

			float newV = _HyperParam_Beta1 * oldV + (1 - _HyperParam_Beta1) * filterBackpropValue;
			float newS = _HyperParam_Beta2 * oldS + (1 - _HyperParam_Beta2) * filterBackpropValue;

			float newVCorrected = newV / (1 - pow(_HyperParam_Beta1, _HyperParam_T));
			float newSCorrected = newS / (1 - pow(_HyperParam_Beta2, _HyperParam_T));

			selected_V_Matrix[index] = newV;
			selected_S_Matrix[index] = newS;

			selected_Corrected_V_Matrix[index] = newVCorrected;
			selected_Corrected_S_Matrix[index] = newSCorrected;

			float oldFilterValue = selectedFilter[index];
			float newF = oldFilterValue - _HyperParam_alpha * (newVCorrected / sqrt(newSCorrected + _HyperParam_Epsilon));

			selectedFilter[index] = newF;
		}
	}
}


TransposeConvolution::TransposeConvolution(int _filterSize, int _paddingSize, int _numberOfInputs, int _numberOfOutputs, int _inputHeight, int _inputWidth)
{
	m_FilterSize = _filterSize;
	m_PaddingSize = _paddingSize;

	L_FORWARD_NumberOf_INPUTS = _numberOfInputs;
	L_FORWARD_NumberOf_OUTPUTS = _numberOfOutputs;

	L_BACKWARD_NumberOf_INPUTS = L_FORWARD_NumberOf_OUTPUTS;
	L_BACKWARD_NumberOf_OUTPUTS = L_FORWARD_NumberOf_INPUTS;

	L_FORWARD_InputLayer_HEIGHT = _inputHeight;
	L_FORWARD_InputLayer_WIDTH = _inputWidth;

	L_FORWARD_OutputLayer_HEIGHT = _inputHeight  +2;
	L_FORWARD_OutputLayer_WIDTH = _inputWidth  +2;

	L_BACKWARD_InputLayer_HEIGHT = L_FORWARD_OutputLayer_HEIGHT;
	L_BACKWARD_InputLayer_WIDTH = L_FORWARD_OutputLayer_WIDTH;

	L_BACKWARD_OutputLayer_HEIGHT = L_FORWARD_InputLayer_HEIGHT;
	L_BACKWARD_OutputLayer_WIDTH = L_FORWARD_InputLayer_WIDTH;

	L_FORWARD_Pass_INPUTS = new float* [L_FORWARD_NumberOf_INPUTS];
	L_FORWARD_Pass_PADDED_INPUTS = new float* [L_FORWARD_NumberOf_INPUTS];
	L_FORWARD_Pass_OUTPUTS = new float* [L_FORWARD_NumberOf_OUTPUTS];

	L_BACKWARD_Pass_INPUTS = new float* [L_BACKWARD_NumberOf_INPUTS];
	L_BACKWARD_Pass_OUTPUTS = new float* [L_BACKWARD_NumberOf_OUTPUTS];

	L_NumberOf_FILTERS = L_FORWARD_NumberOf_INPUTS * L_FORWARD_NumberOf_OUTPUTS;

	L_Filters = new float* [L_NumberOf_FILTERS];
	L_FLIPPED_Filters = new float* [L_NumberOf_FILTERS];
	L_Filter_BACKPROP_RESULTS = new float* [L_NumberOf_FILTERS];

	L_Baises = new float*[L_NumberOf_FILTERS];
	L_PrevBiases = new float* [L_NumberOf_FILTERS];

	L_AdamOptimizer_V_Matrix = new float* [L_NumberOf_FILTERS];
	L_AdamOptimizer_S_Matrix = new float* [L_NumberOf_FILTERS];
	L_AdamOptimizer_Corrected_V_Matrix = new float* [L_NumberOf_FILTERS];
	L_AdamOptimizer_Corrected_S_Matrix = new float* [L_NumberOf_FILTERS];

	L_BIAS_AdamOptimizer_V_Matrix = new float* [L_NumberOf_FILTERS];
	L_BIAS_AdamOptimizer_S_Matrix = new float* [L_NumberOf_FILTERS];
	L_BIAS_AdamOptimizer_Corrected_V_Matrix = new float* [L_NumberOf_FILTERS];
	L_BIAS_AdamOptimizer_Corrected_S_Matrix = new float* [L_NumberOf_FILTERS];
	

	for (int i = 0; i < L_NumberOf_FILTERS; ++i)
	{
		L_AdamOptimizer_V_Matrix[i] = new float[m_FilterSize * m_FilterSize];
		L_AdamOptimizer_S_Matrix[i] = new float[m_FilterSize * m_FilterSize];
		L_AdamOptimizer_Corrected_V_Matrix[i] = new float[m_FilterSize * m_FilterSize];
		L_AdamOptimizer_Corrected_S_Matrix[i] = new float[m_FilterSize * m_FilterSize];

		size_t byteCount = m_FilterSize * m_FilterSize * sizeof(float);
		memset(L_AdamOptimizer_V_Matrix[i], 0, byteCount);
		memset(L_AdamOptimizer_S_Matrix[i], 0, byteCount);
		memset(L_AdamOptimizer_Corrected_V_Matrix[i], 0, byteCount);
		memset(L_AdamOptimizer_Corrected_S_Matrix[i], 0, byteCount);

		L_Biases = new float* [L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH];
		L_BIAS_AdamOptimizer_V_Matrix[i] = new float[L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH];
		L_BIAS_AdamOptimizer_S_Matrix[i] = new float[L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH];
		L_BIAS_AdamOptimizer_Corrected_V_Matrix[i] = new float[L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH];
		L_BIAS_AdamOptimizer_Corrected_S_Matrix[i] = new float[L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH];

		size_t biasByteCount = L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH * sizeof(float);
		memset(L_BIAS_AdamOptimizer_V_Matrix[i], 0, biasByteCount);
		memset(L_BIAS_AdamOptimizer_S_Matrix[i], 0, biasByteCount);
		memset(L_BIAS_AdamOptimizer_Corrected_V_Matrix[i], 0, biasByteCount);
		memset(L_BIAS_AdamOptimizer_Corrected_S_Matrix[i], 0, biasByteCount);
	}

	InitializeFilter();
}


void TransposeConvolution::ForwardPass(float* forwardPassInput, int fwdPassHeight, int fwdPassWidth)
{

	m_InputMatrixHeight = fwdPassHeight;
	m_InputMatrixWidth = fwdPassWidth;

	m_OutputMatrixHeight = m_InputMatrixHeight + 2;
	m_OutputMatrixWidth = m_InputMatrixWidth + 2;

	int arrayLength = fwdPassHeight * fwdPassWidth;
	size_t inputSize = arrayLength * sizeof(float);

	m_InputMatrix = new float[arrayLength];
	m_OutputMatrix = new float[m_OutputMatrixHeight * m_OutputMatrixWidth];

	memcpy(m_InputMatrix, forwardPassInput, inputSize);

	PadInput();

	int rowShifts = m_OutputMatrixHeight;
	int columnShifts = m_OutputMatrixWidth;

	int elementsInPaddedInput = m_PaddedInputHeight * m_PaddedInputWidth;
	int elementsInOutput = m_OutputMatrixHeight * m_OutputMatrixWidth;


	dim3 blockGrid(rowShifts, 1, 1);
	dim3 threads(columnShifts, 1, 1);

	size_t paddedInputElementCount = m_PaddedInputHeight * m_PaddedInputWidth;
	size_t filterMatrixElementCount = m_FilterSize * m_FilterSize;
	size_t outputElementCount = m_OutputMatrixHeight * m_OutputMatrixWidth;

	int paddedInputByteCount = paddedInputElementCount * sizeof(float);
	int filterByteCount = filterMatrixElementCount * sizeof(float);
	int outputByteCount = outputElementCount * sizeof(float);

	//Define pointers for deviceMemory locations
	float* d_Input;
	float* d_Filter;
	float* d_Output;


	//Allocate memory
	hipMalloc((void**)&d_Input, paddedInputByteCount);
	hipMalloc((void**)&d_Filter, filterByteCount);
	hipMalloc((void**)&d_Output, outputByteCount);


	//Copy m_Filter into global device memory m_InputMatrix -> d_Input
	hipMemcpy(d_Input, m_PaddedInput, paddedInputByteCount, hipMemcpyHostToDevice);
	hipMemcpy(d_Filter, m_Filter, filterByteCount, hipMemcpyHostToDevice);

	TransposeConvolutionKernel << <blockGrid, threads >> > (d_Input, d_Filter, d_Output, m_OutputMatrixWidth, m_PaddedInputWidth, m_FilterSize, m_FilterSize);
	hipDeviceSynchronize();

	hipMemcpy(m_OutputMatrix, d_Output, outputByteCount, hipMemcpyDeviceToHost);
}

void TransposeConvolution::LayerForwardPass(float** _inputs)
{
	int inputSize = L_FORWARD_InputLayer_HEIGHT* L_FORWARD_InputLayer_WIDTH;
	
	int filterSize = m_FilterSize * m_FilterSize;

	int biasSize = L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH;

	int outputSize = L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH;

	size_t inputByteCount = inputSize * sizeof(float);
	size_t filterByteCount = filterSize * sizeof(float);
	size_t biasByteCount = biasSize * sizeof(float);
	size_t outputByteCount = outputSize * sizeof(float);

	for (int inputNumber = 0; inputNumber < L_FORWARD_NumberOf_INPUTS; ++inputNumber)
	{
		L_FORWARD_Pass_INPUTS[inputNumber] = new float[inputSize];
		memcpy(L_FORWARD_Pass_INPUTS[inputNumber], _inputs[inputNumber], inputByteCount);
	}
	
	for (int outputNumber = 0; outputNumber < L_FORWARD_NumberOf_OUTPUTS; ++outputNumber)
	{
		L_FORWARD_Pass_OUTPUTS[outputNumber] = new float[outputSize];
	}

	LayerPadInput();

	int paddedInputSize = L_FORWARD_InputLayer_PADDED_HEIGHT * L_FORWARD_InputLayer_PADDED_WIDTH;
	size_t paddedInputByteCount = paddedInputSize * sizeof(float);


	int numberOfBlockx_X = L_FORWARD_OutputLayer_HEIGHT;//L_FORWARD_OutputLayer_HEIGHT;
	int numberOfBlocks_Y = L_FORWARD_OutputLayer_WIDTH;//L_FORWARD_OutputLayer_WIDTH;
	int numberOfBlocks_Z = L_FORWARD_NumberOf_OUTPUTS;//L_FORWARD_NumberOf_OUTPUTS;
	int numberOfThreadsPerBlock = L_FORWARD_NumberOf_INPUTS;//L_FORWARD_NumberOf_INPUTS;

	float** h_Inputs = new float* [L_FORWARD_NumberOf_INPUTS];  //(float**)malloc(L_FORWARD_NumberOf_INPUTS * sizeof(int*));
	float** h_Filters = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(int*));
	float** h_Biases = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(int*));
	float** h_Outputs = new float* [L_FORWARD_NumberOf_OUTPUTS]; //(float**)malloc(L_FORWARD_NumberOf_OUTPUTS * sizeof(int*));

	float** d_BiasesPointerArray;
	hipMalloc((void**)&d_BiasesPointerArray, L_NumberOf_FILTERS * sizeof(int*));

	float** d_InputPointerArray;
	hipMalloc((void**)&d_InputPointerArray, L_FORWARD_NumberOf_INPUTS * sizeof(int*));

	float** d_FilterPointerArray;
	hipMalloc((void**)&d_FilterPointerArray, L_NumberOf_FILTERS * sizeof(int*));

	float** d_OutputPointerArray;
	hipMalloc((void**)&d_OutputPointerArray, L_FORWARD_NumberOf_OUTPUTS * sizeof(int*));

	float* something = new float[paddedInputSize];
	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_FORWARD_NumberOf_INPUTS; i++) {
		hipMalloc(&h_Inputs[i], paddedInputByteCount);
		memcpy(something, L_FORWARD_Pass_PADDED_INPUTS[i], paddedInputByteCount);
		hipMemcpy(h_Inputs[i], L_FORWARD_Pass_PADDED_INPUTS[i], paddedInputByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Filters[i], filterByteCount);
		hipMemcpy(h_Filters[i], L_Filters[i], filterByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Biases[i], filterByteCount);
		hipMemcpy(h_Biases[i], L_Biases[i], biasByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_FORWARD_NumberOf_OUTPUTS; i++) {
		hipMalloc(&h_Outputs[i], outputByteCount);
		hipMemset(&h_Outputs[i], 0, outputByteCount);
	}

	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_InputPointerArray, h_Inputs, L_FORWARD_NumberOf_INPUTS * sizeof(int*), hipMemcpyHostToDevice);
	hipMemcpy(d_FilterPointerArray, h_Filters, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyHostToDevice);
	hipMemcpy(d_BiasesPointerArray, h_Biases, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyHostToDevice);

	hipMemcpy(d_OutputPointerArray, h_Outputs, L_FORWARD_NumberOf_OUTPUTS * sizeof(float*), hipMemcpyHostToDevice);

	dim3 blockGrid(numberOfBlockx_X, numberOfBlocks_Y, numberOfBlocks_Z); ///OK
	dim3 threads(numberOfThreadsPerBlock, 1, 1); ///OK

	LayerTransposeConvolutionKernel << <blockGrid, threads >> > (d_InputPointerArray, d_FilterPointerArray, d_OutputPointerArray,d_BiasesPointerArray , L_FORWARD_NumberOf_OUTPUTS, L_FORWARD_OutputLayer_WIDTH, L_FORWARD_InputLayer_PADDED_WIDTH, m_FilterSize);
	hipDeviceSynchronize();
	float* temp = new float[outputSize];
	hipMemcpy(h_Outputs, d_OutputPointerArray, L_FORWARD_NumberOf_OUTPUTS * sizeof(int*), hipMemcpyDeviceToHost);
	for (size_t i = 0; i < L_FORWARD_NumberOf_OUTPUTS; i++) {

		hipMemcpy(temp, h_Outputs[i], outputByteCount, hipMemcpyDeviceToHost);
		memcpy(L_FORWARD_Pass_OUTPUTS[i], temp, outputByteCount);
		hipFree(h_Outputs[i]);
	}
	hipFree(d_OutputPointerArray);
	delete[] h_Outputs;

	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_FORWARD_NumberOf_INPUTS; i++) {
		hipFree(&d_InputPointerArray[i]);
	}
	hipFree(d_InputPointerArray);
	delete[] h_Inputs;

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipFree(&d_FilterPointerArray[i]);
	}
	hipFree(d_FilterPointerArray);
	delete[] h_Filters;
}


void TransposeConvolution::BackwardPass(float* backpropInput, int backPassHeight, int backPassWidth)
{
	m_BackpropInputMatrixHeight = backPassHeight; ///OK
	m_BackpropInputMatrixWidth = backPassWidth; ///OK

	m_BackpropOutputMatrixHeight = m_BackpropInputMatrixHeight - 2; ///OK
	m_BackpropOutputMatrixWidth = m_BackpropInputMatrixWidth - 2; ///OK

	int arrayLength = backPassHeight * backPassWidth; ///OK
	size_t inputSize = arrayLength * sizeof(float); ///OK

	m_BackPropInputMatrix = new float[arrayLength]; ///OK
	m_BackpropagationOutput = new float[m_BackpropOutputMatrixHeight * m_BackpropOutputMatrixWidth]; ///OK

	memcpy(m_BackPropInputMatrix, backpropInput, inputSize); ///OK

	//Main backprop

	int rowShifts = m_BackpropOutputMatrixHeight; ///OK
	int columnShifts = m_BackpropOutputMatrixWidth; ///OK
	
	dim3 blockGrid(rowShifts, 1, 1); ///OK
	dim3 threads(columnShifts, 1, 1); ///OK

	size_t inputElementCount = m_BackpropInputMatrixHeight * m_BackpropInputMatrixWidth; ///OK
	size_t filterMatrixElementCount = m_FilterSize * m_FilterSize; ///OK
	size_t outputElementCount = m_BackpropOutputMatrixHeight * m_BackpropOutputMatrixWidth; ///OK

	int inputByteCount = inputElementCount * sizeof(float); ///OK
	int filterByteCount = filterMatrixElementCount * sizeof(float); ///OK
	int outputByteCount = outputElementCount * sizeof(float); ///OK

	FlipFilter();  ///OK

	//Define pointers for deviceMemory locations
	float* d_Input; ///OK
	float* d_Filter; ///OK
	float* d_Output; ///OK


	//Allocate memory
	hipMalloc((void**)&d_Input, inputByteCount); ///OK
	hipMalloc((void**)&d_Filter, filterByteCount); ///OK
	hipMalloc((void**)&d_Output, outputByteCount); ///OK


	//Copy m_Filter into global device memory m_InputMatrix -> d_Input
	hipMemcpy(d_Input, m_BackPropInputMatrix, inputByteCount, hipMemcpyHostToDevice); ///OK
	hipMemcpy(d_Filter, m_FlippedFilter, filterByteCount, hipMemcpyHostToDevice); ///OK

	TransposeConvolutionKernel << <blockGrid, threads >> > (d_Input, d_Filter, d_Output, m_BackpropOutputMatrixWidth, m_BackpropInputMatrixWidth, m_FilterSize, m_FilterSize); ///OK
	hipDeviceSynchronize(); ///OK

	hipMemcpy(m_BackpropagationOutput, d_Output, outputByteCount, hipMemcpyDeviceToHost); ///OK

	//Filter backprop
	FilterBackprop(backpropInput, backPassHeight, backPassWidth); ///OK

	hipFree(d_Input);
	hipFree(d_Filter);
	hipFree(d_Output);
}

void TransposeConvolution::LayerBackwardPass(float** _backpropInput)
{
	int inputSize = L_BACKWARD_InputLayer_HEIGHT * L_BACKWARD_InputLayer_WIDTH;

	int filterSize = m_FilterSize * m_FilterSize;

	int outputSize = L_BACKWARD_OutputLayer_HEIGHT * L_BACKWARD_OutputLayer_WIDTH;

	size_t inputByteCount = inputSize * sizeof(float);
	size_t filterByteCount = filterSize * sizeof(float);
	size_t outputByteCount = outputSize * sizeof(float);

	for (int inputNumber = 0; inputNumber < L_BACKWARD_NumberOf_INPUTS; ++inputNumber)
	{
		L_BACKWARD_Pass_INPUTS[inputNumber] = new float[inputSize];
		memcpy(L_BACKWARD_Pass_INPUTS[inputNumber], _backpropInput[inputNumber], inputByteCount);
	}

	for (int outputNumber = 0; outputNumber < L_BACKWARD_NumberOf_OUTPUTS; ++outputNumber)
	{
		L_BACKWARD_Pass_OUTPUTS[outputNumber] = new float[outputSize];
	}

	LayerFlipFilter();

	int numberOfBlockx_X = L_BACKWARD_OutputLayer_HEIGHT;
	int numberOfBlocks_Y = L_BACKWARD_OutputLayer_WIDTH;
	int numberOfBlocks_Z = L_BACKWARD_NumberOf_OUTPUTS;
	int numberOfThreadsPerBlock = L_BACKWARD_NumberOf_INPUTS;

	// create intermediate host array for storage of device row-pointers

	// create top-level device array pointer
	float** h_Inputs = new float* [L_BACKWARD_NumberOf_INPUTS];  //(float**)malloc(L_FORWARD_NumberOf_INPUTS * sizeof(int*));
	float** h_Filters = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(int*));
	float** h_Outputs = new float* [L_BACKWARD_NumberOf_OUTPUTS]; //(float**)malloc(L_FORWARD_NumberOf_OUTPUTS * sizeof(int*));

	float* d_Biases;
	hipMalloc((void**)&d_Biases, L_NumberOf_FILTERS * sizeof(float));


	float** d_InputPointerArray;
	hipMalloc((void**)&d_InputPointerArray, L_BACKWARD_NumberOf_INPUTS * sizeof(int*));

	float** d_FilterPointerArray;
	hipMalloc((void**)&d_FilterPointerArray, L_NumberOf_FILTERS * sizeof(int*));

	float** d_OutputPointerArray;
	hipMalloc((void**)&d_OutputPointerArray, L_BACKWARD_NumberOf_OUTPUTS * sizeof(int*));


	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_BACKWARD_NumberOf_INPUTS; i++) {
		hipMalloc(&h_Inputs[i], inputByteCount);
		hipMemcpy(h_Inputs[i], L_BACKWARD_Pass_INPUTS[i], inputByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Filters[i], filterByteCount);
		hipMemcpy(h_Filters[i], L_FLIPPED_Filters[i], filterByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_BACKWARD_NumberOf_OUTPUTS; i++) {
		hipMalloc(&h_Outputs[i], outputByteCount);
		hipMemset(&h_Outputs[i], 0, outputByteCount);
	}

	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_InputPointerArray, h_Inputs, L_BACKWARD_NumberOf_INPUTS * sizeof(int*), hipMemcpyHostToDevice);
	hipMemcpy(d_FilterPointerArray, h_Filters, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyHostToDevice);
	hipMemcpy(d_OutputPointerArray, h_Outputs, L_BACKWARD_NumberOf_OUTPUTS * sizeof(float*), hipMemcpyHostToDevice);

	dim3 blockGrid(numberOfBlockx_X, numberOfBlocks_Y, numberOfBlocks_Z); ///OK
	dim3 threads(numberOfThreadsPerBlock, 1, 1); ///OK

	LayerTransposeConvolutionBackPropKernel << <blockGrid, threads >> > (d_InputPointerArray, d_FilterPointerArray, d_OutputPointerArray, L_BACKWARD_NumberOf_INPUTS, L_BACKWARD_OutputLayer_WIDTH, L_BACKWARD_InputLayer_WIDTH, m_FilterSize);
	hipDeviceSynchronize();
	float* temp = new float[outputSize];
	hipMemcpy(h_Outputs, d_OutputPointerArray, L_BACKWARD_NumberOf_OUTPUTS * sizeof(int*), hipMemcpyDeviceToHost);
	for (size_t i = 0; i < L_BACKWARD_NumberOf_OUTPUTS; i++) {

		hipMemcpy(temp, h_Outputs[i], outputByteCount, hipMemcpyDeviceToHost);
		memcpy(L_BACKWARD_Pass_OUTPUTS[i], temp, outputByteCount);
		hipFree(h_Outputs[i]);
	}
	hipFree(d_OutputPointerArray);
	delete[] h_Outputs;

	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_BACKWARD_NumberOf_INPUTS; i++) {
		hipFree(&d_InputPointerArray[i]);
	}
	hipFree(d_InputPointerArray);
	delete[] h_Inputs;

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipFree(&d_FilterPointerArray[i]);
	}
	hipFree(d_FilterPointerArray);
	delete[] h_Filters;
}

void TransposeConvolution::LayerFilterBackprop()
{
	int inputSize = L_FORWARD_InputLayer_PADDED_WIDTH* L_FORWARD_InputLayer_PADDED_WIDTH;

	int filterEquivalentSize = L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH;

	int outputSize = m_FilterSize * m_FilterSize;

	size_t inputByteCount = inputSize * sizeof(float);
	size_t filterEquivalentByteCount = filterEquivalentSize * sizeof(float);
	size_t outputByteCount = outputSize * sizeof(float);


	for (int outputNumber = 0; outputNumber < L_NumberOf_FILTERS; ++outputNumber)
	{
		L_Filter_BACKPROP_RESULTS[outputNumber] = new float[outputSize];
	}


	int numberOfBlockx_X = m_FilterSize;
	int numberOfBlocks_Y = m_FilterSize;
	int numberOfBlocks_Z = L_BACKWARD_NumberOf_INPUTS;
	int numberOfThreadsPerBlock = L_FORWARD_NumberOf_INPUTS;

	// create intermediate host array for storage of device row-pointers

	// create top-level device array pointer
	float** h_Inputs = new float* [L_FORWARD_NumberOf_INPUTS];  //(float**)malloc(L_FORWARD_NumberOf_INPUTS * sizeof(int*));
	float** h_Filters = new float* [L_BACKWARD_NumberOf_INPUTS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(int*));
	float** h_Outputs = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_FORWARD_NumberOf_OUTPUTS * sizeof(int*));



	float** d_InputPointerArray;
	hipMalloc((void**)&d_InputPointerArray, L_FORWARD_NumberOf_INPUTS * sizeof(int*));

	float** d_FilterPointerArray;
	hipMalloc((void**)&d_FilterPointerArray, L_BACKWARD_NumberOf_INPUTS * sizeof(int*));

	float** d_OutputPointerArray;
	hipMalloc((void**)&d_OutputPointerArray, L_NumberOf_FILTERS * sizeof(int*));


	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_FORWARD_NumberOf_INPUTS; i++) {
		hipMalloc(&h_Inputs[i], inputByteCount);
		hipMemcpy(h_Inputs[i], L_FORWARD_Pass_PADDED_INPUTS[i], inputByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_BACKWARD_NumberOf_INPUTS; i++) {
		hipMalloc(&h_Filters[i], filterEquivalentByteCount);
		hipMemcpy(h_Filters[i], L_BACKWARD_Pass_INPUTS[i], filterEquivalentByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Outputs[i], outputByteCount);
		hipMemset(&h_Outputs[i], 0, outputByteCount);
	}

	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_InputPointerArray, h_Inputs, L_FORWARD_NumberOf_INPUTS * sizeof(int*), hipMemcpyHostToDevice);
	hipMemcpy(d_FilterPointerArray, h_Filters, L_BACKWARD_NumberOf_INPUTS * sizeof(int*), hipMemcpyHostToDevice);
	hipMemcpy(d_OutputPointerArray, h_Outputs, L_NumberOf_FILTERS * sizeof(float*), hipMemcpyHostToDevice);

	dim3 blockGrid(numberOfBlockx_X, numberOfBlocks_Y, numberOfBlocks_Z); ///OK
	dim3 threads(numberOfThreadsPerBlock, 1, 1); ///OK

	LayerTransposeConvFilterBackpropKernel << <blockGrid, threads >> > (d_InputPointerArray, d_OutputPointerArray, d_FilterPointerArray, L_FORWARD_InputLayer_PADDED_WIDTH, m_FilterSize, L_BACKWARD_InputLayer_HEIGHT, L_BACKWARD_InputLayer_WIDTH, L_FORWARD_NumberOf_INPUTS);
	hipDeviceSynchronize();

	float* temp = new float[outputSize];
	hipMemcpy(h_Outputs, d_OutputPointerArray, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyDeviceToHost);
	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {

		hipMemcpy(temp, h_Outputs[i], outputByteCount, hipMemcpyDeviceToHost);
		memcpy(L_Filter_BACKPROP_RESULTS[i], temp, outputByteCount);
		hipFree(h_Outputs[i]);
	}
	hipFree(d_OutputPointerArray);
	delete[] h_Outputs;

	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_FORWARD_NumberOf_INPUTS; i++) {
		hipFree(&d_InputPointerArray[i]);
	}
	hipFree(d_InputPointerArray);
	delete[] h_Inputs;

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipFree(&d_FilterPointerArray[i]);
	}
	hipFree(d_FilterPointerArray);
	delete[] h_Filters;
}

void TransposeConvolution::LayerFlipFilter()
{
	int inputSize = m_FilterSize * m_FilterSize;

	int outputSize = m_FilterSize * m_FilterSize;

	size_t inputByteCount = inputSize * sizeof(float);
	size_t outputByteCount = outputSize * sizeof(float);

	for (int filterNumber = 0; filterNumber < L_NumberOf_FILTERS; ++filterNumber)
	{
		L_FLIPPED_Filters[filterNumber] = new float[inputSize];
	}

	int numberOfBlockx_X = L_NumberOf_FILTERS;

	// create intermediate host array for storage of device row-pointers

	// create top-level device array pointer
	float** h_Inputs = new float* [L_NumberOf_FILTERS];

	float** h_Outputs = new float* [L_NumberOf_FILTERS];

	float** d_InputPointerArray;
	hipMalloc((void**)&d_InputPointerArray, L_NumberOf_FILTERS * sizeof(int*));


	float** d_OutputPointerArray;
	hipMalloc((void**)&d_OutputPointerArray, L_NumberOf_FILTERS * sizeof(int*));


	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Inputs[i], inputByteCount);
		hipMemcpy(h_Inputs[i], L_Filters[i], inputByteCount, hipMemcpyHostToDevice);
	}

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Outputs[i], outputByteCount);
	}

	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_InputPointerArray, h_Inputs, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyHostToDevice);
	hipMemcpy(d_OutputPointerArray, h_Outputs, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyHostToDevice);

	dim3 blockGrid(numberOfBlockx_X, 1, 1); ///OK
	dim3 threads(1, 1, 1); ///OK

	LayerTransposeConvFilterFlipKernel << <blockGrid, threads >> > (d_InputPointerArray, d_OutputPointerArray, m_FilterSize);
	hipDeviceSynchronize();

	float* temp = new float[outputSize];

	hipMemcpy(h_Outputs, d_OutputPointerArray, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyDeviceToHost);

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {

		hipMemcpy(temp, h_Outputs[i], outputByteCount, hipMemcpyDeviceToHost);
		memcpy(L_FLIPPED_Filters[i], temp, outputByteCount);
		hipFree(&h_Outputs[i]);
	}
	hipFree(d_OutputPointerArray);
	delete[] h_Outputs;

	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipFree(&d_InputPointerArray[i]);
	}
	hipFree(d_InputPointerArray);
	delete[] h_Inputs;

	delete[] temp;
}

void TransposeConvolution::FilterBackprop(float* backpropInput, int backPassHeight, int backPassWidth)
{
	size_t fwdInputElementCount = m_PaddedInputHeight * m_PaddedInputWidth;  ///OK
	size_t filterEqivElementCount = m_OutputMatrixHeight * m_OutputMatrixWidth; ///OK
	size_t  filterOutputElementCount = m_FilterSize * m_FilterSize; ///OK

	int fwdInputByteCount = fwdInputElementCount * sizeof(float); ///OK
	int filterEqivByteCount = filterEqivElementCount * sizeof(float); ///OK
	int filterOutputByteCount = filterOutputElementCount * sizeof(float); ///OK

	m_FilterBackpropResult = new float[filterOutputElementCount];

	float* d_FwdInput; ///OK
	float* d_FilterEquiv; ///OK
	float* d_FilterOutput; ///OK

	//Allocate memory
	hipMalloc((void**)&d_FwdInput, fwdInputByteCount); ///OK
	hipMalloc((void**)&d_FilterEquiv, filterEqivByteCount); ///OK
	hipMalloc((void**)&d_FilterOutput, filterOutputByteCount); ///OK

	//Copy m_Filter into global device memory m_InputMatrix -> d_Input
	hipMemcpy(d_FwdInput, m_PaddedInput, fwdInputByteCount, hipMemcpyHostToDevice); ///OK
	hipMemcpy(d_FilterEquiv, m_BackPropInputMatrix, filterEqivByteCount, hipMemcpyHostToDevice); ///OK

	int rowShifts = m_FilterSize; ///OK
	int columnShifts = m_FilterSize; ///OK

	dim3 blockGrid(rowShifts, 1, 1); ///OK
	dim3 threads(columnShifts, 1, 1); ///OK

	TransposeConvolutionKernel << <blockGrid, threads >> > (d_FwdInput, d_FilterEquiv, d_FilterOutput, m_FilterSize, m_PaddedInputWidth, m_OutputMatrixHeight, m_OutputMatrixWidth);
	hipDeviceSynchronize();

	hipMemcpy(m_FilterBackpropResult, d_FilterOutput, filterOutputByteCount, hipMemcpyDeviceToHost); ///OK
}


void TransposeConvolution::UpdateModule()
{
	for (int rowIndex = 0; rowIndex < m_FilterSize; ++rowIndex)
	{
		for (int columnIndex = 0; columnIndex < m_FilterSize; ++columnIndex)
		{
			int index = rowIndex * m_FilterSize + columnIndex;

			float filterBackpropValue = m_FilterBackpropResult[index];
			float oldV = m_AdamOptimizer_VMatrix[index];
			float oldS = m_AdamOptimizer_SMatrix[index];

			float newV = m_HyperParam_Beta1 * oldV + (1 - m_HyperParam_Beta1) * filterBackpropValue;
			float newS = m_HyperParam_Beta2 * oldS + (1 - m_HyperParam_Beta2) * filterBackpropValue;

			float newVCorrected = newV / (1 - pow(m_HyperParam_Beta1, m_HyperParam_T));
			float newSCorrected = newS / (1 - pow(m_HyperParam_Beta2, m_HyperParam_T));

			m_AdamOptimizer_VMatrix[index] = newV;
			m_AdamOptimizer_SMatrix[index] = newS;

			m_AdamOptimizer_Corrected_VMatrix[index] = newVCorrected;
			m_AdamOptimizer_Corrected_SMatrix[index] = newSCorrected;

			float oldFilterValue = m_Filter[index];
			float newF = oldFilterValue - m_HyperParam_alpha * (newVCorrected / sqrt(newSCorrected + m_HyperParam_Epsilon));

			m_Filter[index] = newF;
		}
	}
}

void TransposeConvolution::LayerUpdate()
{
	int filterSize = m_FilterSize * m_FilterSize;

	size_t filterByteCount = filterSize * sizeof(float);


	int numberOfBlocks_X = L_NumberOf_FILTERS;
	int numberOfThreadsPerBlock = 1;

	// create intermediate host array for storage of device row-pointers

	// create top-level device array pointer
	float** h_Filters = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(int*));
	float** h_FilterGradients = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(int*));
	float** h_V_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(int*));
	float** h_S_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(int*));
	float** h_V_CORRECTED_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(int*));
	float** h_S_CORRECTED_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(int*));


	float** d_FilterPointers;
	hipMalloc((void**)&d_FilterPointers, L_NumberOf_FILTERS * sizeof(int*));

	float** d_FilterGradientPointers;
	hipMalloc((void**)&d_FilterGradientPointers, L_NumberOf_FILTERS * sizeof(int*));

	float** d_V_Matricies;
	hipMalloc((void**)&d_V_Matricies, L_NumberOf_FILTERS * sizeof(int*));

	float** d_S_Matricies;
	hipMalloc((void**)&d_S_Matricies, L_NumberOf_FILTERS * sizeof(int*));

	float** d_V_CORRECTED_Matricies;
	hipMalloc((void**)&d_V_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(int*));

	float** d_S_CORRECTED_Matricies;
	hipMalloc((void**)&d_S_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(int*));



	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Filters[i], filterByteCount);
		hipMemcpy(h_Filters[i], L_Filters[i], filterByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_FilterGradients[i], filterByteCount);
		hipMemcpy(h_FilterGradients[i], L_Filter_BACKPROP_RESULTS[i], filterByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_V_Matricies[i], filterByteCount);
		hipMemcpy(h_V_Matricies[i], L_AdamOptimizer_V_Matrix[i], filterByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_S_Matricies[i], filterByteCount);
		hipMemcpy(h_S_Matricies[i], L_AdamOptimizer_S_Matrix[i], filterByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_V_CORRECTED_Matricies[i], filterByteCount);
		hipMemcpy(h_V_CORRECTED_Matricies[i], L_AdamOptimizer_Corrected_V_Matrix[i], filterByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_S_CORRECTED_Matricies[i], filterByteCount);
		hipMemcpy(h_S_CORRECTED_Matricies[i], L_AdamOptimizer_Corrected_S_Matrix[i], filterByteCount, hipMemcpyHostToDevice);
	}


	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_FilterPointers, h_Filters, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyHostToDevice);
	hipMemcpy(d_FilterGradientPointers, h_FilterGradients, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyHostToDevice);
	hipMemcpy(d_V_Matricies, h_V_Matricies, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyHostToDevice);
	hipMemcpy(d_S_Matricies, h_S_Matricies, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyHostToDevice);
	hipMemcpy(d_V_CORRECTED_Matricies, h_V_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyHostToDevice);
	hipMemcpy(d_S_CORRECTED_Matricies, h_S_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyHostToDevice);


	dim3 blockGrid(numberOfBlocks_X, 1, 1); ///OK
	dim3 threads(numberOfThreadsPerBlock, 1, 1); ///OK

	TransposeConvFilterUpdateKernel << <blockGrid, threads >> > (d_FilterPointers, d_FilterGradientPointers, d_V_Matricies, d_S_Matricies, d_V_CORRECTED_Matricies, d_S_CORRECTED_Matricies, m_FilterSize, m_HyperParam_Beta1, m_HyperParam_Beta2, m_HyperParam_T, m_HyperParam_alpha, m_HyperParam_Epsilon);
	hipDeviceSynchronize();

	float* temp = new float[filterByteCount];

	hipMemcpy(h_Filters, d_FilterPointers, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyDeviceToHost);
	hipMemcpy(h_FilterGradients, d_FilterGradientPointers, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyDeviceToHost);
	hipMemcpy(h_V_Matricies, d_V_Matricies, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyDeviceToHost);
	hipMemcpy(h_S_Matricies, d_S_Matricies, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyDeviceToHost);
	hipMemcpy(h_V_CORRECTED_Matricies, h_V_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyDeviceToHost);
	hipMemcpy(h_S_CORRECTED_Matricies, h_S_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyDeviceToHost);

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {

		hipMemcpy(temp, h_Filters[i], filterByteCount, hipMemcpyDeviceToHost);
		memcpy(L_Filters[i], temp, filterByteCount);
		hipFree(h_Filters[i]);

		hipMemcpy(temp, h_V_Matricies[i], filterByteCount, hipMemcpyDeviceToHost);
		memcpy(L_AdamOptimizer_V_Matrix[i], temp, filterByteCount);
		hipFree(h_V_Matricies[i]);

		hipMemcpy(temp, h_S_Matricies[i], filterByteCount, hipMemcpyDeviceToHost);
		memcpy(L_AdamOptimizer_S_Matrix[i], temp, filterByteCount);
		hipFree(h_S_Matricies[i]);

		hipMemcpy(temp, h_V_CORRECTED_Matricies[i], filterByteCount, hipMemcpyDeviceToHost);
		memcpy(L_AdamOptimizer_Corrected_V_Matrix[i], temp, filterByteCount);
		hipFree(h_V_CORRECTED_Matricies[i]);

		hipMemcpy(temp, h_S_CORRECTED_Matricies[i], filterByteCount, hipMemcpyDeviceToHost);
		memcpy(L_AdamOptimizer_Corrected_S_Matrix[i], temp, filterByteCount);
		hipFree(h_S_CORRECTED_Matricies[i]);
	}

	hipFree(d_FilterPointers);
	hipFree(d_FilterGradientPointers);
	hipFree(d_V_Matricies);
	hipFree(d_S_Matricies);
	hipFree(h_V_CORRECTED_Matricies);
	hipFree(h_S_CORRECTED_Matricies);

	delete[] h_Filters;
	delete[] h_FilterGradients;
	delete[] h_V_Matricies;
	delete[] h_S_Matricies;
	delete[] h_V_CORRECTED_Matricies;
	delete[] h_S_CORRECTED_Matricies;
}

void TransposeConvolution::LayerBiasUpdate()
{
	int biasMatrixSize = L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH;

	size_t biasMatrixByteCount = biasMatrixSize * sizeof(float);


	int numberOfBlocks_X = L_NumberOf_FILTERS;
	int numberOfThreadsPerBlock = 1;

	// create intermediate host array for storage of device row-pointers

	// create top-level device array pointer
	float** h_Biases = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(int*));
	float** h_BiasGradients = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(int*));
	float** h_V_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(int*));
	float** h_S_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(int*));
	float** h_V_CORRECTED_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(int*));
	float** h_S_CORRECTED_Matricies = new float* [L_NumberOf_FILTERS]; //(float**)malloc(L_NumberOf_FILTERS * sizeof(int*));


	float** d_BiasPointers;
	hipMalloc((void**)&d_BiasPointers, L_NumberOf_FILTERS * sizeof(int*));

	float** d_BiasGradientPointers;
	hipMalloc((void**)&d_BiasGradientPointers, L_NumberOf_FILTERS * sizeof(int*));

	float** d_V_Matricies;
	hipMalloc((void**)&d_V_Matricies, L_NumberOf_FILTERS * sizeof(int*));

	float** d_S_Matricies;
	hipMalloc((void**)&d_S_Matricies, L_NumberOf_FILTERS * sizeof(int*));

	float** d_V_CORRECTED_Matricies;
	hipMalloc((void**)&d_V_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(int*));

	float** d_S_CORRECTED_Matricies;
	hipMalloc((void**)&d_S_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(int*));



	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {
		hipMalloc(&h_Biases[i], biasMatrixByteCount);
		hipMemcpy(h_Biases[i], L_Biases[i], biasMatrixByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_BiasGradients[i], biasMatrixByteCount);
		hipMemcpy(h_BiasGradients[i], L_BACKWARD_Pass_INPUTS[i], biasMatrixByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_V_Matricies[i], biasMatrixByteCount);
		hipMemcpy(h_V_Matricies[i], L_BIAS_AdamOptimizer_V_Matrix[i], biasMatrixByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_S_Matricies[i], biasMatrixByteCount);
		hipMemcpy(h_S_Matricies[i], L_BIAS_AdamOptimizer_S_Matrix[i], biasMatrixByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_V_CORRECTED_Matricies[i], biasMatrixByteCount);
		hipMemcpy(h_V_CORRECTED_Matricies[i], L_BIAS_AdamOptimizer_Corrected_V_Matrix[i], biasMatrixByteCount, hipMemcpyHostToDevice);

		hipMalloc(&h_S_CORRECTED_Matricies[i], biasMatrixByteCount);
		hipMemcpy(h_S_CORRECTED_Matricies[i], L_BIAS_AdamOptimizer_Corrected_S_Matrix[i], biasMatrixByteCount, hipMemcpyHostToDevice);
	}


	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_BiasPointers, h_Biases, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyHostToDevice);
	hipMemcpy(d_BiasGradientPointers, h_BiasGradients, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyHostToDevice);
	hipMemcpy(d_V_Matricies, h_V_Matricies, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyHostToDevice);
	hipMemcpy(d_S_Matricies, h_S_Matricies, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyHostToDevice);
	hipMemcpy(d_V_CORRECTED_Matricies, h_V_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyHostToDevice);
	hipMemcpy(d_S_CORRECTED_Matricies, h_S_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyHostToDevice);


	dim3 blockGrid(numberOfBlocks_X, 1, 1); ///OK
	dim3 threads(numberOfThreadsPerBlock, 1, 1); ///OK

	TransposeConvBiasUpdateKernel << <blockGrid, threads >> > (d_BiasPointers, d_BiasGradientPointers, d_V_Matricies, d_S_Matricies, d_V_CORRECTED_Matricies, d_S_CORRECTED_Matricies, L_FORWARD_OutputLayer_HEIGHT, L_FORWARD_OutputLayer_WIDTH, m_HyperParam_Beta1, m_HyperParam_Beta2, m_HyperParam_T, m_HyperParam_alpha, m_HyperParam_Epsilon);
	hipDeviceSynchronize();

	float* temp = new float[biasMatrixByteCount];

	hipMemcpy(h_Biases, d_BiasPointers, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyDeviceToHost);
	hipMemcpy(h_BiasGradients, d_BiasGradientPointers, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyDeviceToHost);
	hipMemcpy(h_V_Matricies, d_V_Matricies, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyDeviceToHost);
	hipMemcpy(h_S_Matricies, d_S_Matricies, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyDeviceToHost);
	hipMemcpy(h_V_CORRECTED_Matricies, h_V_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyDeviceToHost);
	hipMemcpy(h_S_CORRECTED_Matricies, h_S_CORRECTED_Matricies, L_NumberOf_FILTERS * sizeof(int*), hipMemcpyDeviceToHost);

	for (size_t i = 0; i < L_NumberOf_FILTERS; i++) {

		hipMemcpy(temp, h_Biases[i], biasMatrixByteCount, hipMemcpyDeviceToHost);
		memcpy(L_Biases[i], temp, biasMatrixByteCount);
		hipFree(h_Biases[i]);

		hipMemcpy(temp, h_V_Matricies[i], biasMatrixByteCount, hipMemcpyDeviceToHost);
		memcpy(L_BIAS_AdamOptimizer_V_Matrix[i], temp, biasMatrixByteCount);
		hipFree(h_V_Matricies[i]);

		hipMemcpy(temp, h_S_Matricies[i], biasMatrixByteCount, hipMemcpyDeviceToHost);
		memcpy(L_BIAS_AdamOptimizer_S_Matrix[i], temp, biasMatrixByteCount);
		hipFree(h_S_Matricies[i]);

		hipMemcpy(temp, h_V_CORRECTED_Matricies[i], biasMatrixByteCount, hipMemcpyDeviceToHost);
		memcpy(L_BIAS_AdamOptimizer_Corrected_V_Matrix[i], temp, biasMatrixByteCount);
		hipFree(h_V_CORRECTED_Matricies[i]);

		hipMemcpy(temp, h_S_CORRECTED_Matricies[i], biasMatrixByteCount, hipMemcpyDeviceToHost);
		memcpy(L_BIAS_AdamOptimizer_Corrected_S_Matrix[i], temp, biasMatrixByteCount);
		hipFree(h_S_CORRECTED_Matricies[i]);
	}

	hipFree(d_BiasPointers);
	hipFree(d_BiasGradientPointers);
	hipFree(d_V_Matricies);
	hipFree(d_S_Matricies);
	hipFree(h_V_CORRECTED_Matricies);
	hipFree(h_S_CORRECTED_Matricies);

	delete[] h_Biases;
	delete[] h_BiasGradients;
	delete[] h_V_Matricies;
	delete[] h_S_Matricies;
	delete[] h_V_CORRECTED_Matricies;
	delete[] h_S_CORRECTED_Matricies;
}

void TransposeConvolution::PadInput()
{
	m_PaddedInputHeight = m_InputMatrixHeight + 2 * m_PaddingSize;
	m_PaddedInputWidth = m_InputMatrixWidth + 2 * m_PaddingSize;
	int elementsInPaddedInput = m_PaddedInputHeight * m_PaddedInputWidth;

	m_PaddedInput = new float[elementsInPaddedInput];

	memset(m_PaddedInput, 0, elementsInPaddedInput * sizeof(float));

	float* d_Output;
	float* d_UnpaddedInput;

	size_t outputByteCount = elementsInPaddedInput * sizeof(float);
	size_t unpaddedInputByteCount = (m_InputMatrixHeight * m_InputMatrixWidth) * sizeof(float);

	hipMalloc((void**)&d_Output, outputByteCount);
	hipMalloc((void**)&d_UnpaddedInput, unpaddedInputByteCount);

	hipMemcpy(d_UnpaddedInput, m_InputMatrix, unpaddedInputByteCount, hipMemcpyHostToDevice);
	hipMemcpy(d_Output, m_PaddedInput, unpaddedInputByteCount, hipMemcpyHostToDevice);

	int numberOfBlocks = m_InputMatrixHeight / 2;

	dim3 blockGrid(numberOfBlocks, 1, 1);
	dim3 threads(2, 1, 1);

	PaddingKernel << <blockGrid, threads >> > (d_UnpaddedInput, d_Output, m_PaddedInputWidth, m_InputMatrixWidth, m_InputMatrixHeight);
	hipDeviceSynchronize();

	hipMemcpy(m_PaddedInput, d_Output, outputByteCount, hipMemcpyDeviceToHost);
}

void TransposeConvolution::LayerPadInput()
{
	L_FORWARD_InputLayer_PADDED_HEIGHT = L_FORWARD_InputLayer_HEIGHT + 2 * m_PaddingSize;
	L_FORWARD_InputLayer_PADDED_WIDTH = L_FORWARD_InputLayer_WIDTH + 2 * m_PaddingSize;

	int inputSize = L_FORWARD_InputLayer_HEIGHT * L_FORWARD_InputLayer_WIDTH;

	int outputSize = L_FORWARD_InputLayer_PADDED_HEIGHT * L_FORWARD_InputLayer_PADDED_WIDTH;

	size_t inputByteCount = inputSize * sizeof(float);
	size_t outputByteCount = outputSize * sizeof(float);


	for (int inputNumber = 0; inputNumber < L_FORWARD_NumberOf_INPUTS; ++inputNumber)
	{
		L_FORWARD_Pass_PADDED_INPUTS[inputNumber] = new float[outputSize];
	}

	int numberOfBlockx_X = L_FORWARD_InputLayer_HEIGHT / 2;
	int numberOfBlocks_Y = L_FORWARD_NumberOf_INPUTS;
	int numberOfThreadsPerBlock = 2;

	// create intermediate host array for storage of device row-pointers

	// create top-level device array pointer
	float** h_Inputs = new float* [L_FORWARD_NumberOf_INPUTS];  //(float**)malloc(L_FORWARD_NumberOf_INPUTS * sizeof(int*));
	float** h_Outputs = new float* [L_FORWARD_NumberOf_INPUTS]; //(float**)malloc(L_FORWARD_NumberOf_OUTPUTS * sizeof(int*));


	float** d_InputPointerArray;
	hipMalloc((void**)&d_InputPointerArray, L_FORWARD_NumberOf_INPUTS * sizeof(int*));

	float** d_OutputPointerArray;
	hipMalloc((void**)&d_OutputPointerArray, L_FORWARD_NumberOf_INPUTS * sizeof(int*));


	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_FORWARD_NumberOf_INPUTS; i++) {
		hipMalloc(&h_Inputs[i], inputByteCount);
		hipMemcpy(h_Inputs[i], L_FORWARD_Pass_INPUTS[i], inputByteCount, hipMemcpyHostToDevice);
	}


	for (size_t i = 0; i < L_FORWARD_NumberOf_INPUTS; i++) {
		hipMalloc(&h_Outputs[i], outputByteCount);
		hipMemset(&h_Outputs[i], 0, outputByteCount);
	}

	// fixup top level device array pointer to point to array of device row-pointers
	hipMemcpy(d_InputPointerArray, h_Inputs, L_FORWARD_NumberOf_INPUTS * sizeof(int*), hipMemcpyHostToDevice);
	hipMemcpy(d_OutputPointerArray, h_Outputs, L_FORWARD_NumberOf_INPUTS * sizeof(float*), hipMemcpyHostToDevice);

	dim3 blockGrid(numberOfBlockx_X, numberOfBlocks_Y, 1); ///OK
	dim3 threads(numberOfThreadsPerBlock, 1, 1); ///OK

	LayerTransposeInputPaddingKernel << <blockGrid, threads >> > (d_InputPointerArray, d_OutputPointerArray, L_FORWARD_InputLayer_WIDTH, L_FORWARD_InputLayer_PADDED_WIDTH);
	hipDeviceSynchronize();

	float* temp = new float[outputSize];
	hipMemcpy(h_Outputs, d_OutputPointerArray, L_FORWARD_NumberOf_INPUTS * sizeof(int*), hipMemcpyDeviceToHost);
	for (size_t i = 0; i < L_FORWARD_NumberOf_INPUTS; i++) {

		hipMemcpy(temp, h_Outputs[i], outputByteCount, hipMemcpyDeviceToHost);
		memcpy(L_FORWARD_Pass_PADDED_INPUTS[i], temp, outputByteCount);
		hipFree(h_Outputs[i]);
	}
	hipFree(d_OutputPointerArray);
	delete[] h_Outputs;

	// allocate each device row-pointer, then copy host data to it
	for (size_t i = 0; i < L_BACKWARD_NumberOf_INPUTS; i++) {
		hipFree(h_Inputs[i]);
	}
	hipFree(d_InputPointerArray);
	delete h_Inputs;
}

void TransposeConvolution::InitializeFilter()
{
	std::random_device rd{};
	std::mt19937 gen{ rd() };
	std::normal_distribution<> distribution{ 1.42,2 };
	m_Filter = new float[m_FilterSize * m_FilterSize];

	for (int i = 0; i < m_FilterSize * m_FilterSize; ++i)
	{
		m_Filter[i] = distribution(gen);
	}
}

void TransposeConvolution::LayerFilterInitialization()
{
	std::random_device rd{};
	std::mt19937 gen{ rd() };
	std::normal_distribution<> distribution{ 1,2 };

	int filterElementCount = m_FilterSize * m_FilterSize;

	for (int filterNumber = 0; filterNumber < L_NumberOf_FILTERS; ++filterNumber)
	{
		L_Filters[filterNumber] = new float[filterElementCount];

		for (int i = 0; i < filterElementCount; ++i)
		{
			L_Filters[filterNumber][i] = i + 1; //distribution(gen);
		}
	}
}

void TransposeConvolution::FlipFilter()
{
	int filterArraySize = m_FilterSize * m_FilterSize;
	m_FlippedFilter = new float[filterArraySize];

	int k = 0;

	//Loop from back and assign value to new array
	for (int i = filterArraySize - 1; i >= 0; ) {
		m_FlippedFilter[k++] = m_Filter[i--];
	}
}

void TransposeConvolution::DebugPrintAll()
{
	int newLineCounter = 1;

	cout << "========================================================" << endl;
	cout << "============ Transpose Conv Debug Print All ============" << endl;
	cout << "========================================================" << endl;

	cout << "Squishy: " << endl;
	cout << "Layer ID: " << layerID << endl;
	cout << "Level ID: " << levelID << endl;
	cout << "Hyper parameters: " << endl;
	cout << "Beta 1: " << m_HyperParam_Beta1 << endl;
	cout << "Beta 2: " << m_HyperParam_Beta2 << endl;
	cout << "Epsilon: " << m_HyperParam_Epsilon << endl;
	cout << "Alpha: " << m_HyperParam_alpha << endl;
	cout << "T: " << m_HyperParam_T << endl;


	cout << ">>>> Forward Inputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_FORWARD_NumberOf_INPUTS; ++inputIndex)
	{
		cout << "--- Element " << inputIndex + 1 << "---" << endl;
		for (int elementIndex = 0; elementIndex < L_FORWARD_InputLayer_HEIGHT * L_FORWARD_InputLayer_WIDTH; ++elementIndex)
		{
			cout << L_FORWARD_Pass_INPUTS[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == L_FORWARD_InputLayer_WIDTH + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Normal Filter Inputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_Filters[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}


	cout << ">>>> Flipped Filter Inputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_FLIPPED_Filters[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}


	cout << ">>>> Forward Outputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_FORWARD_NumberOf_OUTPUTS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH; ++elementIndex)
		{
			cout << L_FORWARD_Pass_OUTPUTS[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == L_FORWARD_OutputLayer_WIDTH + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Backward Inputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_BACKWARD_NumberOf_INPUTS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < L_BACKWARD_InputLayer_HEIGHT * L_BACKWARD_InputLayer_WIDTH; ++elementIndex)
		{
			cout << L_BACKWARD_Pass_INPUTS[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == L_BACKWARD_InputLayer_WIDTH + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Padded Backward Inputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_BACKWARD_NumberOf_INPUTS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < L_FORWARD_InputLayer_PADDED_HEIGHT * L_FORWARD_InputLayer_PADDED_WIDTH; ++elementIndex)
		{
			cout << L_FORWARD_Pass_PADDED_INPUTS[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == L_FORWARD_InputLayer_PADDED_WIDTH + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Backward Outputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_BACKWARD_NumberOf_OUTPUTS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < L_BACKWARD_OutputLayer_HEIGHT * L_BACKWARD_OutputLayer_WIDTH; ++elementIndex)
		{
			cout << L_BACKWARD_Pass_OUTPUTS[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == L_BACKWARD_OutputLayer_WIDTH + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Filter Backprop Outputs <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_Filter_BACKPROP_RESULTS[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Bias Outputs Before Update <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH; ++elementIndex)
		{
			cout << L_PrevBiases[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == L_BACKWARD_OutputLayer_WIDTH + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}


	cout << ">>>> Bias Outputs After Update <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < L_FORWARD_OutputLayer_HEIGHT * L_FORWARD_OutputLayer_WIDTH; ++elementIndex)
		{
			cout << L_Biases[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == L_FORWARD_OutputLayer_WIDTH + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Filter Adam Optimizer V Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_AdamOptimizer_V_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Filter Adam Optimizer >CORRECTED< V Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_AdamOptimizer_Corrected_V_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Filter Adam Optimizer S Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_AdamOptimizer_S_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> Filter Adam Optimizer >CORRECTED< S Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_AdamOptimizer_Corrected_S_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> BIAS Adam Optimizer V Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_BIAS_AdamOptimizer_V_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> BIAS Adam Optimizer >CORRECTED< V Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_BIAS_AdamOptimizer_Corrected_V_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> BIAS Adam Optimizer S Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_BIAS_AdamOptimizer_S_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

	cout << ">>>> BIAS Adam Optimizer >CORRECTED< S Matrix <<<<" << endl << endl;

	for (int inputIndex = 0; inputIndex < L_NumberOf_FILTERS; ++inputIndex)
	{
		cout << "- Element " << inputIndex + 1 << "-" << endl;
		for (int elementIndex = 0; elementIndex < m_FilterSize * m_FilterSize; ++elementIndex)
		{
			cout << L_BIAS_AdamOptimizer_Corrected_S_Matrix[inputIndex][elementIndex] << " ";
			newLineCounter++;
			if (newLineCounter == m_FilterSize + 1)
			{
				cout << endl;
				newLineCounter = 1;
			}
		}
		cout << endl;
	}

}
